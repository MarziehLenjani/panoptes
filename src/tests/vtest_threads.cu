#include "hip/hip_runtime.h"
/**
 * Panoptes - A Binary Translation Framework for CUDA
 * (c) 2011-2012 Chris Kennelly <chris@ckennelly.com>
 *
 * This program is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.
 */

#include <boost/thread/barrier.hpp>
#include <boost/thread/thread.hpp>
#include <hip/hip_runtime.h>
#include <gtest/gtest.h>

extern "C" __global__ void k_copy(int * out, const int * in, int n) {
    for (int i = threadIdx.x + blockIdx.x * blockDim.x; i < n;
            i += blockDim.x * gridDim.x) {
        out[i] = in[i];
    }
}

typedef std::vector<void *> ptr_vector_t;

struct worker_data {
    ptr_vector_t * addresses;
    boost::barrier * barrier;
    size_t rank;
};

void worker(worker_data * data) {
    /* Initialize. */
    hipError_t ret;
    hipStream_t stream;

    ret = hipStreamCreate(&stream);
    ASSERT_EQ(hipSuccess, ret);

    const int n_ints = 1 << 20;
    int * mem;
    ret = hipMalloc((void **) &mem, sizeof(*mem) * n_ints);
    ASSERT_EQ(hipSuccess, ret);

    const size_t others = data->addresses->size();
    (*data->addresses)[data->rank] = mem;

    /* Wait. */
    data->barrier->wait();

    for (size_t i = 0; i < 16; i++) {
        const size_t remote = (data->rank + i) % others;
        const int * remote_addr =
            static_cast<const int *>((*data->addresses)[remote]);

        k_copy<<<256, 32, 0, stream>>>(mem, remote_addr, n_ints);

        ret = hipStreamSynchronize(stream);
        EXPECT_EQ(hipSuccess, ret);
    }

    ret = hipStreamDestroy(stream);
    ASSERT_EQ(hipSuccess, ret);

    /* Wait. */
    data->barrier->wait();
    ret = hipFree(mem);
    ASSERT_EQ(hipSuccess, ret);
}

TEST(Threads, PingPong) {
    const size_t n_threads = 2;

    ptr_vector_t addresses(n_threads, NULL);
    boost::barrier barrier(n_threads);

    worker_data default_data;
    default_data.addresses = &addresses;
    default_data.barrier   = &barrier;

    std::vector<worker_data> data(n_threads, default_data);
    std::vector<boost::thread *> threads(n_threads);

    /* Start workers. */
    for (size_t i = 0; i < n_threads; i++) {
        data[i].rank = i;

        threads[i] = new boost::thread(worker, &data[i]);
    }

    /* Stop workers. */
    for (size_t i = 0; i < n_threads; i++) {
        threads[i]->join();
        delete threads[i];
    }
}

int main(int argc, char **argv) {
    ::testing::InitGoogleTest(&argc, argv);
    return RUN_ALL_TESTS();
}
