/**
 * Panoptes - A Binary Translation Framework for CUDA
 * (c) 2011-2012 Chris Kennelly <chris@ckennelly.com>
 *
 * This program is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.
 */

#include <boost/scoped_array.hpp>
#include <boost/static_assert.hpp>
#include <hip/hip_runtime.h>
#include <gtest/gtest.h>
#include <stdint.h>
#include <valgrind/memcheck.h>
#include <cstdio>

TEST(EventSynchronize, SynchronizeAfterDestroy) {
    ::testing::FLAGS_gtest_death_test_style = "threadsafe";

    hipError_t ret;
    hipEvent_t event;
    hipStream_t stream;

    ret = hipEventCreate(&event);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipStreamCreate(&stream);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipEventRecord(event, stream);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipEventDestroy(event);
    EXPECT_EQ(hipSuccess, ret);

    int version;
    ret = hipRuntimeGetVersion(&version);
    ASSERT_EQ(hipSuccess, ret);

    if (version >= 4010 /* 4.1 */) {
        ret = hipEventSynchronize(event);
        EXPECT_EQ(hipErrorUnknown, ret);
    } else {
        EXPECT_EXIT(
            hipEventSynchronize(event),
            ::testing::KilledBySignal(SIGSEGV), "");
    }

    ret = hipStreamDestroy(stream);
    EXPECT_EQ(hipSuccess, ret);
}

int main(int argc, char **argv) {
    ::testing::InitGoogleTest(&argc, argv);
    return RUN_ALL_TESTS();
}
