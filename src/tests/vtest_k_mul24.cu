#include "hip/hip_runtime.h"
/**
 * Panoptes - A Binary Translation Framework for CUDA
 * (c) 2011-2012 Chris Kennelly <chris@ckennelly.com>
 *
 * This program is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.
 */

#include <hip/hip_runtime.h>
#include <gtest/gtest.h>
#include <stdint.h>

__global__ void k_mul24(const int32_t * x, const int32_t * y,
        int32_t * out, int32_t n) {
    for (int32_t i = threadIdx.x + blockIdx.x * blockDim.x;
            i < n; i += blockDim.x * gridDim.x) {
        out[i] = __mul24(x[i], y[i]);
    }
}

__global__ void k_umul24(const uint32_t * x, const uint32_t * y,
        uint32_t * out, int32_t n) {
    for (int32_t i = threadIdx.x + blockIdx.x * blockDim.x;
            i < n; i += blockDim.x * gridDim.x) {
        out[i] = __umul24(x[i], y[i]);
    }
}

TEST(kMul24, Signed) {
    hipError_t ret;
    hipStream_t stream;

    const int32_t n = 1 << 24;
    int32_t * x;
    int32_t * y;
    int32_t * out;

    ret = hipMalloc((void **) &x, sizeof(*x) * n);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipMalloc((void **) &y, sizeof(*y) * n);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipMalloc((void **) &out, sizeof(*out) * n);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipStreamCreate(&stream);
    ASSERT_EQ(hipSuccess, ret);

    k_mul24<<<256, 16, 0, stream>>>(x, y, out, n);

    ret = hipStreamSynchronize(stream);
    EXPECT_EQ(hipSuccess, ret);

    ret = hipStreamDestroy(stream);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipFree(x);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipFree(y);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipFree(out);
    ASSERT_EQ(hipSuccess, ret);
}

TEST(kMul24, Unsigned) {
    hipError_t ret;
    hipStream_t stream;

    const int32_t n = 1 << 24;
    uint32_t * x;
    uint32_t * y;
    uint32_t * out;

    ret = hipMalloc((void **) &x, sizeof(*x) * n);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipMalloc((void **) &y, sizeof(*y) * n);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipMalloc((void **) &out, sizeof(*out) * n);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipStreamCreate(&stream);
    ASSERT_EQ(hipSuccess, ret);

    k_umul24<<<256, 16, 0, stream>>>(x, y, out, n);

    ret = hipStreamSynchronize(stream);
    EXPECT_EQ(hipSuccess, ret);

    ret = hipStreamDestroy(stream);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipFree(x);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipFree(y);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipFree(out);
    ASSERT_EQ(hipSuccess, ret);
}

int main(int argc, char **argv) {
    ::testing::InitGoogleTest(&argc, argv);
    return RUN_ALL_TESTS();
}
