#include "hip/hip_runtime.h"
/**
 * Panoptes - A Binary Translation Framework for CUDA
 * (c) 2011-2012 Chris Kennelly <chris@ckennelly.com>
 *
 * This program is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.
 */

#include <hip/hip_runtime.h>
#include <gtest/gtest.h>

extern "C" __global__ void k_copysignf(const float * a, const float * b,
        float * out, int n) {
    for (int idx = threadIdx.x + blockDim.x * blockIdx.x;
            idx < n; idx += blockDim.x * gridDim.x) {
        const float _a = a[idx];
        const float _b = b[idx];
        float _out;
        /* copysignf does not do the right thing, so for now, use inline PTX */
        asm("copysign.f32 %0, %1, %2;" : "=f"(_out) : "f"(_a), "f"(_b));

        out[idx] = _out;
    }
}

extern "C" __global__ void k_copysign(const double * a, const double * b,
        double * out, int n) {
    for (int idx = threadIdx.x + blockDim.x * blockIdx.x;
            idx < n; idx += blockDim.x * gridDim.x) {
        const double _a = a[idx];
        const double _b = b[idx];
        double _out;

        /* copysign does not do the right thing, so for now, use inline PTX */
        asm("copysign.f64 %0, %1, %2;" : "=d"(_out) : "d"(_a), "d"(_b));

        out[idx] = _out;
    }
}

TEST(kCopySign, SinglePrecision) {
    hipError_t ret;
    hipStream_t stream;

    const int N = 1 << 20;
    const int n_blocks = 32;

    float * a;
    float * b;
    float * out;

    ret = hipMalloc((void **) &a, sizeof(*a) * N);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipMalloc((void **) &b, sizeof(*b) * N);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipMalloc((void **) &out, sizeof(*out) * N);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipStreamCreate(&stream);
    ASSERT_EQ(hipSuccess, ret);

    k_copysignf<<<256, n_blocks, 0, stream>>>(a, b, out, N);

    ret = hipStreamSynchronize(stream);
    EXPECT_EQ(hipSuccess, ret);

    ret = hipStreamDestroy(stream);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipFree(a);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipFree(b);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipFree(out);
    ASSERT_EQ(hipSuccess, ret);
}

TEST(kCopySign, DoublePrecision) {
    hipError_t ret;
    hipStream_t stream;

    const int N = 1 << 20;
    const int n_blocks = 32;

    double * a;
    double * b;
    double * out;

    ret = hipMalloc((void **) &a, sizeof(*a) * N);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipMalloc((void **) &b, sizeof(*b) * N);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipMalloc((void **) &out, sizeof(*out) * N);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipStreamCreate(&stream);
    ASSERT_EQ(hipSuccess, ret);

    k_copysign<<<256, n_blocks, 0, stream>>>(a, b, out, N);

    ret = hipStreamSynchronize(stream);
    EXPECT_EQ(hipSuccess, ret);

    ret = hipStreamDestroy(stream);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipFree(a);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipFree(b);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipFree(out);
    ASSERT_EQ(hipSuccess, ret);
}

int main(int argc, char **argv) {
    ::testing::InitGoogleTest(&argc, argv);
    return RUN_ALL_TESTS();
}
