#include "hip/hip_runtime.h"
/**
 * Panoptes - A Binary Translation Framework for CUDA
 * (c) 2011-2012 Chris Kennelly <chris@ckennelly.com>
 *
 * This program is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.
 */

#include <hip/hip_runtime.h>
#include <gtest/gtest.h>
#include <stdint.h>

enum {
    BDIM = 256
};

extern "C" __global__ void k_shuffle(const uint32_t * in,
        const int N, uint32_t * out) {
    __shared__ volatile uint32_t buf[BDIM];

    for (int idx = threadIdx.x + blockDim.x * blockIdx.x;
            idx < N; idx += blockDim.x * gridDim.x) {
        buf[threadIdx.x] = in[idx];
        __syncthreads();
        out[idx] = buf[blockDim.x - threadIdx.x];
    }
}

TEST(kSyncThreads, Shuffle) {
    hipError_t ret;
    hipStream_t stream;

    const int N = 1 << 20;
    const int n_blocks = 32;

    uint32_t * in;
    uint32_t * out;

    ret = hipMalloc((void **) &in, sizeof(*in) * N);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipMalloc((void **) &out, sizeof(*out) * N);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipStreamCreate(&stream);
    ASSERT_EQ(hipSuccess, ret);

    k_shuffle<<<BDIM, n_blocks, 0, stream>>>(in, N, out);

    ret = hipStreamSynchronize(stream);
    EXPECT_EQ(hipSuccess, ret);

    ret = hipStreamDestroy(stream);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipFree(in);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipFree(out);
    ASSERT_EQ(hipSuccess, ret);
}

int main(int argc, char **argv) {
    ::testing::InitGoogleTest(&argc, argv);
    return RUN_ALL_TESTS();
}
