/**
 * Panoptes - A Binary Translation Framework for CUDA
 * (c) 2011-2012 Chris Kennelly <chris@ckennelly.com>
 *
 * This program is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.
 */

#include <boost/static_assert.hpp>
#include <hip/hip_runtime.h>
#include <gtest/gtest.h>
#include <stdint.h>
#include <unistd.h>
#include <valgrind/memcheck.h>

/**
 * The CUDA documentation for hipMemcpyAsync notes that any host pointers
 * provided to the call must be page-locked.  Pageable addresses cause the
 * call to "return an error."
 *
 * That behavior is not evidenced by this set of tests, so Panoptes treats
 * the pointers just as it would any other.
 */

TEST(MemcpyAsync, CheckReturnValues) {
    /**
     * The API documentation states that
     * hipErrorInvalidDevicePointer is a valid return value for
     * hipMemcpyAsync
     *
     * TODO;  This needs a test.
     */
    hipError_t ret;
    hipStream_t stream;

    ret = hipStreamCreate(&stream);
    ASSERT_EQ(hipSuccess, ret);

    /**
     * Test woefully out of range directions.
     */
    int a = 0;
    ret = hipMemcpyAsync(&a,   &a,   sizeof(a), (hipMemcpyKind) -1, stream);
    EXPECT_EQ(hipErrorInvalidMemcpyDirection, ret);

    ret = hipMemcpyAsync(NULL, NULL, sizeof(a), (hipMemcpyKind) -1, stream);
    EXPECT_EQ(hipErrorInvalidMemcpyDirection, ret);

    ret = hipStreamSynchronize(stream);
    EXPECT_EQ(hipSuccess, ret);

    ret = hipStreamDestroy(stream);
    EXPECT_EQ(hipSuccess, ret);
}

/**
 * CUDA4 introduced the hipMemcpyDefault direction to hipMemcpy.
 */
TEST(MemcpyAsync, CheckDefaultDirection) {
    hipError_t ret;
    hipStream_t stream;

    ret = hipStreamCreate(&stream);
    ASSERT_EQ(hipSuccess, ret);

    int a1 = 0;
    int a2 = 0;
    int * b;
    ret = hipMalloc((void**) &b, sizeof(*b));
    ASSERT_EQ(hipSuccess, ret);

    ret = hipMemcpyAsync(&a1,   &a2,  sizeof(a1), hipMemcpyDefault, stream);
    EXPECT_EQ(hipSuccess, ret);

    ret = hipMemcpyAsync(&a1,    b,   sizeof(a1), hipMemcpyDefault, stream);
    EXPECT_EQ(hipSuccess, ret);

    ret = hipMemcpyAsync( b,    &a1,  sizeof(a1), hipMemcpyDefault, stream);
    EXPECT_EQ(hipSuccess, ret);

    ret = hipMemcpyAsync( b,    b,    sizeof(a1), hipMemcpyDefault, stream);
    EXPECT_EQ(hipSuccess, ret);

    ret = hipStreamSynchronize(stream);
    EXPECT_EQ(hipSuccess, ret);

    ret = hipFree(b);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipStreamDestroy(stream);
    EXPECT_EQ(hipSuccess, ret);
}

/**
 * This test only performs copies in valid directions as to avoid upsetting
 * Valgrind.  The error-causing tests are in test_memcpy.cu.
 */
TEST(MemcpyAsync, AllDirections) {
    hipError_t ret;
    hipStream_t stream;

    ret = hipStreamCreate(&stream);
    ASSERT_EQ(hipSuccess, ret);

    int a1 = 0;
    int a2 = 0;
    int * b;
    ret = hipMalloc((void**) &b, sizeof(*b) * 2);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipMemcpyAsync(&a1,    &a2,    sizeof(a1),
        hipMemcpyHostToHost, stream);
    EXPECT_EQ(hipSuccess, ret);

    ret = hipMemcpyAsync(&a1,     b + 0, sizeof(a1),
        hipMemcpyDeviceToHost, stream);
    EXPECT_EQ(hipSuccess, ret);

    ret = hipMemcpyAsync(&a1,     b + 1, sizeof(a1),
        hipMemcpyDeviceToHost, stream);
    EXPECT_EQ(hipSuccess, ret);

    ret = hipMemcpyAsync( b + 0, &a1,    sizeof(a1),
        hipMemcpyHostToDevice, stream);
    EXPECT_EQ(hipSuccess, ret);

    ret = hipMemcpyAsync( b + 1, &a1,    sizeof(a1),
        hipMemcpyHostToDevice, stream);
    EXPECT_EQ(hipSuccess, ret);

    ret = hipMemcpyAsync( b + 0,  b + 0, sizeof(a1),
        hipMemcpyDeviceToDevice, stream);
    EXPECT_EQ(hipSuccess, ret);

    ret = hipMemcpyAsync( b + 1,  b + 1, sizeof(a1),
        hipMemcpyDeviceToDevice, stream);
    EXPECT_EQ(hipSuccess, ret);

    ret = hipStreamSynchronize(stream);
    EXPECT_EQ(hipSuccess, ret);

    ret = hipFree(b);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipStreamDestroy(stream);
    EXPECT_EQ(hipSuccess, ret);
}

TEST(MemcpyAsync, Validity) {
    hipError_t ret;
    hipStream_t stream;

    ret = hipStreamCreate(&stream);
    ASSERT_EQ(hipSuccess, ret);

    int * device_ptr, src = 0, vsrc, dst, vdst;

    ret = hipMalloc((void **) &device_ptr, sizeof(*device_ptr));
    ASSERT_EQ(hipSuccess, ret);

    /* Only src is valid; *device_ptr and dst are invalid. */

    /* Do transfer */
    ret = hipMemcpyAsync(device_ptr, &src, sizeof(src),
        hipMemcpyHostToDevice, stream);
    ASSERT_EQ(hipSuccess, ret);

    /* Both src and *device_ptr are valid; dst is invalid */
    ret = hipMemcpyAsync(&dst, device_ptr, sizeof(dst),
        hipMemcpyDeviceToHost, stream);
    ASSERT_EQ(hipSuccess, ret);

    EXPECT_EQ(src, dst);

    int valgrind = VALGRIND_GET_VBITS(&src, &vsrc, sizeof(src));
    assert(valgrind == 0 || valgrind == 1);

    if (valgrind == 1) {
        valgrind = VALGRIND_GET_VBITS(&dst, &vdst, sizeof(dst));
        assert(valgrind == 1);

        EXPECT_EQ(vsrc, vdst);
    }

    ret = hipStreamSynchronize(stream);
    EXPECT_EQ(hipSuccess, ret);

    ret = hipFree(device_ptr);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipStreamDestroy(stream);
    ASSERT_EQ(hipSuccess, ret);
}

TEST(MemcpyAsync, Pinned) {
    /**
     * Host memory must be pinned in order to be used as an argument to
     * hipMemcpyAsync.  Panoptes only prints a warning about this error
     * rather than actually return an error via the CUDA API.  This test is
     * written as to check for the absence of an error once the CUDA
     * implementation starts returning one for nonpinned host memory.
     */
    const long page_size_ = sysconf(_SC_PAGESIZE);
    ASSERT_LT(0, page_size_);
    const size_t page_size = page_size_;

    const size_t pages = 3;
    assert(pages > 0);

    hipError_t ret;
    hipStream_t stream;

    uint8_t *device_ptr, *host_ptr;
    ret = hipMalloc((void **) &device_ptr, pages * page_size);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipHostMalloc((void **) &host_ptr, pages * page_size);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipStreamCreate(&stream);
    ASSERT_EQ(hipSuccess, ret);

    /* Page aligned transfers */
    for (size_t i = 0; i < pages; i++) {
        for (size_t j = i; j < pages; j++) {
            ret = hipMemcpyAsync(device_ptr, host_ptr + i * page_size,
                (pages - j) * page_size, hipMemcpyHostToDevice, stream);
            EXPECT_EQ(hipSuccess, ret);

            ret = hipMemcpyAsync(host_ptr + i * page_size, device_ptr,
                (pages - j) * page_size, hipMemcpyDeviceToHost, stream);
            EXPECT_EQ(hipSuccess, ret);
        }
    }

    /* Try a nonaligned transfer. */
    ret = hipMemcpyAsync(device_ptr, host_ptr + (page_size / 2),
        page_size / 2, hipMemcpyHostToDevice, stream);

    ret = hipStreamSynchronize(stream);
    EXPECT_EQ(hipSuccess, ret);

    ret = hipStreamDestroy(stream);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipHostFree(host_ptr);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipFree(device_ptr);
    ASSERT_EQ(hipSuccess, ret);
}

int main(int argc, char **argv) {
    ::testing::InitGoogleTest(&argc, argv);
    return RUN_ALL_TESTS();
}
