#include "hip/hip_runtime.h"
/**
 * Panoptes - A Binary Translation Framework for CUDA
 * (c) 2011-2012 Chris Kennelly <chris@ckennelly.com>
 *
 * This program is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.
 */

#include <boost/scoped_array.hpp>
#include <boost/static_assert.hpp>
#include <hip/hip_runtime.h>
#include <gtest/gtest.h>
#include <stdint.h>
#include <valgrind/memcheck.h>
#include <cstdio>

extern "C" __global__ void k_noop() {

}

TEST(kNOOP, SetCacheConfig) {
    hipError_t ret;

    ret = hipFuncSetCacheConfig(reinterpret_cast<const void*>(k_noop), hipFuncCachePreferNone);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipFuncSetCacheConfig(reinterpret_cast<const void*>(k_noop), hipFuncCachePreferShared);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipFuncSetCacheConfig(reinterpret_cast<const void*>(k_noop), hipFuncCachePreferL1);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipFuncSetCacheConfig(reinterpret_cast<const void*>(k_noop), hipFuncCachePreferEqual);
    ASSERT_EQ(hipSuccess, ret);
}

int main(int argc, char **argv) {
    ::testing::InitGoogleTest(&argc, argv);
    return RUN_ALL_TESTS();
}
