/**
 * Panoptes - A Binary Translation Framework for CUDA
 * (c) 2011-2012 Chris Kennelly <chris@ckennelly.com>
 *
 * This program is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.
 */

#include <boost/static_assert.hpp>
#include <hip/hip_runtime.h>
#include <gtest/gtest.h>

TEST(StreamSynchronize, NoWork) {
    hipError_t ret;
    hipStream_t stream;

    ret = hipStreamCreate(&stream);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipStreamSynchronize(stream);
    EXPECT_EQ(hipSuccess, ret);

    ret = hipStreamDestroy(stream);
    EXPECT_EQ(hipSuccess, ret);
}

TEST(StreamSynchronize, Work) {
    hipError_t ret;
    hipStream_t stream;

    int * src;
    int * dst;
    const size_t allocation_size = 1u << 20;
    ret = hipMalloc((void **) &src, allocation_size * sizeof(*src));
    ASSERT_EQ(hipSuccess, ret);

    ret = hipMalloc((void **) &dst, allocation_size * sizeof(*dst));
    ASSERT_EQ(hipSuccess, ret);

    BOOST_STATIC_ASSERT(sizeof(*src) == sizeof(*dst));

    ret = hipStreamCreate(&stream);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipStreamSynchronize(stream);
    EXPECT_EQ(hipSuccess, ret);

    ret = hipMemcpyAsync(dst, src, allocation_size * sizeof(*src),
        hipMemcpyDeviceToDevice, stream);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipStreamSynchronize(stream);
    EXPECT_EQ(hipSuccess, ret);

    ret = hipStreamDestroy(stream);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipFree(dst);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipFree(src);
    ASSERT_EQ(hipSuccess, ret);
}

int main(int argc, char **argv) {
    ::testing::InitGoogleTest(&argc, argv);
    return RUN_ALL_TESTS();
}
