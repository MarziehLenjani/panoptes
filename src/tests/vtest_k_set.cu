#include "hip/hip_runtime.h"
/**
 * Panoptes - A Binary Translation Framework for CUDA
 * (c) 2011-2012 Chris Kennelly <chris@ckennelly.com>
 *
 * This program is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.
 */

#include <hip/hip_runtime.h>
#include <gtest/gtest.h>
#include <stdint.h>

__global__ void k_set_with_immediate(uint32_t * out) {
    uint32_t out_;
    asm volatile("set.ne.u32.u32 %0, 0, 0;\n" : "=r"(out_));
    *out = out_;
}

TEST(Regression, SetWithImmediates) {
    hipError_t ret;

    uint32_t * out;
    ret = hipMalloc((void **) &out, sizeof(*out));
    ASSERT_EQ(hipSuccess, ret);

    hipStream_t stream;

    ret = hipStreamCreate(&stream);
    ASSERT_EQ(hipSuccess, ret);

    k_set_with_immediate<<<1, 1, 0, stream>>>(out);

    ret = hipStreamSynchronize(stream);
    EXPECT_EQ(hipSuccess, ret);

    ret = hipStreamDestroy(stream);
    ASSERT_EQ(hipSuccess, ret);

    uint32_t hout;
    ret = hipMemcpy(&hout, out, sizeof(hout), hipMemcpyDeviceToHost);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipFree(out);
    ASSERT_EQ(hipSuccess, ret);

    EXPECT_EQ(0x0, hout);
}

__global__ void k_cpred(bool * out) {
    uint32_t out_;

    asm volatile(
        "{\n"
        "  .reg .pred %tmp;\n"
        "  mov.pred %tmp, 1;\n"
        "  set.ne.or.u32.u32 %0, 0, 0, %tmp;\n"
        "}\n" : "=r"(out_));

    *out = out_;
}

TEST(Regression, cPredicate) {
    hipError_t ret;

    bool * out;
    ret = hipMalloc((void **) &out, sizeof(*out));
    ASSERT_EQ(hipSuccess, ret);

    hipStream_t stream;

    ret = hipStreamCreate(&stream);
    ASSERT_EQ(hipSuccess, ret);

    k_cpred<<<1, 1, 0, stream>>>(out);

    ret = hipStreamSynchronize(stream);
    EXPECT_EQ(hipSuccess, ret);

    ret = hipStreamDestroy(stream);
    ASSERT_EQ(hipSuccess, ret);

    bool hout;
    ret = hipMemcpy(&hout, out, sizeof(hout), hipMemcpyDeviceToHost);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipFree(out);
    ASSERT_EQ(hipSuccess, ret);

    EXPECT_TRUE(hout);
}

int main(int argc, char **argv) {
    ::testing::InitGoogleTest(&argc, argv);
    return RUN_ALL_TESTS();
}
