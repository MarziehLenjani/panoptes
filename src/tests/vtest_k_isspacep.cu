#include "hip/hip_runtime.h"
/**
 * Panoptes - A Binary Translation Framework for CUDA
 * (c) 2011-2013 Chris Kennelly <chris@ckennelly.com>
 *
 * This program is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.
 */

#include "common.h"
#include <hip/hip_runtime.h>
#include <gtest/gtest.h>

__global__ void k_is_global(bool * out, const void * ptr) {
    int ret;
    asm volatile(
        "{ .reg .pred %tmp;\n"
        "isspacep.global %tmp, %1;\n"
        "selp.s32 %0, 1, 0, %tmp;}\n" :
        "=r"(ret) : PTRC(ptr));
    *out = ret;
}

__global__ void k_is_local(bool * out, const void * ptr) {
    int ret;
    asm volatile(
        "{ .reg .pred %tmp;\n"
        "isspacep.local %tmp, %1;\n"
        "selp.s32 %0, 1, 0, %tmp;}\n" :
        "=r"(ret) : PTRC(ptr));
    *out = ret;
}

__global__ void k_is_shared(bool * out, const void * ptr) {
    int ret;
    asm volatile(
        "{ .reg .pred %tmp;\n"
        "isspacep.shared %tmp, %1;\n"
        "selp.s32 %0, 1, 0, %tmp;}\n" :
        "=r"(ret) : PTRC(ptr));
    *out = ret;
}

__global__ void k_is_global_const(bool * out) {
    int ret;
    asm volatile(
        "{ .reg .pred %tmp;\n"
        "isspacep.global %tmp, 0;\n"
        "selp.s32 %0, 1, 0, %tmp;}\n" : "=r"(ret));
    *out = ret;
}

TEST(IsSpacePTest, Single) {
    hipError_t ret;
    hipStream_t stream;

    bool * d;
    ret = hipMalloc((void **) &d, 4 * sizeof(*d));
    ASSERT_EQ(hipSuccess, ret);

    ret = hipStreamCreate(&stream);
    ASSERT_EQ(hipSuccess, ret);

    k_is_global      <<<1, 1, 0, stream>>>(d + 0, d);
    k_is_global_const<<<1, 1, 0, stream>>>(d + 1);
    k_is_local       <<<1, 1, 0, stream>>>(d + 2, d);
    k_is_shared      <<<1, 1, 0, stream>>>(d + 3, d);

    ret = hipStreamSynchronize(stream);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipStreamDestroy(stream);
    ASSERT_EQ(hipSuccess, ret);

    bool hd[4];
    ret = hipMemcpy(&hd, d, sizeof(hd), hipMemcpyDeviceToHost);
    ASSERT_EQ(hipSuccess, ret);

    int version;
    ret = hipRuntimeGetVersion(&version);
    ASSERT_EQ(hipSuccess, ret);

    if (version != 5000 /* 5.0 */) {
        /**
         * As of CUDA 5.0, this particular test fails while running NATIVELY.
         * While we could compare against the version and modify our
         * expectation accordingly (for the sake of ensuring that programs
         * cannot tell whether they are being run under Panoptes), the code
         * generated Panoptes (and then compiled via the driver API) *works*.
         *
         * To complicate things further, a very similar test in GPU Ocelot
         * fails with CUDA 5.0 and a GTX480:
         *
         * https://code.google.com/p/gpuocelot/source/browse/trunk/ocelot/ocelot/cuda/test/driver/generic.cpp
         */
        EXPECT_FALSE(hd[0]);
    }
    EXPECT_TRUE(hd[1]);
    EXPECT_FALSE(hd[2]);
    EXPECT_FALSE(hd[3]);

    ret = hipFree(d);
    ASSERT_EQ(hipSuccess, ret);
}

/**
 * TODO:  Add a validity check to see that we propagate invalid bits.
 */

int main(int argc, char **argv) {
    ::testing::InitGoogleTest(&argc, argv);
    return RUN_ALL_TESTS();
}
