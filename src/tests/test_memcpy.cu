/**
 * Panoptes - A Binary Translation Framework for CUDA
 * (c) 2011-2012 Chris Kennelly <chris@ckennelly.com>
 *
 * This program is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.
 */

#include <boost/static_assert.hpp>
#include <hip/hip_runtime.h>
#include <gtest/gtest.h>
#include <stdint.h>
#include <valgrind/memcheck.h>
#include <cstdio>

TEST(MemcpyDeathTest, AllDirections) {
    ::testing::FLAGS_gtest_death_test_style = "threadsafe";

    int a = 0;
    int * b;
    ASSERT_EQ(hipSuccess, hipMalloc((void**) &b, sizeof(*b)));

    int version;
    ASSERT_EQ(hipSuccess, hipRuntimeGetVersion(&version));

    /* Panoptes catches this.
    EXPECT_EQ(hipErrorInvalidValue,
        hipMemcpy(&a,   &a,   sizeof(a), hipMemcpyDeviceToDevice));
    hipGetLastError(); */
    /* Panoptes catches this.
    EXPECT_EQ(hipErrorInvalidValue,
        hipMemcpy(&a,   &a,   sizeof(a), hipMemcpyDeviceToHost));
    hipGetLastError(); */
    /* Panoptes catches this.
    EXPECT_EQ(hipErrorInvalidValue,
        hipMemcpy(&a,   &a,   sizeof(a), hipMemcpyHostToDevice));
    hipGetLastError(); */

    /* Panoptes catches this.
    EXPECT_EQ(hipErrorInvalidValue,
        hipMemcpy(&a,    b,   sizeof(a), hipMemcpyDeviceToDevice));
    hipGetLastError(); */
    /* Panoptes catches this.
    EXPECT_EQ(hipErrorInvalidValue,
        hipMemcpy(&a,    b,   sizeof(a), hipMemcpyHostToDevice));
    hipGetLastError(); */
    EXPECT_EXIT(
        hipMemcpy(&a,    b,   sizeof(a), hipMemcpyHostToHost),
        ::testing::KilledBySignal(SIGSEGV), "");
    hipGetLastError();

    /* Panoptes catches this.
    EXPECT_EQ(hipErrorInvalidValue,
        hipMemcpy( b,   &a,   sizeof(a), hipMemcpyDeviceToDevice));
    hipGetLastError(); */
    /* Error caught by Panoptes
    EXPECT_EQ(hipErrorInvalidValue,
        hipMemcpy( b,   &a,   sizeof(a), hipMemcpyDeviceToHost)); */
    hipGetLastError();
    EXPECT_EXIT(
        hipMemcpy( b,   &a,   sizeof(a), hipMemcpyHostToHost),
        ::testing::KilledBySignal(SIGSEGV), "");
    hipGetLastError();

    /* Caught by Panoptes
    EXPECT_EXIT(
        hipMemcpy( b,    b,   sizeof(a), hipMemcpyDeviceToHost),
        ::testing::KilledBySignal(SIGSEGV), "");
    hipGetLastError(); */
    /* Caught by Panoptes
    EXPECT_EXIT(
        hipMemcpy( b,    b,   sizeof(a), hipMemcpyHostToDevice),
        ::testing::KilledBySignal(SIGSEGV), "");
    hipGetLastError(); */
    if (version >= 4010 /* 4.1 */) {
       EXPECT_EQ(hipSuccess, hipMemcpy(b, b, sizeof(a),
            hipMemcpyHostToHost));
    } else {
        EXPECT_EXIT(
            hipMemcpy( b,    b,   sizeof(a), hipMemcpyHostToHost),
            ::testing::KilledBySignal(SIGSEGV), "");
    }
    hipGetLastError();

    /* Caught by Panoptes.
    EXPECT_EQ(hipErrorInvalidValue,
        hipMemcpy(&a,   NULL, sizeof(a), hipMemcpyDeviceToDevice));
    hipGetLastError(); */
    /* Error, caught by Panoptes
    EXPECT_EQ(hipErrorInvalidValue,
        hipMemcpy(&a,   NULL, sizeof(a), hipMemcpyDeviceToHost));
    hipGetLastError(); */
    /* Caught by Panoptes
    EXPECT_EQ(hipErrorInvalidValue,
        hipMemcpy(&a,   NULL, sizeof(a), hipMemcpyHostToDevice));
    hipGetLastError(); */
    EXPECT_EXIT(
        hipMemcpy(&a,   NULL, sizeof(a), hipMemcpyHostToHost),
        ::testing::KilledBySignal(SIGSEGV), "");
    hipGetLastError();

    /* Caught by Panoptes
    EXPECT_EQ(hipErrorInvalidValue,
        hipMemcpy(NULL, &a,   sizeof(a), hipMemcpyDeviceToDevice));
    hipGetLastError(); */
    /* Error, caught by Panoptes
    EXPECT_EQ(hipErrorInvalidValue,
        hipMemcpy(NULL, &a,   sizeof(a), hipMemcpyDeviceToHost));
    hipGetLastError(); */
    /* Caught by Panoptes
    EXPECT_EQ(hipErrorInvalidValue,
        hipMemcpy(NULL, &a,   sizeof(a), hipMemcpyHostToDevice));
    hipGetLastError(); */
    EXPECT_EXIT(
        hipMemcpy(NULL, &a,   sizeof(a), hipMemcpyHostToHost),
        ::testing::KilledBySignal(SIGSEGV), "");
    hipGetLastError();

    /* Caught by Panoptes
    EXPECT_EQ(hipErrorInvalidValue,
        hipMemcpy( b,   NULL, sizeof(a), hipMemcpyDeviceToDevice));
    hipGetLastError(); */
    /* Error, caught by Panoptes
    EXPECT_EQ(hipErrorInvalidValue,
        hipMemcpy( b,   NULL, sizeof(a), hipMemcpyDeviceToHost)); */
    EXPECT_EXIT(
        hipMemcpy( b,   NULL, sizeof(a), hipMemcpyHostToHost),
        ::testing::KilledBySignal(SIGSEGV), "");
    hipGetLastError();

    /* Caught by Panoptes
    EXPECT_EQ(hipErrorInvalidValue,
        hipMemcpy(NULL,  b,   sizeof(a), hipMemcpyDeviceToDevice));
    hipGetLastError(); */
    /* Caught by Panoptes
    EXPECT_EQ(hipErrorInvalidValue,
        hipMemcpy(NULL,  b,   sizeof(a), hipMemcpyHostToDevice));
    hipGetLastError(); */
    EXPECT_EXIT(
        hipMemcpy(NULL,  b,   sizeof(a), hipMemcpyHostToHost),
        ::testing::KilledBySignal(SIGSEGV), "");
    hipGetLastError();

    /* Caught by Panoptes
    EXPECT_EQ(hipErrorInvalidValue,
        hipMemcpy(NULL, NULL, sizeof(a), hipMemcpyDeviceToDevice));
    hipGetLastError(); */
    /* Error, caught by Panoptes
    EXPECT_EQ(hipErrorInvalidValue,
        hipMemcpy(NULL, NULL, sizeof(a), hipMemcpyDeviceToHost));
    hipGetLastError(); */
    /* Caught by Panoptes
    EXPECT_EQ(hipErrorInvalidValue,
        hipMemcpy(NULL, NULL, sizeof(a), hipMemcpyHostToDevice));
    hipGetLastError(); */
    if (version >= 4010 /* 4.1 */) {
        EXPECT_EQ(hipSuccess, hipMemcpy(NULL, NULL, sizeof(a),
            hipMemcpyHostToHost));
    } else {
        EXPECT_EXIT(
            hipMemcpy(NULL, NULL, sizeof(a), hipMemcpyHostToHost),
            ::testing::KilledBySignal(SIGSEGV), "");
    }
    hipGetLastError();

    ASSERT_EQ(hipSuccess, hipFree(b));
}

int main(int argc, char **argv) {
    ::testing::InitGoogleTest(&argc, argv);
    return RUN_ALL_TESTS();
}
