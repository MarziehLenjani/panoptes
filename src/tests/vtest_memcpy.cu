/**
 * Panoptes - A Binary Translation Framework for CUDA
 * (c) 2011-2012 Chris Kennelly <chris@ckennelly.com>
 *
 * This program is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.
 */

#include <boost/static_assert.hpp>
#include <hip/hip_runtime.h>
#include <gtest/gtest.h>
#include <stdint.h>
#include <valgrind/memcheck.h>
#include <cstdio>

TEST(Memcpy, CheckReturnValues) {
    /**
     * The API documentation states that
     * hipErrorInvalidDevicePointer is a valid return value for hipMemcpy
     *
     * TODO;  This needs a test.
     */

    /**
     * Test woefully out of range directions.
     */
    int a = 0;
    EXPECT_EQ(hipErrorInvalidMemcpyDirection,
        hipMemcpy(&a,   &a,   sizeof(a), (hipMemcpyKind) -1));
    EXPECT_EQ(hipErrorInvalidMemcpyDirection,
        hipMemcpy(NULL, NULL, sizeof(a), (hipMemcpyKind) -1));
}

/**
 * CUDA4 introduced the hipMemcpyDefault direction to hipMemcpy.
 */
TEST(Memcpy, CheckDefaultDirection) {
    int a1 = 0;
    int a2 = 0;
    int * b;
    ASSERT_EQ(hipSuccess, hipMalloc((void**) &b, sizeof(*b)));

    EXPECT_EQ(hipSuccess,
        hipMemcpy(&a1,   &a2,  sizeof(a1), hipMemcpyDefault));
    EXPECT_EQ(hipSuccess,
        hipMemcpy(&a1,    b,   sizeof(a1), hipMemcpyDefault));
    EXPECT_EQ(hipSuccess,
        hipMemcpy( b,    &a1,  sizeof(a1), hipMemcpyDefault));
    EXPECT_EQ(hipSuccess,
        hipMemcpy( b,    b,    sizeof(a1), hipMemcpyDefault));

    ASSERT_EQ(hipSuccess, hipFree(b));
}

/**
 * This test only performs copies in valid directions as to avoid upsetting
 * Valgrind.  The error-causing tests are in test_memcpy.cu.
 */
TEST(Memcpy, AllDirections) {
    int a1 = 0;
    int a2 = 0;
    int * b;
    ASSERT_EQ(hipSuccess, hipMalloc((void**) &b, sizeof(*b) * 2));

    EXPECT_EQ(hipSuccess,
        hipMemcpy(&a1,    &a2,    sizeof(a1), hipMemcpyHostToHost));
    EXPECT_EQ(hipSuccess,
        hipMemcpy(&a1,     b + 0, sizeof(a1), hipMemcpyDeviceToHost));
    EXPECT_EQ(hipSuccess,
        hipMemcpy(&a1,     b + 1, sizeof(a1), hipMemcpyDeviceToHost));
    EXPECT_EQ(hipSuccess,
        hipMemcpy( b + 0, &a1,    sizeof(a1), hipMemcpyHostToDevice));
    EXPECT_EQ(hipSuccess,
        hipMemcpy( b + 1, &a1,    sizeof(a1), hipMemcpyHostToDevice));
    EXPECT_EQ(hipSuccess,
        hipMemcpy( b + 0,  b + 0, sizeof(a1), hipMemcpyDeviceToDevice));
    EXPECT_EQ(hipSuccess,
        hipMemcpy( b + 1,  b + 1, sizeof(a1), hipMemcpyDeviceToDevice));

    ASSERT_EQ(hipSuccess, hipFree(b));
}

int main(int argc, char **argv) {
    ::testing::InitGoogleTest(&argc, argv);
    return RUN_ALL_TESTS();
}
