#include "hip/hip_runtime.h"
/**
 * Panoptes - A Binary Translation Framework for CUDA
 * (c) 2011-2012 Chris Kennelly <chris@ckennelly.com>
 *
 * This program is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.
 */

#include <hip/hip_runtime.h>
#include <gtest/gtest.h>

extern "C" __global__ void k_popc(const unsigned * data, const int N,
        int * popc_values) {
    for (int idx = threadIdx.x + blockDim.x * blockIdx.x;
            idx < N; idx += blockDim.x * gridDim.x) {
        popc_values[idx] = __popc(data[idx]);
    }
}

extern "C" __global__ void k_popcll(const unsigned long long * data,
        const int N, int * popc_values) {
    for (int idx = threadIdx.x + blockDim.x * blockIdx.x;
            idx < N; idx += blockDim.x * gridDim.x) {
        popc_values[idx] = __popc(data[idx]);
    }
}

TEST(kPOPC, POPC) {
    hipError_t ret;
    hipStream_t stream;

    const int N = 1 << 20;
    const int n_blocks = 32;

    unsigned * data;
    int * popc_values;

    ret = hipMalloc((void **) &data, sizeof(*data) * N);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipMalloc((void **) &popc_values, sizeof(*popc_values) * N);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipStreamCreate(&stream);
    ASSERT_EQ(hipSuccess, ret);

    k_popc<<<256, n_blocks, 0, stream>>>(data, N, popc_values);

    ret = hipStreamSynchronize(stream);
    EXPECT_EQ(hipSuccess, ret);

    ret = hipStreamDestroy(stream);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipFree(data);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipFree(popc_values);
    ASSERT_EQ(hipSuccess, ret);
}

TEST(kPOPC, POPCLL) {
    hipError_t ret;
    hipStream_t stream;

    const int N = 1 << 20;
    const int n_blocks = 32;

    unsigned long long * data;
    int * popc_values;

    ret = hipMalloc((void **) &data, sizeof(*data) * N);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipMalloc((void **) &popc_values, sizeof(*popc_values) * N);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipStreamCreate(&stream);
    ASSERT_EQ(hipSuccess, ret);

    k_popcll<<<256, n_blocks, 0, stream>>>(data, N, popc_values);

    ret = hipStreamSynchronize(stream);
    EXPECT_EQ(hipSuccess, ret);

    ret = hipStreamDestroy(stream);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipFree(data);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipFree(popc_values);
    ASSERT_EQ(hipSuccess, ret);
}

int main(int argc, char **argv) {
    ::testing::InitGoogleTest(&argc, argv);
    return RUN_ALL_TESTS();
}
