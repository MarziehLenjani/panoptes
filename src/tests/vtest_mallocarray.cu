/**
 * Panoptes - A Binary Translation Framework for CUDA
 * (c) 2011-2012 Chris Kennelly <chris@ckennelly.com>
 *
 * This program is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.
 */

#include <hip/hip_runtime.h>
#include <gtest/gtest.h>

TEST(MallocArray, NullArguments) {
    struct hipArray * ary;
    struct hipChannelFormatDesc dsc;
    dsc.x = dsc.y = dsc.z = dsc.w = 8;
    dsc.f = hipChannelFormatKindSigned;

    // Commented out cases segfault.

    EXPECT_EQ(hipErrorInvalidValue, hipMallocArray(NULL, NULL, 0, 0, 0));
    EXPECT_EQ(hipErrorInvalidValue, hipMallocArray(NULL, NULL, 0, 8, 0));
    // EXPECT_EQ(hipErrorInvalidValue, hipMallocArray(NULL, NULL, 8, 0, 0), 0));
    // EXPECT_EQ(hipErrorInvalidValue, hipMallocArray(NULL, NULL, 8, 8, 0));

    EXPECT_EQ(hipSuccess,           hipMallocArray(&ary, NULL, 0, 0, 0));
    EXPECT_EQ(hipSuccess,           hipFreeArray(ary));

    EXPECT_EQ(hipSuccess,           hipMallocArray(&ary, NULL, 0, 8, 0));
    EXPECT_EQ(hipSuccess,           hipFreeArray(ary));

    // EXPECT_EQ(hipErrorInvalidValue, hipMallocArray(&ary, NULL, 8, 0, 0));

    // EXPECT_EQ(hipErrorInvalidValue, hipMallocArray(&ary, NULL, 8, 8, 0));

    EXPECT_EQ(hipErrorInvalidValue, hipMallocArray(NULL, &dsc, 0, 0, 0));
    EXPECT_EQ(hipErrorInvalidValue, hipMallocArray(NULL, &dsc, 0, 8, 0));
    // EXPECT_EQ(hipErrorInvalidValue, hipMallocArray(NULL, &dsc, 8, 0, 0));
    // EXPECT_EQ(hipErrorInvalidValue, hipMallocArray(NULL, &dsc, 8, 8, 0));
}

TEST(MallocArray, Limits) {
    struct hipArray * ary;
    struct hipChannelFormatDesc dsc;
    dsc.x = dsc.y = dsc.z = dsc.w = 8;
    dsc.f = hipChannelFormatKindSigned;

    hipError_t ret;

    ret = hipMallocArray(&ary, &dsc, 0, 0, 0);
    EXPECT_EQ(hipSuccess, ret);
    if (ret == hipSuccess) {
        EXPECT_EQ(hipSuccess, hipFreeArray(ary));
    }

    int device;
    ret = hipGetDevice(&device);
    ASSERT_EQ(hipSuccess, ret);

    struct hipDeviceProp_t prop;
    ret = hipGetDeviceProperties(&prop, device);
    ASSERT_EQ(hipSuccess, ret);

    /* Adapt to what's available by a safe margin */
    size_t targetable = prop.totalGlobalMem / 8;

    if ((size_t) prop.maxTexture1D < targetable) {
        ret = hipMallocArray(&ary, &dsc, prop.maxTexture1D, 0, 0);
        EXPECT_EQ(hipSuccess, ret);
        if (ret == hipSuccess) {
            EXPECT_EQ(hipSuccess, hipFreeArray(ary));
        }

        ret = hipMallocArray(&ary, &dsc, prop.maxTexture1D + 1, 0, 0);
        EXPECT_EQ(hipErrorInvalidValue, ret);
        if (ret == hipSuccess) {
            EXPECT_EQ(hipSuccess, hipFreeArray(ary));
        }
    }

    if ((size_t) prop.maxTexture2D[0] < targetable) {
        ret = hipMallocArray(&ary, &dsc, prop.maxTexture2D[0], 1, 0);
        EXPECT_EQ(hipSuccess, ret);
        if (ret == hipSuccess) {
            EXPECT_EQ(hipSuccess, hipFreeArray(ary));
        }

        ret = hipMallocArray(&ary, &dsc, prop.maxTexture2D[0] + 1, 1, 0);
        EXPECT_EQ(hipErrorInvalidValue, ret);
        if (ret == hipSuccess) {
            EXPECT_EQ(hipSuccess, hipFreeArray(ary));
        }
    }

    if ((size_t) prop.maxTexture2D[1] < targetable) {
        ret = hipMallocArray(&ary, &dsc, 1, prop.maxTexture2D[1], 0);
        EXPECT_EQ(hipSuccess, ret);
        if (ret == hipSuccess) {
            EXPECT_EQ(hipSuccess, hipFreeArray(ary));
        }

        ret = hipMallocArray(&ary, &dsc, 1, prop.maxTexture2D[1] + 1, 0);
        EXPECT_EQ(hipErrorInvalidValue, ret);
        if (ret == hipSuccess) {
            EXPECT_EQ(hipSuccess, hipFreeArray(ary));
        }
    }

    if ((size_t) prop.maxTexture2D[0] * prop.maxTexture2D[1] < targetable) {
        ret = hipMallocArray(&ary, &dsc, prop.maxTexture2D[0],
                prop.maxTexture2D[1], 0);
        EXPECT_EQ(hipSuccess, ret);
        if (ret == hipSuccess) {
            EXPECT_EQ(hipSuccess, hipFreeArray(ary));
        }

        ret = hipMallocArray(&ary, &dsc, prop.maxTexture2D[0],
                prop.maxTexture2D[1] + 1, 0);
        EXPECT_EQ(hipErrorInvalidValue, ret);
        if (ret == hipSuccess) {
            EXPECT_EQ(hipSuccess, hipFreeArray(ary));
        }

        ret = hipMallocArray(&ary, &dsc, prop.maxTexture2D[0] + 1,
                prop.maxTexture2D[1], 0);
        EXPECT_EQ(hipErrorInvalidValue, ret);
        if (ret == hipSuccess) {
            EXPECT_EQ(hipSuccess, hipFreeArray(ary));
        }

        ret = hipMallocArray(&ary, &dsc, prop.maxTexture2D[0] + 1,
                prop.maxTexture2D[1] + 1, 0);
        EXPECT_EQ(hipErrorInvalidValue, ret);
        if (ret == hipSuccess) {
            EXPECT_EQ(hipSuccess, hipFreeArray(ary));
        }
    } else if ((size_t) prop.maxTexture2D[0] * prop.maxTexture2D[1] >
            prop.totalGlobalMem) {
        EXPECT_EQ(hipErrorOutOfMemory,
            hipMallocArray(&ary, &dsc, prop.maxTexture2D[0],
            prop.maxTexture2D[1], 0));
    }
}

TEST(MallocArray, Attributes) {
    struct hipArray * ary;
    struct hipChannelFormatDesc dsc;
    dsc.x = dsc.y = dsc.z = dsc.w = 8;
    dsc.f = hipChannelFormatKindSigned;

    hipError_t ret;

    ret = hipMallocArray(&ary, &dsc, 1, 1, 0);
    ASSERT_EQ(hipSuccess, ret);

    struct hipPointerAttribute_t attr;
    ret = hipPointerGetAttributes(&attr, ary);
    EXPECT_EQ(hipErrorInvalidValue, ret);

    EXPECT_EQ(hipSuccess, hipFreeArray(ary));
}

TEST(MallocArray, NegativeChannels) {
    struct hipArray * ary;
    struct hipChannelFormatDesc dsc;
    dsc.x = dsc.y = dsc.z = 8;
    dsc.w = -8;
    dsc.f = hipChannelFormatKindSigned;

    hipError_t ret;

    ret = hipMallocArray(&ary, &dsc, 1, 1, 0);
    ASSERT_EQ(hipErrorInvalidChannelDescriptor, ret);
}

TEST(MallocArray, Mismatch) {
    struct hipArray * ary;
    struct hipChannelFormatDesc dsc;
    dsc.x = dsc.y = dsc.z = dsc.w = 8;
    dsc.f = hipChannelFormatKindSigned;

    hipError_t ret;
    ret = hipMallocArray(&ary, &dsc, 1, 1, 0);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipFree(ary);
    EXPECT_EQ(hipErrorInvalidDevicePointer, ret);

    ret = hipFreeArray(ary);
    ASSERT_EQ(hipSuccess, ret);
}

int main(int argc, char **argv) {
    ::testing::InitGoogleTest(&argc, argv);
    return RUN_ALL_TESTS();
}
