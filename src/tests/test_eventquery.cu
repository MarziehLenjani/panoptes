/**
 * Panoptes - A Binary Translation Framework for CUDA
 * (c) 2011-2012 Chris Kennelly <chris@ckennelly.com>
 *
 * This program is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.
 */

#include <hip/hip_runtime.h>
#include <gtest/gtest.h>

TEST(EventQuery, QueryAfterDestroy) {
    ::testing::FLAGS_gtest_death_test_style = "threadsafe";

    hipError_t ret;
    hipEvent_t event;

    ret = hipEventCreate(&event);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipEventDestroy(event);
    EXPECT_EQ(hipSuccess, ret);

    EXPECT_EXIT(
        hipEventQuery(event),
        ::testing::KilledBySignal(SIGSEGV), "");
}

int main(int argc, char **argv) {
    ::testing::InitGoogleTest(&argc, argv);
    return RUN_ALL_TESTS();
}
