/**
 * Panoptes - A Binary Translation Framework for CUDA
 * (c) 2011-2012 Chris Kennelly <chris@ckennelly.com>
 *
 * This program is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.
 */

#include <boost/static_assert.hpp>
#include <hip/hip_runtime.h>
#include <gtest/gtest.h>
#include <stdint.h>
#include <valgrind/memcheck.h>

typedef uint32_t symbol_t;
__device__   symbol_t device_symbol;
__device__   symbol_t device_symbol2;

TEST(MemcpyFromSymbol, Invalid) {
    const char missing[] = "this_symbol_does_not_exist";
    hipError_t ret;
    symbol_t target;

    ret = hipMemcpyFromSymbol(NULL, HIP_SYMBOL(missing), sizeof(symbol_t), 0,
        hipMemcpyDeviceToHost);
    EXPECT_EQ(hipErrorInvalidSymbol, ret);

    ret = hipMemcpyFromSymbol(&target, HIP_SYMBOL(missing), sizeof(symbol_t), 0,
        hipMemcpyDeviceToHost);
    EXPECT_EQ(hipErrorInvalidSymbol, ret);

    ret = hipMemcpyFromSymbol(NULL, HIP_SYMBOL(NULL), sizeof(symbol_t), 0,
        hipMemcpyDeviceToHost);
    EXPECT_EQ(hipErrorInvalidSymbol, ret);

    ret = hipMemcpyFromSymbol(&target, HIP_SYMBOL(NULL), sizeof(symbol_t), 0,
        hipMemcpyDeviceToHost);
    EXPECT_EQ(hipErrorInvalidSymbol, ret);
}

TEST(MemcpyFromSymbol, Basic) {
    hipError_t ret;
    void * ptr;

    ret = hipGetSymbolAddress(&ptr, HIP_SYMBOL(device_symbol));
    ASSERT_EQ(hipSuccess, device_symbol);

    ret = hipMemset(ptr, 0xAA, sizeof(symbol_t));
    ASSERT_EQ(hipSuccess, device_symbol);

    symbol_t atarget, target, vtarget;
    BOOST_STATIC_ASSERT(sizeof(atarget) == sizeof(device_symbol));
    ret = hipMemcpyFromSymbol(&atarget, HIP_SYMBOL("device_symbol"),
        sizeof(symbol_t), 0, hipMemcpyDeviceToHost);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipMemcpyFromSymbol( &target, HIP_SYMBOL(device_symbol),
        sizeof(symbol_t), 0, hipMemcpyDeviceToHost);
    ASSERT_EQ(hipSuccess, ret);

    int valgrind = VALGRIND_GET_VBITS(&atarget, &vtarget, sizeof(symbol_t));
    assert(valgrind == 0 || valgrind == 1);

    if (valgrind == 1) {
        const symbol_t valid = 0;

        EXPECT_EQ(valid, vtarget);
    } else {
        return;
    }

    valgrind = VALGRIND_GET_VBITS(&target, &vtarget, sizeof(symbol_t));
    assert(valgrind == 0 || valgrind == 1);

    if (valgrind == 1) {
        const symbol_t valid = 0;

        EXPECT_EQ(valid, vtarget);
    }
}

TEST(MemcpyFromSymbol, ByAddress) {
    hipError_t ret;
    void * ptr;

    ret = hipGetSymbolAddress(&ptr, HIP_SYMBOL(device_symbol));
    ASSERT_EQ(hipSuccess, device_symbol);

    ret = hipMemset(ptr, 0xAA, sizeof(symbol_t));
    ASSERT_EQ(hipSuccess, device_symbol);

    symbol_t target;
    ret = hipMemcpyFromSymbol(&target, HIP_SYMBOL(ptr), sizeof(symbol_t), 0,
        hipMemcpyDeviceToHost);
    EXPECT_EQ(hipErrorInvalidSymbol, ret);
}

TEST(MemcpyFromSymbol, DeviceToDevice) {
    hipError_t ret;
    void * device_ptr;
    void * symbol_ptr;

    ret = hipMalloc(&device_ptr, sizeof(symbol_t));
    ASSERT_EQ(hipSuccess, ret);

    ret = hipGetSymbolAddress(&symbol_ptr, HIP_SYMBOL(device_symbol));
    ASSERT_EQ(hipSuccess, device_symbol);

    const int pattern = 0xAA;
    ret = hipMemset(symbol_ptr, pattern, sizeof(symbol_t));
    ASSERT_EQ(hipSuccess, device_symbol);

    ret = hipMemcpyFromSymbol(device_ptr, HIP_SYMBOL(device_symbol),
        sizeof(symbol_t), 0, hipMemcpyDeviceToDevice);
    EXPECT_EQ(hipSuccess, ret);

    if (ret == hipSuccess) {
        symbol_t expected, target, vtarget;
        BOOST_STATIC_ASSERT(sizeof(symbol_t) == sizeof(device_symbol));
        ret = hipMemcpy(&target, device_ptr, sizeof(symbol_t),
            hipMemcpyDeviceToHost);
        ASSERT_EQ(hipSuccess, ret);

        memset(&expected, pattern, sizeof(expected));

        int valgrind = VALGRIND_GET_VBITS(&target, &vtarget, sizeof(symbol_t));
        assert(valgrind == 0 || valgrind == 1);

        /*
         * Suppress validity warnings on target.
         */
        if (valgrind == 1) {
            VALGRIND_MAKE_MEM_DEFINED(&target, sizeof(target));
        }

        EXPECT_EQ(expected, target);

        if (valgrind == 1) {
            const uint32_t valid = 0;
            BOOST_STATIC_ASSERT(sizeof(valid) == sizeof(target));

            EXPECT_EQ(valid, vtarget);
        }
    }

    ret = hipFree(device_ptr);
    ASSERT_EQ(hipSuccess, ret);
}

TEST(MemcpyFromSymbol, InvalidDirections) {
    hipError_t ret;
    symbol_t target;

    ret = hipMemcpyFromSymbol(&target, HIP_SYMBOL(device_symbol),
        sizeof(symbol_t), 0, hipMemcpyHostToDevice);
    EXPECT_EQ(hipErrorInvalidMemcpyDirection, ret);

    ret = hipMemcpyFromSymbol(&target, HIP_SYMBOL(device_symbol),
        sizeof(symbol_t), 0, hipMemcpyHostToHost);
    EXPECT_EQ(hipErrorInvalidMemcpyDirection, ret);

    /* hipMemcpyDefault implemented in test_memcpyfromsymbol.cu */
}

TEST(MemcpyFromSymbol, NonSymbol) {
    hipError_t ret;
    symbol_t target;
    void * device_ptr;

    ret = hipMalloc(&device_ptr, sizeof(symbol_t));
    ASSERT_EQ(hipSuccess, ret);

    ret = hipMemcpyFromSymbol(&target, HIP_SYMBOL(device_ptr),
        sizeof(symbol_t), 0, hipMemcpyDeviceToHost);
    EXPECT_EQ(hipErrorInvalidSymbol, ret);

    ret = hipFree(device_ptr);
    ASSERT_EQ(hipSuccess, ret);
}

TEST(MemcpyFromSymbol, OutOfBounds) {
    hipError_t ret;
    symbol_t target[2];

    ret = hipMemcpyFromSymbol(target, HIP_SYMBOL(device_symbol),
        sizeof(symbol_t), sizeof(symbol_t), hipMemcpyDeviceToHost);
    ASSERT_EQ(hipErrorInvalidValue, ret);
}

TEST(MemcpyFromSymbol, SymbolToSymbol) {
    hipError_t ret;
    void * device_symbol2_ptr;

    ret = hipGetSymbolAddress(&device_symbol2_ptr, HIP_SYMBOL(device_symbol2));
    ASSERT_EQ(hipSuccess, ret);

    BOOST_STATIC_ASSERT(sizeof(device_symbol) == sizeof(device_symbol2));
    ret = hipMemcpyFromSymbol(device_symbol2_ptr, HIP_SYMBOL(device_symbol),
        sizeof(symbol_t), 0, hipMemcpyDeviceToDevice);
    EXPECT_EQ(hipSuccess, ret);
}

__device__   symbol_t initialized_symbol = 0xDEADBEEF;

TEST(MemcpyFromSymbol, InitializedSymbol) {
    hipError_t ret;
    void * ptr;

    symbol_t target;
    BOOST_STATIC_ASSERT(sizeof(target) == sizeof(initialized_symbol));
    ret = hipMemcpyFromSymbol(&target, HIP_SYMBOL(initialized_symbol),
        sizeof(target), 0, hipMemcpyDeviceToHost);
    ASSERT_EQ(hipSuccess, ret);

    symbol_t vtarget;
    unsigned valgrind = VALGRIND_GET_VBITS(&target, &vtarget, sizeof(target));
    assert(valgrind == 0 || valgrind == 1);
    if (valgrind == 1) {
        const symbol_t vexpected = 0;
        EXPECT_EQ(vexpected, vtarget);
    }

    VALGRIND_MAKE_MEM_DEFINED_IF_ADDRESSABLE(&target, sizeof(target));
    EXPECT_EQ(0xDEADBEEF, target);
}

int main(int argc, char **argv) {
    ::testing::InitGoogleTest(&argc, argv);
    return RUN_ALL_TESTS();
}
