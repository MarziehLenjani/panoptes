/**
 * Panoptes - A Binary Translation Framework for CUDA
 * (c) 2011-2012 Chris Kennelly <chris@ckennelly.com>
 *
 * This program is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.
 */

#include <boost/scoped_array.hpp>
#include <boost/static_assert.hpp>
#include <hip/hip_runtime.h>
#include <gtest/gtest.h>
#include <stdint.h>
#include <valgrind/memcheck.h>
#include <cstdio>

typedef int symbol_t;
__device__   symbol_t device_symbol;

TEST(MemcpyToSymbolDeathTest, InvalidDirections) {
    ::testing::FLAGS_gtest_death_test_style = "threadsafe";

    symbol_t target;

    /**
     * Panoptes catches this.
     */
    EXPECT_EXIT(
        hipMemcpyToSymbol(HIP_SYMBOL(device_symbol), &target,
        sizeof(device_symbol), 0, hipMemcpyDefault),
        ::testing::KilledBySignal(SIGSEGV), "");
     /* */
}

int main(int argc, char **argv) {
    ::testing::InitGoogleTest(&argc, argv);
    return RUN_ALL_TESTS();
}
