/**
 * Panoptes - A Binary Translation Framework for CUDA
 * (c) 2011-2012 Chris Kennelly <chris@ckennelly.com>
 *
 * This program is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.
 */

#include <hip/hip_runtime.h>
#include <gtest/gtest.h>
#include <stdint.h>

texture<int32_t, 1, hipReadModeElementType> tex_src;

TEST(BindTexture, Simple) {
    hipError_t ret;
    const struct textureReference * texref;

    const uint32_t bytes = 1u << 20;
    int32_t * data;
    ret = hipMalloc((void **) &data, sizeof(*data) * bytes);
    ASSERT_EQ(hipSuccess, ret);

    int version;
    ret = hipRuntimeGetVersion(&version);
    ASSERT_EQ(hipSuccess, ret);

    #if CUDA_VERSION >= 5000
    if (version < 5000 /* 5.0 */) {
    #endif
        ret = hipGetTextureReference(&texref, HIP_SYMBOL("tex_src"));
    #if CUDA_VERSION >= 5000
    } else {
        ret = hipGetTextureReference(&texref, HIP_SYMBOL(&tex_src));
    }
    #endif
    ASSERT_EQ(hipSuccess, ret);

    struct hipChannelFormatDesc desc;
    desc.f = hipChannelFormatKindSigned;
    desc.x = CHAR_BIT * sizeof(*data);
    desc.y = desc.z = desc.w = 0;
    ret = hipBindTexture(NULL, texref, data, &desc, bytes);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipUnbindTexture(tex_src);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipFree(data);
    ASSERT_EQ(hipSuccess, ret);
}

TEST(BindTexture, Adjacent) {
    hipError_t ret;
    const struct textureReference * texref;

    uint32_t bytes = 1u << 20;
    uint8_t * data[2];

    while (bytes > 0) {
        ret = hipMalloc((void **) &data[0], bytes);
        ASSERT_EQ(hipSuccess, ret);

        ret = hipMalloc((void **) &data[1], bytes);
        ASSERT_EQ(hipSuccess, ret);

        if (data[1] < data[0]) {
            std::swap(data[0], data[1]);
        }

        if ((size_t) (data[1] - data[0]) == bytes) {
            break;
        } else {
            ret = hipFree(data[0]);
            ASSERT_EQ(hipSuccess, ret);

            ret = hipFree(data[1]);
            ASSERT_EQ(hipSuccess, ret);

            bytes = bytes >> 1u;
        }
    }

    if (bytes == 0) {
        return;
    }

    ASSERT_LT(0, bytes);

    int version;
    ret = hipRuntimeGetVersion(&version);
    ASSERT_EQ(hipSuccess, ret);

    #if CUDA_VERSION >= 5000
    if (version < 5000 /* 5.0 */) {
    #endif
        ret = hipGetTextureReference(&texref, HIP_SYMBOL("tex_src"));
    #if CUDA_VERSION >= 5000
    } else {
        ret = hipGetTextureReference(&texref, HIP_SYMBOL(&tex_src));
    }
    #endif
    ASSERT_EQ(hipSuccess, ret);

    struct hipChannelFormatDesc desc;
    desc.f = hipChannelFormatKindSigned;
    desc.x = CHAR_BIT;
    desc.y = desc.z = desc.w = 0;
    ret = hipBindTexture(NULL, texref, data[0], &desc, 2u * bytes);
    EXPECT_EQ(hipSuccess, ret);

    ret = hipUnbindTexture(tex_src);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipFree(data[0]);
    EXPECT_EQ(hipSuccess, ret);

    ret = hipFree(data[1]);
    EXPECT_EQ(hipSuccess, ret);
}

TEST(BindTexture, DoubleBind) {
    hipError_t ret;
    const struct textureReference * texref;

    const uint32_t bytes = 1u << 20;
    int32_t * data;
    ret = hipMalloc((void **) &data, sizeof(*data) * bytes);
    ASSERT_EQ(hipSuccess, ret);

    int version;
    ret = hipRuntimeGetVersion(&version);
    ASSERT_EQ(hipSuccess, ret);

    #if CUDA_VERSION >= 5000
    if (version < 5000 /* 5.0 */) {
    #endif
        ret = hipGetTextureReference(&texref, HIP_SYMBOL("tex_src"));
    #if CUDA_VERSION >= 5000
    } else {
        ret = hipGetTextureReference(&texref, HIP_SYMBOL(&tex_src));
    }
    #endif
    ASSERT_EQ(hipSuccess, ret);

    struct hipChannelFormatDesc desc;
    desc.f = hipChannelFormatKindSigned;
    desc.x = CHAR_BIT * sizeof(*data);
    desc.y = desc.z = desc.w = 0;

    ret = hipBindTexture(NULL, texref, data, &desc, bytes);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipBindTexture(NULL, texref, data, &desc, bytes);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipUnbindTexture(tex_src);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipFree(data);
    ASSERT_EQ(hipSuccess, ret);
}

TEST(BindTexture, FreeBeforeUnbind) {
    hipError_t ret;
    const struct textureReference * texref;

    const uint32_t bytes = 1u << 20;
    int32_t * data;
    ret = hipMalloc((void **) &data, sizeof(*data) * bytes);
    ASSERT_EQ(hipSuccess, ret);

    int version;
    ret = hipRuntimeGetVersion(&version);
    ASSERT_EQ(hipSuccess, ret);

    #if CUDA_VERSION >= 5000
    if (version < 5000 /* 5.0 */) {
    #endif
        ret = hipGetTextureReference(&texref, HIP_SYMBOL("tex_src"));
    #if CUDA_VERSION >= 5000
    } else {
        ret = hipGetTextureReference(&texref, HIP_SYMBOL(&tex_src));
    }
    #endif
    ASSERT_EQ(hipSuccess, ret);

    struct hipChannelFormatDesc desc;
    desc.f = hipChannelFormatKindSigned;
    desc.x = CHAR_BIT * sizeof(*data);
    desc.y = desc.z = desc.w = 0;
    ret = hipBindTexture(NULL, texref, data, &desc, bytes);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipFree(data);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipUnbindTexture(tex_src);
    ASSERT_EQ(hipSuccess, ret);
}

TEST(BindTexture, Overrun) {
    hipError_t ret;
    const struct textureReference * texref;

    const uint32_t bytes = 1u << 20;
    int32_t * data;
    ret = hipMalloc((void **) &data, sizeof(*data) * bytes);
    ASSERT_EQ(hipSuccess, ret);

    int version;
    ret = hipRuntimeGetVersion(&version);
    ASSERT_EQ(hipSuccess, ret);

    #if CUDA_VERSION >= 5000
    if (version < 5000 /* 5.0 */) {
    #endif
        ret = hipGetTextureReference(&texref, HIP_SYMBOL("tex_src"));
    #if CUDA_VERSION >= 5000
    } else {
        ret = hipGetTextureReference(&texref, HIP_SYMBOL(&tex_src));
    }
    #endif
    ASSERT_EQ(hipSuccess, ret);

    struct hipChannelFormatDesc desc;
    desc.f = hipChannelFormatKindSigned;
    desc.x = CHAR_BIT * sizeof(*data);
    desc.y = desc.z = desc.w = 0;
    ret = hipBindTexture(NULL, texref, data, &desc, 2 * bytes * sizeof(*data));
    ASSERT_EQ(hipSuccess, ret);

    ret = hipUnbindTexture(tex_src);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipFree(data);
    ASSERT_EQ(hipSuccess, ret);
}

TEST(BindTexture, NullArguments) {
    hipError_t ret;
    const struct textureReference * texref;

    const uint32_t bytes = 1u << 20;
    int32_t * data;
    ret = hipMalloc((void **) &data, sizeof(*data) * bytes);
    ASSERT_EQ(hipSuccess, ret);

    int version;
    ret = hipRuntimeGetVersion(&version);
    ASSERT_EQ(hipSuccess, ret);

    #if CUDA_VERSION >= 5000
    if (version < 5000 /* 5.0 */) {
    #endif
        ret = hipGetTextureReference(&texref, HIP_SYMBOL("tex_src"));
    #if CUDA_VERSION >= 5000
    } else {
        ret = hipGetTextureReference(&texref, HIP_SYMBOL(&tex_src));
    }
    #endif
    ASSERT_EQ(hipSuccess, ret);

    struct hipChannelFormatDesc desc;
    desc.f = hipChannelFormatKindSigned;
    desc.x = CHAR_BIT * sizeof(*data);
    desc.y = desc.z = desc.w = 0;

    ret = hipBindTexture(NULL, texref, NULL, &desc, bytes);
    EXPECT_EQ(hipErrorUnknown, ret);

    ret = hipBindTexture(NULL, texref, NULL, NULL,  bytes);
    EXPECT_EQ(hipErrorUnknown, ret);

    ret = hipBindTexture(NULL, NULL,   data, &desc, bytes);
    EXPECT_EQ(hipErrorInvalidTexture, ret);

    ret = hipBindTexture(NULL, NULL,   data, NULL,  bytes);
    EXPECT_EQ(hipErrorInvalidTexture, ret);

    ret = hipBindTexture(NULL, NULL,   NULL, &desc, bytes);
    EXPECT_EQ(hipErrorInvalidTexture, ret);

    ret = hipBindTexture(NULL, NULL,   NULL, NULL,  bytes);
    EXPECT_EQ(hipErrorInvalidTexture, ret);

    ret = hipFree(data);
    ASSERT_EQ(hipSuccess, ret);
}

TEST(BindTexture, Offsets) {
    hipError_t ret;
    const struct textureReference * texref;

    const uint32_t bytes = 1u << 20;
    int32_t * data;
    ret = hipMalloc((void **) &data, sizeof(*data) * bytes);
    ASSERT_EQ(hipSuccess, ret);

    int32_t * offset_data = data + 1;

    int version;
    ret = hipRuntimeGetVersion(&version);
    ASSERT_EQ(hipSuccess, ret);

    #if CUDA_VERSION >= 5000
    if (version < 5000 /* 5.0 */) {
    #endif
        ret = hipGetTextureReference(&texref, HIP_SYMBOL("tex_src"));
    #if CUDA_VERSION >= 5000
    } else {
        ret = hipGetTextureReference(&texref, HIP_SYMBOL(&tex_src));
    }
    #endif
    ASSERT_EQ(hipSuccess, ret);

    struct hipChannelFormatDesc desc;
    desc.f = hipChannelFormatKindSigned;
    desc.x = CHAR_BIT * sizeof(*data);
    desc.y = desc.z = desc.w = 0;

    ret = hipBindTexture(NULL, texref, offset_data, &desc, bytes);
    ASSERT_EQ(hipErrorInvalidValue, ret);

    size_t offset;
    ret = hipBindTexture(&offset, texref, offset_data, &desc, bytes);
    ASSERT_EQ(hipSuccess, ret);
    EXPECT_EQ((offset_data - data) * sizeof(*data), offset);

    ret = hipUnbindTexture(tex_src);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipFree(data);
    ASSERT_EQ(hipSuccess, ret);
}

int main(int argc, char **argv) {
    ::testing::InitGoogleTest(&argc, argv);
    return RUN_ALL_TESTS();
}
