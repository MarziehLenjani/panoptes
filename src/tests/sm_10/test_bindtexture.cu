/**
 * Panoptes - A Binary Translation Framework for CUDA
 * (c) 2011-2012 Chris Kennelly <chris@ckennelly.com>
 *
 * This program is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.
 */

#include <hip/hip_runtime.h>
#include <gtest/gtest.h>
#include <stdint.h>

texture<int32_t, 1, hipReadModeElementType> tex_src;

TEST(BindTexture, NullArguments) {
    ::testing::FLAGS_gtest_death_test_style = "threadsafe";

    hipError_t ret;
    const struct textureReference * texref;

    const uint32_t bytes = 1u << 20;
    int32_t * data;
    ret = hipMalloc((void **) &data, sizeof(*data) * bytes);
    ASSERT_EQ(hipSuccess, ret);

    int version;
    ret = hipRuntimeGetVersion(&version);
    ASSERT_EQ(hipSuccess, ret);

    if (version < 5000 /* 5.0 */) {
        ret = hipGetTextureReference(&texref, HIP_SYMBOL("tex_src"));
    } else {
        ret = hipGetTextureReference(&texref, HIP_SYMBOL(&tex_src));
    }
    ASSERT_EQ(hipSuccess, ret);

    EXPECT_EXIT(
        hipBindTexture(NULL, texref, data, NULL,  bytes),
        ::testing::KilledBySignal(SIGSEGV), "");

    ret = hipFree(data);
    ASSERT_EQ(hipSuccess, ret);
}

int main(int argc, char **argv) {
    ::testing::InitGoogleTest(&argc, argv);
    return RUN_ALL_TESTS();
}
