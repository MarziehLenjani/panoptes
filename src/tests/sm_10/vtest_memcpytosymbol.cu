/**
 * Panoptes - A Binary Translation Framework for CUDA
 * (c) 2011-2012 Chris Kennelly <chris@ckennelly.com>
 *
 * This program is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.
 */

#include <boost/static_assert.hpp>
#include <hip/hip_runtime.h>
#include <gtest/gtest.h>
#include <stdint.h>
#include <valgrind/memcheck.h>

typedef uint32_t symbol_t;
__device__   symbol_t device_symbol;
__device__   symbol_t device_symbol2;

TEST(MemcpyToSymbol, Invalid) {
    const char missing[] = "this_symbol_does_not_exist";
    hipError_t ret;
    symbol_t target;

    ret = hipMemcpyToSymbol(HIP_SYMBOL(missing), NULL, sizeof(symbol_t), 0,
        hipMemcpyHostToDevice);
    EXPECT_EQ(hipErrorInvalidSymbol, ret);

    ret = hipMemcpyToSymbol(HIP_SYMBOL(missing), &target, sizeof(symbol_t), 0,
        hipMemcpyHostToDevice);
    EXPECT_EQ(hipErrorInvalidSymbol, ret);

    ret = hipMemcpyToSymbol(HIP_SYMBOL(NULL), NULL, sizeof(symbol_t), 0,
        hipMemcpyHostToDevice);
    EXPECT_EQ(hipErrorInvalidSymbol, ret);

    ret = hipMemcpyToSymbol(HIP_SYMBOL(NULL), &target, sizeof(symbol_t), 0,
        hipMemcpyHostToDevice);
    EXPECT_EQ(hipErrorInvalidSymbol, ret);
}

__device__ symbol_t device_symbol_basic;

TEST(MemcpyToSymbol, Basic) {
    hipError_t ret;
    void * ptr;

    ret = hipGetSymbolAddress(&ptr, HIP_SYMBOL(device_symbol_basic));
    ASSERT_EQ(hipSuccess, ret);

    symbol_t target, dtarget, ftarget, vtarget;
    memset(&target, 0xAA, sizeof(symbol_t));

    ret = hipMemcpyToSymbol(HIP_SYMBOL(device_symbol_basic), &target,
        sizeof(symbol_t), 0, hipMemcpyHostToDevice);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipMemcpy(&dtarget, ptr, sizeof(symbol_t),
        hipMemcpyDeviceToHost);
    ASSERT_EQ(hipSuccess, ret);

    int valgrind = VALGRIND_GET_VBITS(&dtarget, &vtarget, sizeof(symbol_t));
    assert(valgrind == 0 || valgrind == 1);

    EXPECT_EQ(0xAAAAAAAA, dtarget);

    ret = hipMemcpyFromSymbol(&ftarget, HIP_SYMBOL(device_symbol_basic),
        sizeof(symbol_t), 0, hipMemcpyDeviceToHost);
    ASSERT_EQ(hipSuccess, ret);

    if (valgrind == 1) {
        const symbol_t valid = 0;

        EXPECT_EQ(valid, vtarget);
    } else {
        return;
    }

    valgrind = VALGRIND_GET_VBITS(&ftarget, &vtarget, sizeof(symbol_t));
    assert(valgrind == 0 || valgrind == 1);

    EXPECT_EQ(0xAAAAAAAA, ftarget);

    if (valgrind == 1) {
        const symbol_t valid = 0;

        EXPECT_EQ(valid, vtarget);
    } else {
        return;
    }
}

TEST(MemcpyToSymbol, ByAddress) {
    hipError_t ret;
    void * ptr;

    ret = hipGetSymbolAddress(&ptr, HIP_SYMBOL(device_symbol));
    ASSERT_EQ(hipSuccess, device_symbol);

    ret = hipMemset(ptr, 0xAA, sizeof(symbol_t));
    ASSERT_EQ(hipSuccess, device_symbol);

    symbol_t target;
    ret = hipMemcpyToSymbol(HIP_SYMBOL(ptr), &target, sizeof(symbol_t), 0,
        hipMemcpyHostToDevice);
    EXPECT_EQ(hipErrorInvalidSymbol, ret);
}

TEST(MemcpyToSymbol, DeviceToDevice) {
    hipError_t ret;
    void * device_ptr;
    void * symbol_ptr;

    ret = hipMalloc(&device_ptr, sizeof(symbol_t));
    ASSERT_EQ(hipSuccess, ret);

    ret = hipGetSymbolAddress(&symbol_ptr, HIP_SYMBOL(device_symbol));
    ASSERT_EQ(hipSuccess, device_symbol);

    const int pattern = 0xAA;
    ret = hipMemset(device_ptr, pattern, sizeof(symbol_t));
    ASSERT_EQ(hipSuccess, ret);

    ret = hipMemcpyToSymbol(HIP_SYMBOL(device_symbol), device_ptr,
        sizeof(symbol_t), 0, hipMemcpyDeviceToDevice);
    EXPECT_EQ(hipSuccess, ret);

    if (ret == hipSuccess) {
        symbol_t expected, target, vtarget;
        BOOST_STATIC_ASSERT(sizeof(symbol_t) == sizeof(device_symbol));
        ret = hipMemcpy(&target, symbol_ptr, sizeof(symbol_t),
            hipMemcpyDeviceToHost);
        ASSERT_EQ(hipSuccess, ret);

        memset(&expected, pattern, sizeof(expected));

        int valgrind = VALGRIND_GET_VBITS(&target, &vtarget, sizeof(symbol_t));
        assert(valgrind == 0 || valgrind == 1);

        EXPECT_EQ(expected, target);

        if (valgrind == 1) {
            const uint32_t valid = 0;
            BOOST_STATIC_ASSERT(sizeof(valid) == sizeof(target));

            EXPECT_EQ(valid, vtarget);
        }
    }

    ret = hipFree(device_ptr);
    ASSERT_EQ(hipSuccess, ret);
}

TEST(MemcpyToSymbol, InvalidDirections) {
    hipError_t ret;
    symbol_t target;

    ret = hipMemcpyToSymbol(HIP_SYMBOL(device_symbol), &target,
        sizeof(symbol_t), 0, hipMemcpyDeviceToHost);
    EXPECT_EQ(hipErrorInvalidMemcpyDirection, ret);

    ret = hipMemcpyToSymbol(HIP_SYMBOL(device_symbol), &target,
        sizeof(symbol_t), 0, hipMemcpyHostToHost);
    EXPECT_EQ(hipErrorInvalidMemcpyDirection, ret);

    /* hipMemcpyDefault implemented in test_memcpytosymbol.cu */
}

TEST(MemcpyToSymbol, NonSymbol) {
    hipError_t ret;
    symbol_t target;
    void * device_ptr;

    ret = hipMalloc(&device_ptr, sizeof(symbol_t));
    ASSERT_EQ(hipSuccess, ret);

    ret = hipMemcpyToSymbol(HIP_SYMBOL(device_ptr), &target,
        sizeof(symbol_t), 0, hipMemcpyHostToDevice);
    EXPECT_EQ(hipErrorInvalidSymbol, ret);

    ret = hipFree(device_ptr);
    ASSERT_EQ(hipSuccess, ret);
}

TEST(MemcpyToSymbol, OutOfBounds) {
    hipError_t ret;
    symbol_t target[2];

    ret = hipMemcpyToSymbol(HIP_SYMBOL(device_symbol), target,
        sizeof(symbol_t), sizeof(symbol_t), hipMemcpyHostToDevice);
    ASSERT_EQ(hipErrorInvalidValue, ret);
}

TEST(MemcpyToSymbol, SymbolToSymbol) {
    hipError_t ret;
    void * device_symbol2_ptr;

    ret = hipGetSymbolAddress(&device_symbol2_ptr, HIP_SYMBOL(device_symbol2));
    ASSERT_EQ(hipSuccess, ret);

    BOOST_STATIC_ASSERT(sizeof(device_symbol) == sizeof(device_symbol2));
    ret = hipMemcpyToSymbol(HIP_SYMBOL(device_symbol), device_symbol2_ptr,
        sizeof(symbol_t), 0, hipMemcpyDeviceToDevice);
    EXPECT_EQ(hipSuccess, ret);
}

int main(int argc, char **argv) {
    ::testing::InitGoogleTest(&argc, argv);
    return RUN_ALL_TESTS();
}
