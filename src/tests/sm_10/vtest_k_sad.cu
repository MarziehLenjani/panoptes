#include "hip/hip_runtime.h"
/**
 * Panoptes - A Binary Translation Framework for CUDA
 * (c) 2011-2012 Chris Kennelly <chris@ckennelly.com>
 *
 * This program is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.
 */

#include <hip/hip_runtime.h>
#include <gtest/gtest.h>
#include <stdint.h>
#include <valgrind/memcheck.h>

__global__ void k_sad(const int * x, const int * y, const unsigned int * z,
        unsigned int * out, int32_t n) {
    for (int32_t i = threadIdx.x + blockIdx.x * blockDim.x;
            i < n; i += blockDim.x * gridDim.x) {
        out[i] = __sad(x[i], y[i], z[i]);
    }
}

__global__ void k_sad_allconst(unsigned int * out) {
    unsigned int _out;
    asm("sad.u32 %0, 1, 2, 3;\n" : "=r"(_out));
    *out = _out;
}

TEST(kSAD, SADConstant) {
    hipError_t ret;
    hipStream_t stream;

    unsigned int * out;
    ret = hipMalloc((void **) &out, sizeof(*out));
    ASSERT_EQ(hipSuccess, ret);

    ret = hipStreamCreate(&stream);
    ASSERT_EQ(hipSuccess, ret);

    k_sad_allconst<<<1, 1, 0, stream>>>(out);

    ret = hipStreamSynchronize(stream);
    EXPECT_EQ(hipSuccess, ret);

    ret = hipStreamDestroy(stream);
    ASSERT_EQ(hipSuccess, ret);

    unsigned int hout;
    ret = hipMemcpy(&hout, out, sizeof(hout), hipMemcpyDeviceToHost);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipFree(out);
    ASSERT_EQ(hipSuccess, ret);

    EXPECT_EQ(4u, hout);
}

__global__ void k_sad_nonconstA(unsigned int * out, unsigned int A) {
    unsigned int _out;
    asm("sad.u32 %0, %1, 2, 3;\n" : "=r"(_out) : "r"(A));
    *out = _out;
}

__global__ void k_sad_nonconstB(unsigned int * out, unsigned int B) {
    unsigned int _out;
    asm("sad.u32 %0, 1, %1, 3;\n" : "=r"(_out) : "r"(B));
    *out = _out;
}

__global__ void k_sad_nonconstC(unsigned int * out, unsigned int C) {
    unsigned int _out;
    asm("sad.u32 %0, 1, 2, %1;\n" : "=r"(_out) : "r"(C));
    *out = _out;
}

TEST(kSAD, SADPartialConstant) {
    hipError_t ret;
    hipStream_t stream;

    const unsigned int A   = 1;
          unsigned int Ain = A;
    const unsigned int B   = 2;
          unsigned int Bin = B;
    const unsigned int C   = 3;
          unsigned int Cin = C;

    VALGRIND_MAKE_MEM_UNDEFINED(&Ain, sizeof(Ain));
    VALGRIND_MAKE_MEM_UNDEFINED(&Bin, sizeof(Bin));
    VALGRIND_MAKE_MEM_UNDEFINED(&Cin, sizeof(Cin));

    unsigned int * out;
    ret = hipMalloc((void **) &out, 6 * sizeof(*out));
    ASSERT_EQ(hipSuccess, ret);

    ret = hipStreamCreate(&stream);
    ASSERT_EQ(hipSuccess, ret);

    k_sad_nonconstA<<<1, 1, 0, stream>>>(out + 0, A  );
    k_sad_nonconstA<<<1, 1, 0, stream>>>(out + 1, Ain);
    k_sad_nonconstB<<<1, 1, 0, stream>>>(out + 2, B  );
    k_sad_nonconstB<<<1, 1, 0, stream>>>(out + 3, Bin);
    k_sad_nonconstC<<<1, 1, 0, stream>>>(out + 4, C  );
    k_sad_nonconstC<<<1, 1, 0, stream>>>(out + 5, Cin);

    ret = hipStreamSynchronize(stream);
    EXPECT_EQ(hipSuccess, ret);

    ret = hipStreamDestroy(stream);
    ASSERT_EQ(hipSuccess, ret);

    unsigned int hout[6];
    ret = hipMemcpy(&hout, out, sizeof(hout), hipMemcpyDeviceToHost);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipFree(out);
    ASSERT_EQ(hipSuccess, ret);

    EXPECT_EQ(4u, hout[0]);
    EXPECT_EQ(4u, hout[2]);
    EXPECT_EQ(4u, hout[4]);

    unsigned int vout[6];
    const int vret = VALGRIND_GET_VBITS(hout, vout, sizeof(hout));
    if (vret == 1) {
        EXPECT_EQ( 0, vout[0]);
        EXPECT_EQ(-1, vout[1]);
        EXPECT_EQ( 0, vout[2]);
        EXPECT_EQ(-1, vout[3]);
        EXPECT_EQ( 0, vout[4]);
        EXPECT_EQ(-1, vout[5]);
    }
}

TEST(kSAD, SAD) {
    hipError_t ret;
    hipStream_t stream;

    const int32_t n = 1 << 24;
    int * x;
    int * y;
    unsigned int * z;
    unsigned int * out;

    ret = hipMalloc((void **) &x, sizeof(*x) * n);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipMalloc((void **) &y, sizeof(*y) * n);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipMalloc((void **) &z, sizeof(*z) * n);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipMalloc((void **) &out, sizeof(*out) * n);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipStreamCreate(&stream);
    ASSERT_EQ(hipSuccess, ret);

    k_sad<<<256, 16, 0, stream>>>(x, y, z, out, n);

    ret = hipStreamSynchronize(stream);
    EXPECT_EQ(hipSuccess, ret);

    ret = hipStreamDestroy(stream);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipFree(x);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipFree(y);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipFree(z);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipFree(out);
    ASSERT_EQ(hipSuccess, ret);
}

__global__ void k_usad(const unsigned int * x, const unsigned int * y,
        const unsigned int * z, unsigned int * out, int32_t n) {
    for (int32_t i = threadIdx.x + blockIdx.x * blockDim.x;
            i < n; i += blockDim.x * gridDim.x) {
        out[i] = __sad(x[i], y[i], z[i]);
    }
}

TEST(kSAD, USAD) {
    hipError_t ret;
    hipStream_t stream;

    const int32_t n = 1 << 24;
    unsigned int * x;
    unsigned int * y;
    unsigned int * z;
    unsigned int * out;

    ret = hipMalloc((void **) &x, sizeof(*x) * n);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipMalloc((void **) &y, sizeof(*y) * n);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipMalloc((void **) &z, sizeof(*z) * n);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipMalloc((void **) &out, sizeof(*out) * n);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipStreamCreate(&stream);
    ASSERT_EQ(hipSuccess, ret);

    k_usad<<<256, 16, 0, stream>>>(x, y, z, out, n);

    ret = hipStreamSynchronize(stream);
    EXPECT_EQ(hipSuccess, ret);

    ret = hipStreamDestroy(stream);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipFree(x);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipFree(y);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipFree(z);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipFree(out);
    ASSERT_EQ(hipSuccess, ret);
}

int main(int argc, char **argv) {
    ::testing::InitGoogleTest(&argc, argv);
    return RUN_ALL_TESTS();
}
