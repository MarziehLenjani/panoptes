#include "hip/hip_runtime.h"
/**
 * Panoptes - A Binary Translation Framework for CUDA
 * (c) 2011-2012 Chris Kennelly <chris@ckennelly.com>
 *
 * This program is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.
 */

#include <hip/hip_runtime.h>
#include <gtest/gtest.h>
#include <stdint.h>
#include <valgrind/memcheck.h>

static __global__ void k_not_const1(uint32_t * out) {
    uint32_t _out;
    asm("not.b32 %0, 1234567890;\n" : "=r"(_out));
    *out = _out;
}

static __global__ void k_not_constA(uint32_t * out, uint32_t in) {
    uint32_t _out;
    asm("not.b32 %0, %1;\n" : "=r"(_out) : "r"(in));
    *out = _out;
}

TEST(Not, BinaryConstant) {
    hipError_t ret;
    hipStream_t stream;

    uint32_t * out;
    ret = hipMalloc((void **) &out, 3 * sizeof(*out));
    ASSERT_EQ(hipSuccess, ret);

    ret = hipStreamCreate(&stream);
    ASSERT_EQ(hipSuccess, ret);

    const uint32_t in = 987654321;
    uint32_t invalid_in = in;
    VALGRIND_MAKE_MEM_UNDEFINED(&invalid_in, sizeof(invalid_in));
    k_not_const1<<<1, 1, 0, stream>>>(out + 0);
    k_not_constA<<<1, 1, 0, stream>>>(out + 1, in);
    k_not_constA<<<1, 1, 0, stream>>>(out + 2, invalid_in);

    ret = hipStreamSynchronize(stream);
    EXPECT_EQ(hipSuccess, ret);

    ret = hipStreamDestroy(stream);
    ASSERT_EQ(hipSuccess, ret);

    uint32_t hout[3];
    ret = hipMemcpy(&hout, out, sizeof(hout), hipMemcpyDeviceToHost);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipFree(out);
    ASSERT_EQ(hipSuccess, ret);

    EXPECT_EQ(~1234567890, hout[0]);
    EXPECT_EQ(~987654321,  hout[1]);

    uint32_t vout[3];
    const int vret = VALGRIND_GET_VBITS(&hout, &vout, sizeof(hout));
    if (vret == 1) {
        EXPECT_EQ(0x00000000, vout[0]);
        EXPECT_EQ(0x00000000, vout[1]);
        EXPECT_EQ(0xFFFFFFFF, vout[2]);
    }
}

static __global__ void k_not_constp0(uint32_t * out) {
    uint32_t _out;
    asm("{ .reg .pred %tmp;\n"
        "not.pred %tmp, 0;\n"
        "selp.u32 %0, 1, 0, %tmp; }\n" : "=r"(_out));
    *out = _out;
}

static __global__ void k_not_constp1(uint32_t * out) {
    uint32_t _out;
    asm("{ .reg .pred %tmp;\n"
        "not.pred %tmp, 1;\n"
        "selp.u32 %0, 1, 0, %tmp; }\n" : "=r"(_out));
    *out = _out;
}

static __global__ void k_not_constpA(uint32_t * out, const uint32_t in) {
    uint32_t _out;
    asm("{ .reg .pred %tmp<2>;\n"
        "setp.ne.u32 %tmp0, %1, 0;\n"
        "not.pred %tmp1, %tmp0;\n"
        "selp.u32 %0, 1, 0, %tmp1; }\n" : "=r"(_out) : "r"(in));
    *out = _out;
}

TEST(Not, PredicateConstant) {
    hipError_t ret;
    hipStream_t stream;

    uint32_t * out;
    ret = hipMalloc((void **) &out, 4 * sizeof(*out));
    ASSERT_EQ(hipSuccess, ret);

    ret = hipStreamCreate(&stream);
    ASSERT_EQ(hipSuccess, ret);

    const uint32_t in   = 1;
    uint32_t invalid_in = in;
    VALGRIND_MAKE_MEM_UNDEFINED(&invalid_in, sizeof(invalid_in));
    k_not_constp0<<<1, 1, 0, stream>>>(out + 0);
    k_not_constp1<<<1, 1, 0, stream>>>(out + 1);
    k_not_constpA<<<1, 1, 0, stream>>>(out + 2, in);
    k_not_constpA<<<1, 1, 0, stream>>>(out + 3, invalid_in);

    ret = hipStreamSynchronize(stream);
    EXPECT_EQ(hipSuccess, ret);

    ret = hipStreamDestroy(stream);
    ASSERT_EQ(hipSuccess, ret);

    uint32_t hout[4];
    ret = hipMemcpy(&hout, out, sizeof(hout), hipMemcpyDeviceToHost);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipFree(out);
    ASSERT_EQ(hipSuccess, ret);

    EXPECT_EQ(1,           hout[0]);
    EXPECT_EQ(0,           hout[1]);
    EXPECT_EQ((~in) & 0x1, hout[2]);

    uint32_t vout[4];
    const int vret = VALGRIND_GET_VBITS(&hout, &vout, sizeof(hout));
    if (vret == 1) {
        EXPECT_EQ(0x00000000, vout[0]);
        EXPECT_EQ(0x00000000, vout[1]);
        EXPECT_EQ(0x00000000, vout[2]);
        EXPECT_EQ(0xFFFFFFFF, vout[3]);
    }
}

int main(int argc, char **argv) {
    ::testing::InitGoogleTest(&argc, argv);
    return RUN_ALL_TESTS();
}
