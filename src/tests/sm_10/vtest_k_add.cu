#include "hip/hip_runtime.h"
/**
 * Panoptes - A Binary Translation Framework for CUDA
 * (c) 2011-2012 Chris Kennelly <chris@ckennelly.com>
 *
 * This program is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.
 */

#include <hip/hip_runtime.h>
#include <gtest/gtest.h>
#include <stdint.h>
#include <valgrind/memcheck.h>

static __global__ void k_add_const11(float * out) {
    float _out;
    asm volatile("add.f32 %0, 0f3f800000, 0f3f800000;\n" : "=f"(_out));
    *out = _out;
}

static __global__ void k_add_constA1(float * out, float in) {
    float _out;
    asm volatile("add.f32 %0, %1, 0f3f800000;\n" : "=f"(_out) : "f"(in));
    *out = _out;
}

static __global__ void k_add_const1B(float * out, float in) {
    float _out;
    asm volatile("add.f32 %0, 0f3f800000, %1;\n" : "=f"(_out) : "f"(in));
    *out = _out;
}

TEST(Add, Constant) {
    hipError_t ret;
    hipStream_t stream;

    float * out;
    ret = hipMalloc((void **) &out, 5 * sizeof(*out));
    ASSERT_EQ(hipSuccess, ret);

    ret = hipStreamCreate(&stream);
    ASSERT_EQ(hipSuccess, ret);

    const float in = 1.f;
    float invalid_in = in;
    VALGRIND_MAKE_MEM_UNDEFINED(&invalid_in, sizeof(invalid_in));
    k_add_const11<<<1, 1, 0, stream>>>(out + 0);
    k_add_constA1<<<1, 1, 0, stream>>>(out + 1, in);
    k_add_constA1<<<1, 1, 0, stream>>>(out + 2, invalid_in);
    k_add_const1B<<<1, 1, 0, stream>>>(out + 3, in);
    k_add_const1B<<<1, 1, 0, stream>>>(out + 4, invalid_in);

    ret = hipStreamSynchronize(stream);
    EXPECT_EQ(hipSuccess, ret);

    ret = hipStreamDestroy(stream);
    ASSERT_EQ(hipSuccess, ret);

    float hout[5];
    ret = hipMemcpy(&hout, out, sizeof(hout), hipMemcpyDeviceToHost);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipFree(out);
    ASSERT_EQ(hipSuccess, ret);

    EXPECT_EQ(2.f,      hout[0]);
    EXPECT_EQ(1.f + in, hout[1]);
    EXPECT_EQ(1.f + in, hout[1]);

    uint32_t vout[5];
    const int vret = VALGRIND_GET_VBITS(&hout, &vout, sizeof(hout));
    if (vret == 1) {
        EXPECT_EQ(0x00000000, vout[0]);
        EXPECT_EQ(0x00000000, vout[1]);
        EXPECT_EQ(0xFFFFFFFF, vout[2]);
        EXPECT_EQ(0x00000000, vout[3]);
        EXPECT_EQ(0xFFFFFFFF, vout[4]);
    }
}

int main(int argc, char **argv) {
    ::testing::InitGoogleTest(&argc, argv);
    return RUN_ALL_TESTS();
}
