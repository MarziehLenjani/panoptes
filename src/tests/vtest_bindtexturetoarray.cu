/**
 * Panoptes - A Binary Translation Framework for CUDA
 * (c) 2011-2012 Chris Kennelly <chris@ckennelly.com>
 *
 * This program is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.
 */

#include <hip/hip_runtime.h>
#include <gtest/gtest.h>
#include <stdint.h>

texture<int32_t, 1, hipReadModeElementType> tex_src;

TEST(BindTextureToArray, Simple) {
    hipError_t ret;
    const struct textureReference * texref;
    hipArray * array;

    const uint32_t ints = 1u << 16;

    struct hipChannelFormatDesc desc;
    desc.f = hipChannelFormatKindSigned;
    desc.x = 32;
    desc.y = desc.z = desc.w = 0;

    ret = hipMallocArray(&array, &desc, ints, 0, 0);
    ASSERT_EQ(hipSuccess, ret);

    int version;
    ret = hipRuntimeGetVersion(&version);
    ASSERT_EQ(hipSuccess, ret);

    const void * ptr;
    if (version < 5000 /* 5.0 */) {
        ptr = "tex_src";
    } else {
        ptr = &tex_src;
    }
    ret = hipGetTextureReference(&texref, HIP_SYMBOL(ptr));
    ASSERT_EQ(hipSuccess, ret);

    ret = hipBindTextureToArray(texref, array, &desc);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipUnbindTexture(tex_src);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipFreeArray(array);
    ASSERT_EQ(hipSuccess, ret);
}

TEST(BindTextureToArray, FreeBeforeUnbind) {
    hipError_t ret;
    const struct textureReference * texref;
    hipArray * array;

    const uint32_t ints = 1u << 16;

    struct hipChannelFormatDesc desc;
    desc.f = hipChannelFormatKindSigned;
    desc.x = 32;
    desc.y = desc.z = desc.w = 0;

    ret = hipMallocArray(&array, &desc, ints, 0, 0);
    ASSERT_EQ(hipSuccess, ret);

    int version;
    ret = hipRuntimeGetVersion(&version);
    ASSERT_EQ(hipSuccess, ret);

    const void * ptr;
    if (version < 5000 /* 5.0 */) {
        ptr = "tex_src";
    } else {
        ptr = &tex_src;
    }
    ret = hipGetTextureReference(&texref, HIP_SYMBOL(ptr));
    ASSERT_EQ(hipSuccess, ret);

    ret = hipBindTextureToArray(texref, array, &desc);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipFreeArray(array);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipUnbindTexture(tex_src);
    ASSERT_EQ(hipSuccess, ret);
}

TEST(BindTextureToArray, NullArguments) {
    hipError_t ret;
    const struct textureReference * texref;
    hipArray * array;

    const uint32_t ints = 1u << 16;

    struct hipChannelFormatDesc desc;
    desc.f = hipChannelFormatKindSigned;
    desc.x = 32;
    desc.y = desc.z = desc.w = 0;

    ret = hipMallocArray(&array, &desc, ints, 0, 0);
    ASSERT_EQ(hipSuccess, ret);

    int version;
    ret = hipRuntimeGetVersion(&version);
    ASSERT_EQ(hipSuccess, ret);

    const void * ptr;
    if (version < 5000 /* 5.0 */) {
        ptr = "tex_src";
    } else {
        ptr = &tex_src;
    }
    ret = hipGetTextureReference(&texref, HIP_SYMBOL(ptr));
    ASSERT_EQ(hipSuccess, ret);

/* SIGSEGV
    ret = hipBindTextureToArray(texref, array, NULL);
    EXPECT_EQ(hipSuccess, ret);
    */

    ret = hipBindTextureToArray(texref, NULL,  &desc);
    EXPECT_EQ(hipErrorInvalidHandle, ret);
/* SIGSEGV
    ret = hipBindTextureToArray(texref, NULL,  NULL);
    EXPECT_EQ(hipSuccess, ret);
*/
    ret = hipBindTextureToArray(NULL,   array, &desc);
    EXPECT_EQ(hipErrorInvalidTexture, ret);

    ret = hipBindTextureToArray(NULL,   array, NULL);
    EXPECT_EQ(hipErrorInvalidTexture, ret);

    ret = hipBindTextureToArray(NULL,   NULL,  &desc);
    EXPECT_EQ(hipErrorInvalidTexture, ret);

    ret = hipBindTextureToArray(NULL,   NULL,  NULL);
    EXPECT_EQ(hipErrorInvalidTexture, ret);

    /* We never bound anything successfully, so no need for
       hipUnbindTexture. */

    ret = hipFreeArray(array);
    ASSERT_EQ(hipSuccess, ret);
}

int main(int argc, char **argv) {
    ::testing::InitGoogleTest(&argc, argv);
    return RUN_ALL_TESTS();
}
