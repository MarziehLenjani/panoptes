#include "hip/hip_runtime.h"
/**
 * Panoptes - A Binary Translation Framework for CUDA
 * (c) 2011-2012 Chris Kennelly <chris@ckennelly.com>
 *
 * This program is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.
 */

#include <hip/hip_runtime.h>
#include <gtest/gtest.h>

extern "C" __global__ void k_rcpf(const float * in,
        const int N, float * out) {
    for (int idx = threadIdx.x + blockDim.x * blockIdx.x;
            idx < N; idx += blockDim.x * gridDim.x) {
        out[idx] = 1.f / in[idx];
    }
}

extern "C" __global__ void k_rcpfrz(const float * in,
        const int N, float * out) {
    for (int idx = threadIdx.x + blockDim.x * blockIdx.x;
            idx < N; idx += blockDim.x * gridDim.x) {
        out[idx] = __frcp_rz(in[idx]);
    }
}

extern "C" __global__ void k_rcp(const double * in,
        const int N, double * out) {
    for (int idx = threadIdx.x + blockDim.x * blockIdx.x;
            idx < N; idx += blockDim.x * gridDim.x) {
        out[idx] = 1. / in[idx];
    }
}

TEST(kRCP, SinglePrecision) {
    hipError_t ret;
    hipStream_t stream;

    const int N = 1 << 20;
    const int n_blocks = 32;

    float * in;
    float * out;

    ret = hipMalloc((void **) &in, sizeof(*in) * N);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipMalloc((void **) &out, sizeof(*out) * N);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipStreamCreate(&stream);
    ASSERT_EQ(hipSuccess, ret);

    k_rcpf<<<256, n_blocks, 0, stream>>>(in, N, out);

    ret = hipStreamSynchronize(stream);
    EXPECT_EQ(hipSuccess, ret);

    ret = hipStreamDestroy(stream);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipFree(in);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipFree(out);
    ASSERT_EQ(hipSuccess, ret);
}

TEST(kRCP, RZ) {
    hipError_t ret;
    hipStream_t stream;

    const int N = 1 << 20;
    const int n_blocks = 32;

    float * in;
    float * out;

    ret = hipMalloc((void **) &in, sizeof(*in) * N);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipMalloc((void **) &out, sizeof(*out) * N);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipStreamCreate(&stream);
    ASSERT_EQ(hipSuccess, ret);

    k_rcpfrz<<<256, n_blocks, 0, stream>>>(in, N, out);

    ret = hipStreamSynchronize(stream);
    EXPECT_EQ(hipSuccess, ret);

    ret = hipStreamDestroy(stream);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipFree(in);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipFree(out);
    ASSERT_EQ(hipSuccess, ret);
}

TEST(kRCP, DoublePrecision) {
    hipError_t ret;
    hipStream_t stream;

    const int N = 1 << 20;
    const int n_blocks = 32;

    double * in;
    double * out;

    ret = hipMalloc((void **) &in, sizeof(*in) * N);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipMalloc((void **) &out, sizeof(*out) * N);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipStreamCreate(&stream);
    ASSERT_EQ(hipSuccess, ret);

    k_rcp<<<256, n_blocks, 0, stream>>>(in, N, out);

    ret = hipStreamSynchronize(stream);
    EXPECT_EQ(hipSuccess, ret);

    ret = hipStreamDestroy(stream);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipFree(in);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipFree(out);
    ASSERT_EQ(hipSuccess, ret);
}

int main(int argc, char **argv) {
    ::testing::InitGoogleTest(&argc, argv);
    return RUN_ALL_TESTS();
}
