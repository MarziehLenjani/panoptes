#include "hip/hip_runtime.h"
/**
 * Panoptes - A Binary Translation Framework for CUDA
 * (c) 2011-2012 Chris Kennelly <chris@ckennelly.com>
 *
 * This program is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.
 */

#include <boost/static_assert.hpp>
#include <hip/hip_runtime.h>
#include <gtest/gtest.h>
#include <stdint.h>

template<typename T>
class BitfieldTestFixture : public ::testing::Test {
public:
    BitfieldTestFixture() { }
    ~BitfieldTestFixture() { }

    void SetUp() {
        hipError_t ret;
        ret = hipMalloc((void **) &d, sizeof(*d) * n);
        ASSERT_EQ(hipSuccess, ret);

        ret = hipMalloc((void **) &a, sizeof(*a) * n);
        ASSERT_EQ(hipSuccess, ret);

        ret = hipMalloc((void **) &f, sizeof(*a) * n);
        ASSERT_EQ(hipSuccess, ret);

        ret = hipStreamCreate(&stream);
        ASSERT_EQ(hipSuccess, ret);
    }

    void TearDown() {
        hipError_t ret;
        ret = hipFree(d);
        ASSERT_EQ(hipSuccess, ret);

        ret = hipFree(a);
        ASSERT_EQ(hipSuccess, ret);

        ret = hipFree(f);
        ASSERT_EQ(hipSuccess, ret);

        ret = hipStreamSynchronize(stream);
        ASSERT_EQ(hipSuccess, ret);

        ret = hipStreamDestroy(stream);
        ASSERT_EQ(hipSuccess, ret);
    }

    static const int32_t n = 1 << 20;
    /**
     * When testing insertions, we use d as field b.
     */
    T * d;
    T * a;
    T * f;
    T tmp;
    hipStream_t stream;
};

TYPED_TEST_CASE_P(BitfieldTestFixture);

template<typename T>
static __device__ __inline__ uint32_t bitfind(const T & a) {
    BOOST_STATIC_ASSERT(sizeof(T) == 0);
    return 0; /* Suppress warning */
}

template<>
__device__ __inline__ uint32_t bitfind(const uint32_t & a) {
    uint32_t ret;
    asm volatile("bfind.u32 %0, %1;\n" : "=r"(ret) : "r"(a));
    return ret;
}

template<>
__device__ __inline__ uint32_t bitfind(const uint64_t & a) {
    uint32_t ret;
    asm volatile("bfind.u64 %0, %1;\n" : "=r"(ret) : "l"(a));
    return ret;
}

template<>
__device__ __inline__ uint32_t bitfind(const int32_t & a) {
    uint32_t ret;
    asm volatile("bfind.s32 %0, %1;\n" : "=r"(ret) : "r"(a));
    return ret;
}

template<>
__device__ __inline__ uint32_t bitfind(const int64_t & a) {
    uint32_t ret;
    asm volatile("bfind.s64 %0, %1;\n" : "=r"(ret) : "l"(a));
    return ret;
}

template<typename T>
static __device__ __inline__ uint32_t bitfindshift(const T & a) {
    BOOST_STATIC_ASSERT(sizeof(T) == 0);
    return 0; /* Suppress warning */
}

template<>
__device__ __inline__ uint32_t bitfindshift(const uint32_t & a) {
    uint32_t ret;
    asm volatile("bfind.shiftamt.u32 %0, %1;\n" : "=r"(ret) : "r"(a));
    return ret;
}

template<>
__device__ __inline__ uint32_t bitfindshift(const uint64_t & a) {
    uint32_t ret;
    asm volatile("bfind.shiftamt.u64 %0, %1;\n" : "=r"(ret) : "l"(a));
    return ret;
}

template<>
__device__ __inline__ uint32_t bitfindshift(const int32_t & a) {
    uint32_t ret;
    asm volatile("bfind.shiftamt.s32 %0, %1;\n" : "=r"(ret) : "r"(a));
    return ret;
}

template<>
__device__ __inline__ uint32_t bitfindshift(const int64_t & a) {
    uint32_t ret;
    asm volatile("bfind.shiftamt.s64 %0, %1;\n" : "=r"(ret) : "l"(a));
    return ret;
}

template<typename T>
__global__ void k_bitfind(uint32_t * d, T a) {
    uint32_t normal = bitfind(a);
    uint32_t shift  = bitfindshift(a);
    d[0] = normal;
    d[1] = sizeof(T) * CHAR_BIT - (1 + shift + normal);
}

TYPED_TEST_P(BitfieldTestFixture, FindSingle) {
    hipError_t ret;

    this->tmp = 5;
    uint32_t exp[2] = {2, 0};
    uint32_t * d;
    ret = hipMalloc((void **) &d, sizeof(exp));
    assert(ret == hipSuccess);

    k_bitfind<<<1, 1, 0, this->stream>>>(d, this->tmp);

    ret = hipStreamSynchronize(this->stream);
    assert(ret == hipSuccess);

    uint32_t hd[2];
    BOOST_STATIC_ASSERT(sizeof(hd[0]) == sizeof(d[0]));
    BOOST_STATIC_ASSERT(sizeof(hd) == sizeof(exp));
    ret = hipMemcpy(&hd, d, sizeof(hd), hipMemcpyDeviceToHost);
    assert(ret == hipSuccess);

    assert(hd[0] == exp[0]);
    assert(hd[1] == exp[1]);

    ret = hipFree(d);
    assert(ret == hipSuccess);
}

REGISTER_TYPED_TEST_CASE_P(BitfieldTestFixture, FindSingle);

typedef ::testing::Types<int32_t, uint32_t, int64_t, uint64_t> MyTypes;
INSTANTIATE_TYPED_TEST_CASE_P(My, BitfieldTestFixture, MyTypes);

static __global__ void k_bitfind_const(uint4 * out) {
    uint4 _out;
    asm volatile(
        "bfind.u32 %0, 1;\n"
        "bfind.s32 %1, -1;\n"
        "bfind.u64 %2, 4;\n"
        "bfind.s64 %3, -2;\n" : "=r"(_out.x),  "=r"(_out.y),
                                  "=r"(_out.z),  "=r"(_out.w));
    *out = _out;
}

TEST(Bitfind, Constant) {
    hipError_t ret;
    hipStream_t stream;

    uint4 * out;
    ret = hipMalloc((void **) &out, sizeof(*out));
    ASSERT_EQ(hipSuccess, ret);

    ret = hipStreamCreate(&stream);
    ASSERT_EQ(hipSuccess, ret);

    k_bitfind_const<<<1, 1, 0, stream>>>(out);

    ret = hipStreamSynchronize(stream);
    EXPECT_EQ(hipSuccess, ret);

    ret = hipStreamDestroy(stream);
    ASSERT_EQ(hipSuccess, ret);

    uint4 hout;
    ret = hipMemcpy(&hout, out, sizeof(hout), hipMemcpyDeviceToHost);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipFree(out);
    ASSERT_EQ(hipSuccess, ret);

    EXPECT_EQ(0x00000000, hout.x);
    EXPECT_EQ(0xFFFFFFFF, hout.y);
    EXPECT_EQ(0x00000002, hout.z);
    EXPECT_EQ(0x00000000, hout.w);
}

int main(int argc, char **argv) {
    ::testing::InitGoogleTest(&argc, argv);
    return RUN_ALL_TESTS();
}
