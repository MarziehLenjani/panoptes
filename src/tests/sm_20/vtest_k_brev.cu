#include "hip/hip_runtime.h"
/**
 * Panoptes - A Binary Translation Framework for CUDA
 * (c) 2011-2012 Chris Kennelly <chris@ckennelly.com>
 *
 * This program is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.
 */

#include <hip/hip_runtime.h>
#include <gtest/gtest.h>
#include <stdint.h>
#include <valgrind/memcheck.h>

extern "C" __global__ void k_brev(const int32_t * in, int * out, int n) {
    for (int idx = threadIdx.x + blockDim.x * blockIdx.x;
            idx < n; idx += blockDim.x * gridDim.x) {
        out[idx] = __brev(in[idx]);
    }
}

extern "C" __global__ void k_brevll(const int64_t * in, int * out, int n) {
    for (int idx = threadIdx.x + blockDim.x * blockIdx.x;
            idx < n; idx += blockDim.x * gridDim.x) {
        out[idx] = __brevll(in[idx]);
    }
}

TEST(kBREV, Int32) {
    hipError_t ret;
    hipStream_t stream;

    const int N = 1 << 20;
    const int n_blocks = 32;

    int32_t * in;
    int * out;

    ret = hipMalloc((void **) &in, sizeof(*in) * N);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipMalloc((void **) &out, sizeof(*out) * N);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipStreamCreate(&stream);
    ASSERT_EQ(hipSuccess, ret);

    k_brev<<<256, n_blocks, 0, stream>>>(in, out, N);

    ret = hipStreamSynchronize(stream);
    EXPECT_EQ(hipSuccess, ret);

    ret = hipStreamDestroy(stream);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipFree(in);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipFree(out);
    ASSERT_EQ(hipSuccess, ret);
}

TEST(kBREV, Int64) {
    hipError_t ret;
    hipStream_t stream;

    const int N = 1 << 20;
    const int n_blocks = 32;

    int64_t * in;
    int * out;

    ret = hipMalloc((void **) &in, sizeof(*in) * N);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipMalloc((void **) &out, sizeof(*out) * N);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipStreamCreate(&stream);
    ASSERT_EQ(hipSuccess, ret);

    k_brevll<<<256, n_blocks, 0, stream>>>(in, out, N);

    ret = hipStreamSynchronize(stream);
    EXPECT_EQ(hipSuccess, ret);

    ret = hipStreamDestroy(stream);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipFree(in);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipFree(out);
    ASSERT_EQ(hipSuccess, ret);
}

extern "C" __global__ void k_brev_const(int32_t * out) {
    int32_t ret;
    asm volatile("brev.b32 %0, 16909060;\n" : "=r"(ret));
    *out = ret;
}

TEST(kBREV, Constant) {
    hipError_t ret;
    hipStream_t stream;

    int32_t * out;

    ret = hipMalloc((void **) &out, sizeof(*out));
    ASSERT_EQ(hipSuccess, ret);

    ret = hipStreamCreate(&stream);
    ASSERT_EQ(hipSuccess, ret);

    k_brev_const<<<1, 1, 0, stream>>>(out);

    ret = hipStreamSynchronize(stream);
    EXPECT_EQ(hipSuccess, ret);

    ret = hipStreamDestroy(stream);
    ASSERT_EQ(hipSuccess, ret);

    int32_t hout;
    ret = hipMemcpy(&hout, out, sizeof(hout), hipMemcpyDeviceToHost);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipFree(out);
    ASSERT_EQ(hipSuccess, ret);

    EXPECT_EQ(0x20C04080, hout);
    int32_t vout;
    const int vret = VALGRIND_GET_VBITS(&hout, &vout, sizeof(hout));
    if (vret == 1) {
        EXPECT_EQ(0x0, vout);
    }
}

int main(int argc, char **argv) {
    ::testing::InitGoogleTest(&argc, argv);
    return RUN_ALL_TESTS();
}
