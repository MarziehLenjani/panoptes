#include "hip/hip_runtime.h"
/**
 * Panoptes - A Binary Translation Framework for CUDA
 * (c) 2011-2012 Chris Kennelly <chris@ckennelly.com>
 *
 * This program is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.
 */

#include <hip/hip_runtime.h>
#include <gtest/gtest.h>

extern "C" __global__ void k_threadfence() {
    __threadfence();
}

extern "C" __global__ void k_threadfence_block() {
    __threadfence_block();
}

extern "C" __global__ void k_threadfence_system() {
    __threadfence_system();
}

TEST(kThreadFence, Global) {
    hipError_t ret;
    hipStream_t stream;

    ret = hipStreamCreate(&stream);
    ASSERT_EQ(hipSuccess, ret);

    k_threadfence<<<1, 1, 0, stream>>>();

    ret = hipStreamSynchronize(stream);
    EXPECT_EQ(hipSuccess, ret);

    ret = hipStreamDestroy(stream);
    ASSERT_EQ(hipSuccess, ret);
}

TEST(kThreadFence, Block) {
    hipError_t ret;
    hipStream_t stream;

    ret = hipStreamCreate(&stream);
    ASSERT_EQ(hipSuccess, ret);

    k_threadfence_block<<<1, 1, 0, stream>>>();

    ret = hipStreamSynchronize(stream);
    EXPECT_EQ(hipSuccess, ret);

    ret = hipStreamDestroy(stream);
    ASSERT_EQ(hipSuccess, ret);
}

TEST(kThreadFence, System) {
    hipError_t ret;
    hipStream_t stream;

    ret = hipStreamCreate(&stream);
    ASSERT_EQ(hipSuccess, ret);

    k_threadfence_system<<<1, 1, 0, stream>>>();

    ret = hipStreamSynchronize(stream);
    EXPECT_EQ(hipSuccess, ret);

    ret = hipStreamDestroy(stream);
    ASSERT_EQ(hipSuccess, ret);
}

int main(int argc, char **argv) {
    ::testing::InitGoogleTest(&argc, argv);
    return RUN_ALL_TESTS();
}
