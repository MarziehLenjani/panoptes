#include "hip/hip_runtime.h"
/**
 * Panoptes - A Binary Translation Framework for CUDA
 * (c) 2011-2012 Chris Kennelly <chris@ckennelly.com>
 *
 * This program is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.
 */

#include <hip/hip_runtime.h>
#include <gtest/gtest.h>
#include <stdint.h>
#include <valgrind/memcheck.h>

static __device__ bool vote_all(bool in_) {
    const int32_t in = in_;
    int out;

    asm volatile(
        "{\n"
        "  .reg .pred %tmp;\n"
        "  setp.ne.s32 %tmp, %1, 0;\n"
        "  vote.all.pred %tmp, %tmp;\n"
        "  selp.s32 %0, 1, 0, %tmp;\n"
        "}\n" : "=r"(out) : "r"(in));

    return out;
}

static __device__ bool vote_none(bool in_) {
    const int32_t in = in_;
    int out;

    asm volatile(
        "{\n"
        "  .reg .pred %tmp;\n"
        "  setp.ne.s32 %tmp, %1, 0;\n"
        "  vote.all.pred %tmp, !%tmp;\n"
        "  selp.s32 %0, 1, 0, %tmp;\n"
        "}\n" : "=r"(out) : "r"(in));

    return out;
}

static __device__ bool vote_any(bool in_) {
    const int32_t in = in_;
    int out;

    asm volatile(
        "{\n"
        "  .reg .pred %tmp;\n"
        "  setp.ne.s32 %tmp, %1, 0;\n"
        "  vote.any.pred %tmp, %tmp;\n"
        "  selp.s32 %0, 1, 0, %tmp;\n"
        "}\n" : "=r"(out) : "r"(in));

    return out;
}

static __device__ bool vote_notall(bool in_) {
    const int32_t in = in_;
    int out;

    asm volatile(
        "{\n"
        "  .reg .pred %tmp;\n"
        "  setp.ne.s32 %tmp, %1, 0;\n"
        "  vote.any.pred %tmp, !%tmp;\n"
        "  selp.s32 %0, 1, 0, %tmp;\n"
        "}\n" : "=r"(out) : "r"(in));

    return out;
}

__global__ void kv_all(bool * out, int threads, int test) {
    if (threadIdx.x >= threads) {
        return;
    }

    *out = vote_all(threadIdx.x < test);
}

__global__ void kv_none(bool * out, int threads, int test) {
    if (threadIdx.x >= threads) {
        return;
    }

    *out = vote_none(threadIdx.x < test);
}

__global__ void kv_any(bool * out, int threads, int test) {
    if (threadIdx.x >= threads) {
        return;
    }

    *out = vote_any(threadIdx.x < test);
}

__global__ void kv_notall(bool * out, int threads, int test) {
    if (threadIdx.x >= threads) {
        return;
    }

    *out = vote_notall(threadIdx.x < test);
}

/**
 * If threadIdx.x < test, in[0] is loaded as the input for the balloting.
 * Else, in[1] is loaded.
 */
__global__ void k_ballot(uint32_t * out, uint32_t * in, int test) {
    const uint32_t vote = in[threadIdx.x < test ? 0 : 1];
    *out = ballot(vote);
}

TEST(Vote, All) {
    hipError_t ret;
    int device;
    ret = hipGetDevice(&device);
    ASSERT_EQ(hipSuccess, ret);

    hipDeviceProp_t prop;
    ret = hipGetDeviceProperties(&prop, device);
    ASSERT_EQ(hipSuccess, ret);

    const int warpSize = prop.warpSize;

    bool * out;
    ret = hipMalloc((void **) &out, 5 * sizeof(*out));
    ASSERT_EQ(hipSuccess, ret);

    hipStream_t stream;

    ret = hipStreamCreate(&stream);
    ASSERT_EQ(hipSuccess, ret);

    kv_all<<<1, warpSize, 0, stream>>>(out + 0, warpSize,     warpSize);
    kv_all<<<1, warpSize, 0, stream>>>(out + 1, warpSize - 1, warpSize);
    kv_all<<<1, warpSize, 0, stream>>>(out + 2, warpSize,     warpSize - 1);
    kv_all<<<1, warpSize, 0, stream>>>(out + 3, warpSize - 1, warpSize - 1);
    kv_all<<<1, warpSize, 0, stream>>>(out + 4, warpSize - 1, warpSize - 2);

    ret = hipStreamSynchronize(stream);
    EXPECT_EQ(hipSuccess, ret);

    ret = hipStreamDestroy(stream);
    ASSERT_EQ(hipSuccess, ret);

    bool hout[5];
    ret = hipMemcpy(hout, out, sizeof(hout), hipMemcpyDeviceToHost);
    ASSERT_EQ(hipSuccess, ret);

    EXPECT_TRUE(hout[0]);
    EXPECT_TRUE(hout[1]);
    EXPECT_FALSE(hout[2]);
    EXPECT_TRUE(hout[3]);
    EXPECT_FALSE(hout[4]);

    ret = hipFree(out);
    ASSERT_EQ(hipSuccess, ret);
}

TEST(Vote, None) {
    hipError_t ret;
    int device;
    ret = hipGetDevice(&device);
    ASSERT_EQ(hipSuccess, ret);

    hipDeviceProp_t prop;
    ret = hipGetDeviceProperties(&prop, device);
    ASSERT_EQ(hipSuccess, ret);

    const int warpSize = prop.warpSize;

    bool * out;
    ret = hipMalloc((void **) &out, 7 * sizeof(*out));
    ASSERT_EQ(hipSuccess, ret);

    hipStream_t stream;

    ret = hipStreamCreate(&stream);
    ASSERT_EQ(hipSuccess, ret);

    kv_none<<<1, warpSize, 0, stream>>>(out + 0, warpSize,     warpSize);
    kv_none<<<1, warpSize, 0, stream>>>(out + 1, warpSize - 1, warpSize);
    kv_none<<<1, warpSize, 0, stream>>>(out + 2, warpSize,     warpSize - 1);
    kv_none<<<1, warpSize, 0, stream>>>(out + 3, warpSize - 1, warpSize - 1);
    kv_none<<<1, warpSize, 0, stream>>>(out + 4, warpSize - 1, warpSize - 2);
    kv_none<<<1, warpSize, 0, stream>>>(out + 5, warpSize,     0);
    kv_none<<<1, warpSize, 0, stream>>>(out + 6, warpSize - 1, 0);

    ret = hipStreamSynchronize(stream);
    EXPECT_EQ(hipSuccess, ret);

    ret = hipStreamDestroy(stream);
    ASSERT_EQ(hipSuccess, ret);

    bool hout[7];
    ret = hipMemcpy(hout, out, sizeof(hout), hipMemcpyDeviceToHost);
    ASSERT_EQ(hipSuccess, ret);

    EXPECT_FALSE(hout[0]);
    EXPECT_FALSE(hout[1]);
    EXPECT_FALSE(hout[2]);
    EXPECT_FALSE(hout[3]);
    EXPECT_FALSE(hout[4]);
    EXPECT_TRUE(hout[5]);
    EXPECT_TRUE(hout[6]);

    ret = hipFree(out);
    ASSERT_EQ(hipSuccess, ret);
}

TEST(Vote, Any) {
    hipError_t ret;
    int device;
    ret = hipGetDevice(&device);
    ASSERT_EQ(hipSuccess, ret);

    hipDeviceProp_t prop;
    ret = hipGetDeviceProperties(&prop, device);
    ASSERT_EQ(hipSuccess, ret);

    const int warpSize = prop.warpSize;

    bool * out;
    ret = hipMalloc((void **) &out, 7 * sizeof(*out));
    ASSERT_EQ(hipSuccess, ret);

    hipStream_t stream;

    ret = hipStreamCreate(&stream);
    ASSERT_EQ(hipSuccess, ret);

    kv_any<<<1, warpSize, 0, stream>>>(out + 0, warpSize,     warpSize);
    kv_any<<<1, warpSize, 0, stream>>>(out + 1, warpSize - 1, warpSize);
    kv_any<<<1, warpSize, 0, stream>>>(out + 2, warpSize,     warpSize - 1);
    kv_any<<<1, warpSize, 0, stream>>>(out + 3, warpSize - 1, warpSize - 1);
    kv_any<<<1, warpSize, 0, stream>>>(out + 4, warpSize - 1, warpSize - 2);
    kv_any<<<1, warpSize, 0, stream>>>(out + 5, warpSize,     0);
    kv_any<<<1, warpSize, 0, stream>>>(out + 6, warpSize - 1, 0);

    ret = hipStreamSynchronize(stream);
    EXPECT_EQ(hipSuccess, ret);

    ret = hipStreamDestroy(stream);
    ASSERT_EQ(hipSuccess, ret);

    bool hout[7];
    ret = hipMemcpy(hout, out, sizeof(hout), hipMemcpyDeviceToHost);
    ASSERT_EQ(hipSuccess, ret);

    EXPECT_TRUE(hout[0]);
    EXPECT_TRUE(hout[1]);
    EXPECT_TRUE(hout[2]);
    EXPECT_TRUE(hout[3]);
    EXPECT_TRUE(hout[4]);
    EXPECT_FALSE(hout[5]);
    EXPECT_FALSE(hout[6]);

    ret = hipFree(out);
    ASSERT_EQ(hipSuccess, ret);
}

TEST(Vote, NotAll) {
    hipError_t ret;
    int device;
    ret = hipGetDevice(&device);
    ASSERT_EQ(hipSuccess, ret);

    hipDeviceProp_t prop;
    ret = hipGetDeviceProperties(&prop, device);
    ASSERT_EQ(hipSuccess, ret);

    const int warpSize = prop.warpSize;

    bool * out;
    ret = hipMalloc((void **) &out, 7 * sizeof(*out));
    ASSERT_EQ(hipSuccess, ret);

    hipStream_t stream;

    ret = hipStreamCreate(&stream);
    ASSERT_EQ(hipSuccess, ret);

    kv_notall<<<1, warpSize, 0, stream>>>(out + 0, warpSize,     warpSize);
    kv_notall<<<1, warpSize, 0, stream>>>(out + 1, warpSize - 1, warpSize);
    kv_notall<<<1, warpSize, 0, stream>>>(out + 2, warpSize,     warpSize - 1);
    kv_notall<<<1, warpSize, 0, stream>>>(out + 3, warpSize - 1, warpSize - 1);
    kv_notall<<<1, warpSize, 0, stream>>>(out + 4, warpSize - 1, warpSize - 2);
    kv_notall<<<1, warpSize, 0, stream>>>(out + 5, warpSize,     0);
    kv_notall<<<1, warpSize, 0, stream>>>(out + 6, warpSize - 1, 0);

    ret = hipStreamSynchronize(stream);
    EXPECT_EQ(hipSuccess, ret);

    ret = hipStreamDestroy(stream);
    ASSERT_EQ(hipSuccess, ret);

    bool hout[7];
    ret = hipMemcpy(hout, out, sizeof(hout), hipMemcpyDeviceToHost);
    ASSERT_EQ(hipSuccess, ret);

    EXPECT_FALSE(hout[0]);
    EXPECT_FALSE(hout[1]);
    EXPECT_TRUE(hout[2]);
    EXPECT_FALSE(hout[3]);
    EXPECT_TRUE(hout[4]);
    EXPECT_TRUE(hout[5]);
    EXPECT_TRUE(hout[6]);

    ret = hipFree(out);
    ASSERT_EQ(hipSuccess, ret);
}

TEST(Ballot, Validity) {
    /**
     * We allocate two values but only initialize one.  Depending on our choice
     * of test, k_ballot will cause program behavior to depend on an
     * uninitialized value.
     */
    hipError_t ret;
    int device;
    ret = hipGetDevice(&device);
    ASSERT_EQ(hipSuccess, ret);

    hipDeviceProp_t prop;
    ret = hipGetDeviceProperties(&prop, device);
    ASSERT_EQ(hipSuccess, ret);

    const int warpSize = prop.warpSize;

    uint32_t * out;
    ret = hipMalloc((void **) &out, 2 * sizeof(*out));
    ASSERT_EQ(hipSuccess, ret);

    uint32_t * in;
    ret = hipMalloc((void **) &in, 2 * sizeof(*in));
    ASSERT_EQ(hipSuccess, ret);

    const uint32_t init = 1;
    ret = hipMemcpy(in, &init, sizeof(init), hipMemcpyHostToDevice);
    ASSERT_EQ(hipSuccess, ret);

    hipStream_t stream;

    ret = hipStreamCreate(&stream);
    ASSERT_EQ(hipSuccess, ret);

    k_ballot<<<1, warpSize, 0, stream>>>(out + 0, in, warpSize);
    k_ballot<<<1, warpSize, 0, stream>>>(out + 1, in, warpSize / 2);

    ret = hipStreamSynchronize(stream);
    EXPECT_EQ(hipSuccess, ret);

    ret = hipStreamDestroy(stream);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipFree(in);
    ASSERT_EQ(hipSuccess, ret);

    uint32_t hout[2];
    ret = hipMemcpy(hout, out, sizeof(hout), hipMemcpyDeviceToHost);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipFree(out);
    ASSERT_EQ(hipSuccess, ret);

    const uint32_t expected[2] = {0xFFFFFFFF * init, 0x0000FFFF * init};

    EXPECT_EQ(expected[0], hout[0]);
    EXPECT_EQ(expected[1], hout[1] & 0x0000FFFF);

    uint32_t vout[2];
    const int vret = VALGRIND_GET_VBITS(hout, vout, sizeof(hout));
    if (vret == 1) {
        EXPECT_EQ(0x0, vout[0]);
        EXPECT_EQ(0xFFFF0000, vout[1]);
    }
}

static __global__ void vote_all_const0(bool * out) {
    int _out;
    asm("{\n"
        "  .reg .pred %tmp;\n"
        "  vote.all.pred %tmp, 0;\n"
        "  selp.s32 %0, 1, 0, %tmp;\n"
        "}\n" : "=r"(_out));
    *out = _out;
}

static __global__ void vote_all_const1(bool * out) {
    int _out;
    asm("{\n"
        "  .reg .pred %tmp;\n"
        "  vote.all.pred %tmp, 1;\n"
        "  selp.s32 %0, 1, 0, %tmp;\n"
        "}\n" : "=r"(_out));
    *out = _out;
}

static __global__ void vote_none_const0(bool * out) {
    int _out;
    asm("{\n"
        "  .reg .pred %tmp;\n"
        "  vote.all.pred %tmp, !0;\n"
        "  selp.s32 %0, 1, 0, %tmp;\n"
        "}\n" : "=r"(_out));
    *out = _out;
}

static __global__ void vote_none_const1(bool * out) {
    int _out;
    asm("{\n"
        "  .reg .pred %tmp;\n"
        "  vote.all.pred %tmp, !1;\n"
        "  selp.s32 %0, 1, 0, %tmp;\n"
        "}\n" : "=r"(_out));
    *out = _out;
}

static __global__ void vote_any_const0(bool * out) {
    int _out;
    asm("{\n"
        "  .reg .pred %tmp;\n"
        "  vote.any.pred %tmp, 0;\n"
        "  selp.s32 %0, 1, 0, %tmp;\n"
        "}\n" : "=r"(_out));
    *out = _out;
}

static __global__ void vote_any_const1(bool * out) {
    int _out;
    asm("{\n"
        "  .reg .pred %tmp;\n"
        "  vote.any.pred %tmp, 1;\n"
        "  selp.s32 %0, 1, 0, %tmp;\n"
        "}\n" : "=r"(_out));
    *out = _out;
}

static __global__ void vote_notall_const0(bool * out) {
    int _out;
    asm("{\n"
        "  .reg .pred %tmp;\n"
        "  vote.any.pred %tmp, !0;\n"
        "  selp.s32 %0, 1, 0, %tmp;\n"
        "}\n" : "=r"(_out));
    *out = _out;
}

static __global__ void vote_notall_const1(bool * out) {
    int _out;
    asm("{\n"
        "  .reg .pred %tmp;\n"
        "  vote.any.pred %tmp, !1;\n"
        "  selp.s32 %0, 1, 0, %tmp;\n"
        "}\n" : "=r"(_out));
    *out = _out;
}

TEST(Vote, ConstantArguments) {
    hipError_t ret;
    int device;
    ret = hipGetDevice(&device);
    ASSERT_EQ(hipSuccess, ret);

    hipDeviceProp_t prop;
    ret = hipGetDeviceProperties(&prop, device);
    ASSERT_EQ(hipSuccess, ret);

    const int warpSize = prop.warpSize;

    bool * out;
    ret = hipMalloc((void **) &out, 8 * sizeof(*out));
    ASSERT_EQ(hipSuccess, ret);

    hipStream_t stream;

    ret = hipStreamCreate(&stream);
    ASSERT_EQ(hipSuccess, ret);

    vote_all_const0   <<<1, warpSize, 0, stream>>>(out + 0);
    vote_all_const1   <<<1, warpSize, 0, stream>>>(out + 1);
    vote_none_const0  <<<1, warpSize, 0, stream>>>(out + 2);
    vote_none_const1  <<<1, warpSize, 0, stream>>>(out + 3);
    vote_any_const0   <<<1, warpSize, 0, stream>>>(out + 4);
    vote_any_const1   <<<1, warpSize, 0, stream>>>(out + 5);
    vote_notall_const0<<<1, warpSize, 0, stream>>>(out + 6);
    vote_notall_const1<<<1, warpSize, 0, stream>>>(out + 7);

    ret = hipStreamSynchronize(stream);
    EXPECT_EQ(hipSuccess, ret);

    ret = hipStreamDestroy(stream);
    ASSERT_EQ(hipSuccess, ret);

    bool hout[8];
    ret = hipMemcpy(hout, out, sizeof(hout), hipMemcpyDeviceToHost);
    ASSERT_EQ(hipSuccess, ret);

    EXPECT_FALSE(hout[0]);
    EXPECT_TRUE (hout[1]);
    EXPECT_TRUE (hout[2]);
    EXPECT_FALSE(hout[3]);
    EXPECT_FALSE(hout[4]);
    EXPECT_TRUE (hout[5]);
    EXPECT_TRUE (hout[6]);
    EXPECT_FALSE(hout[7]);

    ret = hipFree(out);
    ASSERT_EQ(hipSuccess, ret);
}

static __global__ void vote_ballot_const0(uint32_t * out) {
    uint32_t _out;
    asm("vote.ballot.b32 %0, 0;\n" : "=r"(_out));
    *out = _out;
}

static __global__ void vote_ballot_const1(uint32_t * out) {
    uint32_t _out;
    asm("vote.ballot.b32 %0, 1;\n" : "=r"(_out));
    *out = _out;
}

static __global__ void vote_ballot_not_const0(uint32_t * out) {
    uint32_t _out;
    asm("vote.ballot.b32 %0, !0;\n" : "=r"(_out));
    *out = _out;
}

static __global__ void vote_ballot_not_const1(uint32_t * out) {
    uint32_t _out;
    asm("vote.ballot.b32 %0, !1;\n" : "=r"(_out));
    *out = _out;
}

TEST(Ballot, ConstantArguments) {
    hipError_t ret;
    int device;
    ret = hipGetDevice(&device);
    ASSERT_EQ(hipSuccess, ret);

    hipDeviceProp_t prop;
    ret = hipGetDeviceProperties(&prop, device);
    ASSERT_EQ(hipSuccess, ret);

    const int warpSize = prop.warpSize;

    uint32_t * out;
    ret = hipMalloc((void **) &out, 8 * sizeof(*out));
    ASSERT_EQ(hipSuccess, ret);

    hipStream_t stream;

    ret = hipStreamCreate(&stream);
    ASSERT_EQ(hipSuccess, ret);

    vote_ballot_const0    <<<1, warpSize,     0, stream>>>(out + 0);
    vote_ballot_const1    <<<1, warpSize,     0, stream>>>(out + 1);
    vote_ballot_not_const0<<<1, warpSize,     0, stream>>>(out + 2);
    vote_ballot_not_const1<<<1, warpSize,     0, stream>>>(out + 3);
    vote_ballot_const0    <<<1, warpSize / 2, 0, stream>>>(out + 4);
    vote_ballot_const1    <<<1, warpSize / 2, 0, stream>>>(out + 5);
    vote_ballot_not_const0<<<1, warpSize / 2, 0, stream>>>(out + 6);
    vote_ballot_not_const1<<<1, warpSize / 2, 0, stream>>>(out + 7);

    ret = hipStreamSynchronize(stream);
    EXPECT_EQ(hipSuccess, ret);

    ret = hipStreamDestroy(stream);
    ASSERT_EQ(hipSuccess, ret);

    uint32_t hout[8];
    ret = hipMemcpy(hout, out, sizeof(hout), hipMemcpyDeviceToHost);
    ASSERT_EQ(hipSuccess, ret);

    /**
     * This is a particularly unexpected outcome that is validated by
     * the control run of the test suite (e.g., without Panoptes).  For
     * immediate arguments, vote.ballot treats the operand as a true predicate
     * value.  Consequently, the result of the ballot is an indicator of which
     * threads participated.
     */
    EXPECT_EQ(0xFFFFFFFF, hout[0]);
    EXPECT_EQ(0xFFFFFFFF, hout[1]);
    EXPECT_EQ(0xFFFFFFFF, hout[2]);
    EXPECT_EQ(0xFFFFFFFF, hout[3]);
    EXPECT_EQ(0x0000FFFF, hout[4]);
    EXPECT_EQ(0x0000FFFF, hout[5]);
    EXPECT_EQ(0x0000FFFF, hout[6]);
    EXPECT_EQ(0x0000FFFF, hout[7]);

    ret = hipFree(out);
    ASSERT_EQ(hipSuccess, ret);
}

int main(int argc, char **argv) {
    ::testing::InitGoogleTest(&argc, argv);
    return RUN_ALL_TESTS();
}
