#include "hip/hip_runtime.h"
/**
 * Panoptes - A Binary Translation Framework for CUDA
 * (c) 2011-2012 Chris Kennelly <chris@ckennelly.com>
 *
 * This program is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.
 */

#include <hip/hip_runtime.h>
#include <gtest/gtest.h>

extern "C" __global__ void k_printf(int N) {
    const int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < N) {
        printf("%d\n", i);
    }
}

TEST(kPrintf, ExplicitStream) {
    hipError_t ret;

    int device;
    ret = hipGetDevice(&device);
    ASSERT_EQ(hipSuccess, ret);

    struct hipDeviceProp_t prop;
    ret = hipGetDeviceProperties(&prop, device);
    ASSERT_EQ(hipSuccess, ret);

    if (prop.major < 2) {
        /* printf is not available. */
        return;
    }

    hipStream_t stream;

    ret = hipStreamCreate(&stream);
    ASSERT_EQ(hipSuccess, ret);

    k_printf<<<1, 1, 0, stream>>>(2);

    ret = hipStreamSynchronize(stream);
    EXPECT_EQ(hipSuccess, ret);

    k_printf<<<256, 16, 0, stream>>>(2);

    ret = hipStreamSynchronize(stream);
    EXPECT_EQ(hipSuccess, ret);

    ret = hipStreamDestroy(stream);
    ASSERT_EQ(hipSuccess, ret);
}

int main(int argc, char **argv) {
    ::testing::InitGoogleTest(&argc, argv);
    return RUN_ALL_TESTS();
}
