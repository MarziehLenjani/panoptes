#include "hip/hip_runtime.h"
/**
 * Panoptes - A Binary Translation Framework for CUDA
 * (c) 2011-2012 Chris Kennelly <chris@ckennelly.com>
 *
 * This program is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.
 */

#include <boost/static_assert.hpp>
#include <boost/type_traits/make_unsigned.hpp>
#include <hip/hip_runtime.h>
#include <gtest/gtest.h>
#include <limits>
#include <stdint.h>
#include <valgrind/memcheck.h>

template<typename T>
class BitfieldTestFixture : public ::testing::Test {
public:
    BitfieldTestFixture() { }
    ~BitfieldTestFixture() { }

    void SetUp() {
        hipError_t ret;
        ret = hipMalloc((void **) &d, sizeof(*d) * n);
        ASSERT_EQ(hipSuccess, ret);

        ret = hipMalloc((void **) &a, sizeof(*a) * n);
        ASSERT_EQ(hipSuccess, ret);

        ret = hipMalloc((void **) &f, sizeof(*a) * n);
        ASSERT_EQ(hipSuccess, ret);

        ret = hipStreamCreate(&stream);
        ASSERT_EQ(hipSuccess, ret);
    }

    void TearDown() {
        hipError_t ret;
        ret = hipFree(d);
        ASSERT_EQ(hipSuccess, ret);

        ret = hipFree(a);
        ASSERT_EQ(hipSuccess, ret);

        ret = hipFree(f);
        ASSERT_EQ(hipSuccess, ret);

        ret = hipStreamSynchronize(stream);
        ASSERT_EQ(hipSuccess, ret);

        ret = hipStreamDestroy(stream);
        ASSERT_EQ(hipSuccess, ret);
    }

    static const int32_t n = 1 << 20;
    /**
     * When testing insertions, we use d as field b.
     */
    T * d;
    T * a;
    T * f;
    T tmp;
    hipStream_t stream;
};

TYPED_TEST_CASE_P(BitfieldTestFixture);

template<typename T>
__global__ void k_bfi(T * f, const T * a, const T * b, uint32_t c,
        uint32_t d, int N) {
    BOOST_STATIC_ASSERT(sizeof(T) == 0);
}

template<>
__global__ void k_bfi(uint32_t * f, const uint32_t * a, const uint32_t * b,
        uint32_t c, uint32_t d, int N) {
    for (int idx = threadIdx.x + blockDim.x * blockIdx.x;
            idx < N; idx += blockDim.x * gridDim.x) {
        uint32_t _f;
        uint32_t _a = a[idx];
        uint32_t _b = b[idx];
        asm volatile("bfi.b32 %0, %1, %2, %3, %4;\n" : "=r"(_f) : "r"(_a),
            "r"(_b), "r"(c), "r"(d));
        f[idx] = _f;
    }
}

template<>
__global__ void k_bfi(uint64_t * f, const uint64_t * a, const uint64_t * b,
        uint32_t c, uint32_t d, int N) {
    for (int idx = threadIdx.x + blockDim.x * blockIdx.x;
            idx < N; idx += blockDim.x * gridDim.x) {
        uint64_t _f;
        uint64_t _a = a[idx];
        uint64_t _b = b[idx];
        asm volatile("bfi.b64 %0, %1, %2, %3, %4;\n" : "=l"(_f) : "l"(_a),
            "l"(_b), "r"(c), "r"(d));
        f[idx] = _f;
    }
}

template<>
__global__ void k_bfi(int32_t * f, const int32_t * a, const int32_t * b,
        uint32_t c, uint32_t d, int N) {
    for (int idx = threadIdx.x + blockDim.x * blockIdx.x;
            idx < N; idx += blockDim.x * gridDim.x) {
        int32_t _f;
        int32_t _a = a[idx];
        int32_t _b = b[idx];
        asm volatile("bfi.b32 %0, %1, %2, %3, %4;\n" : "=r"(_f) : "r"(_a),
            "r"(_b), "r"(c), "r"(d));
        f[idx] = _f;
    }
}

template<>
__global__ void k_bfi(int64_t * f, const int64_t * a, const int64_t * b,
        uint32_t c, uint32_t d, int N) {
    for (int idx = threadIdx.x + blockDim.x * blockIdx.x;
            idx < N; idx += blockDim.x * gridDim.x) {
        int64_t _f;
        int64_t _a = a[idx];
        int64_t _b = b[idx];
        asm volatile("bfi.b64 %0, %1, %2, %3, %4;\n" : "=l"(_f) : "l"(_a),
            "l"(_b), "r"(c), "r"(d));
        f[idx] = _f;
    }
}

TYPED_TEST_P(BitfieldTestFixture, Insert) {
    uint32_t c = 1;
    uint32_t d = 33;
    k_bfi<<<256, 16, 0, this->stream>>>(
        this->f, this->a, this->d, c, d, this->n);
}

template<typename T>
__global__ void k_bfi_const(T * f, const T * a, const T * b, int N) {
    BOOST_STATIC_ASSERT(sizeof(T) == 0);
}

template<>
__global__ void k_bfi_const(uint32_t * f, const uint32_t * a,
        const uint32_t * b, int N) {
    for (int idx = threadIdx.x + blockDim.x * blockIdx.x;
            idx < N; idx += blockDim.x * gridDim.x) {
        uint32_t _f;
        uint32_t _a = a[idx];
        uint32_t _b = b[idx];
        asm volatile("bfi.b32 %0, %1, %2, 1, 5;\n" : "=r"(_f) :
            "r"(_a), "r"(_b));
        f[idx] = _f;
    }
}

template<>
__global__ void k_bfi_const(uint64_t * f, const uint64_t * a,
        const uint64_t * b, int N) {
    for (int idx = threadIdx.x + blockDim.x * blockIdx.x;
            idx < N; idx += blockDim.x * gridDim.x) {
        uint64_t _f;
        uint64_t _a = a[idx];
        uint64_t _b = b[idx];
        asm volatile("bfi.b64 %0, %1, %2, 1, 5;\n" : "=l"(_f) :
            "l"(_a), "l"(_b));
        f[idx] = _f;
    }
}

template<>
__global__ void k_bfi_const(int32_t * f, const int32_t * a,
        const int32_t * b, int N) {
    for (int idx = threadIdx.x + blockDim.x * blockIdx.x;
            idx < N; idx += blockDim.x * gridDim.x) {
        int32_t _f;
        int32_t _a = a[idx];
        int32_t _b = b[idx];
        asm volatile("bfi.b32 %0, %1, %2, 1, 5;\n" : "=r"(_f) :
            "r"(_a), "r"(_b));
        f[idx] = _f;
    }
}

template<>
__global__ void k_bfi_const(int64_t * f, const int64_t * a,
        const int64_t * b, int N) {
    for (int idx = threadIdx.x + blockDim.x * blockIdx.x;
            idx < N; idx += blockDim.x * gridDim.x) {
        int64_t _f;
        int64_t _a = a[idx];
        int64_t _b = b[idx];
        asm volatile("bfi.b64 %0, %1, %2, 1, 5;\n" : "=l"(_f) :
            "l"(_a), "l"(_b));
        f[idx] = _f;
    }
}

TYPED_TEST_P(BitfieldTestFixture, InsertConstant) {
    k_bfi_const<<<256, 16, 0, this->stream>>>(
        this->f, this->a, this->d, this->n);
}

template<typename T>
__global__ void k_bfi_constd(T * f, const T * a, const T * b,
        uint32_t c, int N) {
    BOOST_STATIC_ASSERT(sizeof(T) == 0);
}

template<>
__global__ void k_bfi_constd(uint32_t * f, const uint32_t * a,
        const uint32_t * b, uint32_t c, int N) {
    for (int idx = threadIdx.x + blockDim.x * blockIdx.x;
            idx < N; idx += blockDim.x * gridDim.x) {
        uint32_t _f;
        uint32_t _a = a[idx];
        uint32_t _b = b[idx];
        asm volatile("bfi.b32 %0, %1, %2, %3, 5;\n" : "=r"(_f) : "r"(_a),
            "r"(_b), "r"(c));
        f[idx] = _f;
    }
}

template<>
__global__ void k_bfi_constd(uint64_t * f, const uint64_t * a,
        const uint64_t * b, uint32_t c, int N) {
    for (int idx = threadIdx.x + blockDim.x * blockIdx.x;
            idx < N; idx += blockDim.x * gridDim.x) {
        uint64_t _f;
        uint64_t _a = a[idx];
        uint64_t _b = b[idx];
        asm volatile("bfi.b64 %0, %1, %2, %3, 5;\n" : "=l"(_f) : "l"(_a),
            "l"(_b), "r"(c));
        f[idx] = _f;
    }
}

template<>
__global__ void k_bfi_constd(int32_t * f, const int32_t * a,
        const int32_t * b, uint32_t c, int N) {
    for (int idx = threadIdx.x + blockDim.x * blockIdx.x;
            idx < N; idx += blockDim.x * gridDim.x) {
        int32_t _f;
        int32_t _a = a[idx];
        int32_t _b = b[idx];
        asm volatile("bfi.b32 %0, %1, %2, %3, 5;\n" : "=r"(_f) : "r"(_a),
            "r"(_b), "r"(c));
        f[idx] = _f;
    }
}

template<>
__global__ void k_bfi_constd(int64_t * f, const int64_t * a,
        const int64_t * b, uint32_t c, int N) {
    for (int idx = threadIdx.x + blockDim.x * blockIdx.x;
            idx < N; idx += blockDim.x * gridDim.x) {
        int64_t _f;
        int64_t _a = a[idx];
        int64_t _b = b[idx];
        asm volatile("bfi.b64 %0, %1, %2, %3, 5;\n" : "=l"(_f) : "l"(_a),
            "l"(_b), "r"(c));
        f[idx] = _f;
    }
}

TYPED_TEST_P(BitfieldTestFixture, InsertConstantD) {
    const uint32_t c = 5;
    k_bfi_constd<<<256, 16, 0, this->stream>>>(this->f, this->a, this->d, c,
        this->n);
}

REGISTER_TYPED_TEST_CASE_P(BitfieldTestFixture,
    Insert,  InsertConstant,  InsertConstantD);

typedef ::testing::Types<int32_t, uint32_t, int64_t, uint64_t> MyTypes;
INSTANTIATE_TYPED_TEST_CASE_P(My, BitfieldTestFixture, MyTypes);

template<typename T>
class BitfieldSingle : public ::testing::Test {
public:
    BitfieldSingle() { }
    ~BitfieldSingle() { }

    void SetUp() {
        hipError_t ret;
        ret = hipStreamCreate(&stream);
        ASSERT_EQ(hipSuccess, ret);
    }

    void TearDown() {
        hipError_t ret;
        ret = hipStreamDestroy(stream);
        ASSERT_EQ(hipSuccess, ret);
    }

    hipStream_t stream;
};

TYPED_TEST_CASE_P(BitfieldSingle);

template<typename T>
__global__ void k_bfi_constant_data(T * f, uint32_t c, uint32_t d) {
    BOOST_STATIC_ASSERT(sizeof(T) == 0);
}

template<>
__global__ void k_bfi_constant_data(uint32_t * f, uint32_t c, uint32_t d) {
    uint32_t _out;
    asm("bfi.b32 %0, 2863311530, 1431655765, %1, %2;" : "=r"(_out) :
        "r"(c), "r"(d));
    *f = _out;
}

template<>
__global__ void k_bfi_constant_data(int32_t * f, uint32_t c, uint32_t d) {
    int32_t _out;
    asm("bfi.b32 %0, 2863311530, 1431655765, %1, %2;" : "=r"(_out) :
        "r"(c), "r"(d));
    *f = _out;
}

template<>
__global__ void k_bfi_constant_data(uint64_t * f, uint32_t c, uint32_t d) {
    uint64_t _out;
    asm("bfi.b64 %0, 3074457345618258602, 6148914691236517205, %1, %2;" :
        "=l"(_out) : "r"(c), "r"(d));
    *f = _out;
}

template<>
__global__ void k_bfi_constant_data(int64_t * f, uint32_t c, uint32_t d) {
    int64_t _out;
    asm("bfi.b64 %0, 3074457345618258602, 6148914691236517205, %1, %2;" :
        "=l"(_out) : "r"(c), "r"(d));
    *f = _out;
}

template<typename T>
__global__ void k_bfi_constant(T * f) {
    BOOST_STATIC_ASSERT(sizeof(T) == 0);
}

template<>
__global__ void k_bfi_constant(uint32_t * f) {
    uint32_t _out;
    asm("bfi.b32 %0, 2863311530, 1431655765, 5, 5;" : "=r"(_out));
    *f = _out;
}

template<>
__global__ void k_bfi_constant(int32_t * f) {
    int32_t _out;
    asm("bfi.b32 %0, 2863311530, 1431655765, 5, 5;" : "=r"(_out));
    *f = _out;
}

template<>
__global__ void k_bfi_constant(uint64_t * f) {
    uint64_t _out;
    asm("bfi.b64 %0, 3074457345618258602, 6148914691236517205, 5, 5;" :
        "=l"(_out));
    *f = _out;
}

template<>
__global__ void k_bfi_constant(int64_t * f) {
    int64_t _out;
    asm("bfi.b64 %0, 3074457345618258602, 6148914691236517205, 5, 5;" :
        "=l"(_out));
    *f = _out;
}

TYPED_TEST_P(BitfieldSingle, InsertConstantData) {
    TypeParam * f;

    hipError_t ret;
    ret = hipMalloc((void **) &f, 5 * sizeof(*f));
    assert(hipSuccess == ret);

    const uint32_t c = 5;
    const uint32_t d = 5;

    uint32_t c_invalid = c;
    VALGRIND_MAKE_MEM_UNDEFINED(&c_invalid, sizeof(c_invalid));
    uint32_t d_invalid = d;
    VALGRIND_MAKE_MEM_UNDEFINED(&d_invalid, sizeof(d_invalid));

    k_bfi_constant_data<<<1, 1, 0, this->stream>>>(f + 0, c,         d);
    k_bfi_constant     <<<1, 1, 0, this->stream>>>(f + 1);
    k_bfi_constant_data<<<1, 1, 0, this->stream>>>(f + 2, c,         d_invalid);
    k_bfi_constant_data<<<1, 1, 0, this->stream>>>(f + 3, c_invalid, d);
    k_bfi_constant_data<<<1, 1, 0, this->stream>>>(f + 4, c_invalid, d_invalid);

    ret = hipStreamSynchronize(this->stream);
    assert(hipSuccess == ret);

    TypeParam hf[5];
    ret = hipMemcpy(hf, f, sizeof(hf), hipMemcpyDeviceToHost);

    const TypeParam expected = (sizeof(TypeParam) == 4) ?
         1431655765 : 6148914691236517205;

    assert(expected == hf[0]);
    assert(expected == hf[1]);

    uint32_t vf[10];
    const int vret = VALGRIND_GET_VBITS(hf, vf, sizeof(hf));
    BOOST_STATIC_ASSERT(sizeof(hf) <= sizeof(vf));
    if (vret == 1) {
        assert(0 == vf[0]);
        assert(0 == vf[1]);

        if (sizeof(TypeParam) == 4) {
            assert(0xFFFFFFFF == vf[2]);
            assert(0xFFFFFFFF == vf[3]);
        } else {
            assert(0 == vf[2]);
            assert(0 == vf[3]);
        }

        for (size_t i = 4; i < sizeof(TypeParam); i++) {
            assert(0xFFFFFFFF == vf[i]);
        }
    }

    ret = hipFree(f);
    assert(hipSuccess == ret);
}

REGISTER_TYPED_TEST_CASE_P(BitfieldSingle, InsertConstantData);
INSTANTIATE_TYPED_TEST_CASE_P(My, BitfieldSingle, MyTypes);

int main(int argc, char **argv) {
    ::testing::InitGoogleTest(&argc, argv);
    return RUN_ALL_TESTS();
}
