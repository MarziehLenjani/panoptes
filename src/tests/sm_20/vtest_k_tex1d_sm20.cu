#include "hip/hip_runtime.h"
/**
 * Panoptes - A Binary Translation Framework for CUDA
 * (c) 2011-2012 Chris Kennelly <chris@ckennelly.com>
 *
 * This program is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.
 */

#include <boost/static_assert.hpp>
#include <gtest/gtest.h>
#include <stdint.h>
#include <valgrind/memcheck.h>

typedef int32_t tex_t;
texture<tex_t, 1, hipReadModeElementType> tex_src;

class TextureValues : public ::testing::TestWithParam<int> {
    // Empty Fixture
};

static __global__ void k_set(tex_t * out, int n) {
    for (int i = threadIdx.x + blockIdx.x * blockDim.x; i < n;
            i += blockDim.x * gridDim.x) {
        out[i] = static_cast<tex_t>(i);
    }
}

static __global__ void k_copy(tex_t * out, int n) {
    for (int i = threadIdx.x + blockIdx.x * blockDim.x; i < n;
            i += blockDim.x * gridDim.x) {
        out[i] = tex1Dfetch(tex_src, i);
    }
}

TEST_P(TextureValues, DataCopy) {
    /**
     * Verify we can read the values from a texture.
     */
    const int param = GetParam();
    const int alloc = 1 << param;

    const int n_threads = 256;
    const int n_blocks  = (alloc + n_threads - 1) / n_threads;

    hipError_t ret;
    int device;
    ret = hipGetDevice(&device);
    ASSERT_EQ(hipSuccess, ret);

    struct hipDeviceProp_t prop;
    ret = hipGetDeviceProperties(&prop, device);
    ASSERT_EQ(hipSuccess, ret);

    if (alloc > prop.maxTexture1DLinear) {
        return;
    }

    hipStream_t stream;
    ret = hipStreamCreate(&stream);
    ASSERT_EQ(hipSuccess, ret);

    tex_t *tex;
    ret = hipMalloc((void **) &tex, 2 * sizeof(*tex) * alloc);
    ASSERT_EQ(hipSuccess, ret);

    /* Only half of the values are initialized. */
    k_set<<<n_blocks, n_threads, 0, stream>>>(tex, alloc);
    ret = hipStreamSynchronize(stream);
    ASSERT_EQ(hipSuccess, ret);

    const struct hipChannelFormatDesc desc = hipCreateChannelDesc<tex_t>();
    tex_src.addressMode[0] = hipAddressModeClamp;
    tex_src.filterMode = hipFilterModePoint;
    tex_src.normalized = false;

    ret = hipBindTexture(NULL, tex_src, tex, desc, 2 * sizeof(*tex) * alloc);
    ASSERT_EQ(hipSuccess, ret);

    /* Allocate output. */
    tex_t *out;
    ret = hipMalloc((void **) &out, 2 * sizeof(*out) * alloc);
    ASSERT_EQ(hipSuccess, ret);

    /* Run kernel. */
    k_copy<<<n_blocks, n_threads, 0, stream>>>(out, 2 * alloc);

    ret = hipStreamSynchronize(stream);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipStreamDestroy(stream);
    ASSERT_EQ(hipSuccess, ret);

    std::vector<tex_t> hout(2 * alloc);
    ret = hipMemcpy(&hout[0], out, 2 * sizeof(*out) * alloc,
        hipMemcpyDeviceToHost);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipFree(out);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipUnbindTexture(tex_src);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipFree(tex);
    ASSERT_EQ(hipSuccess, ret);

    if (RUNNING_ON_VALGRIND) {
        /* Check validity bits. */
        std::vector<uint32_t> vout(2 * alloc);
        BOOST_STATIC_ASSERT(sizeof(vout[0]) == sizeof(hout[0]));
        int vret = VALGRIND_GET_VBITS(&hout[0], &vout[0],
            2 * sizeof(hout[0]) * alloc);
        ASSERT_EQ(1, vret);

        bool error = false;
        int i;
        /* First half is fully initialized. */
        for (i = 0; i < alloc; i++) {
            error |= vout[i] != 0;
        }
        EXPECT_FALSE(error);

        /* Second half is uninitialized. */
        for (; i < 2 * alloc; i++) {
            error |= vout[i] != 0xFFFFFFFF;
        }
        EXPECT_FALSE(error);
    }

    bool error = false;
    for (int i = 0; i < alloc; i++) {
        const tex_t expected = static_cast<tex_t>(i);
        error |= expected != hout[i];
    }
    EXPECT_FALSE(error);
}

INSTANTIATE_TEST_CASE_P(TextureInst, TextureValues, ::testing::Range(1, 22));

static __global__ void k_load(tex_t * out, int index) {
    *out = tex1Dfetch(tex_src, index);
}

TEST(Textures, WildLoad) {
    /**
     * Allocate a buffer and initialize it.  Then perform a read from the
     * texture at an uninitialized index.
     */
    if (!(RUNNING_ON_VALGRIND)) {
        return;
    }

    hipError_t ret;
    hipStream_t stream;
    ret = hipStreamCreate(&stream);
    ASSERT_EQ(hipSuccess, ret);

    const size_t n_elements = 1u << 16;
    const size_t n_threads  = 256;
    const size_t n_blocks   = (n_elements + n_threads - 1) / n_threads;

    tex_t *tex;
    ret = hipMalloc((void **) &tex, sizeof(*tex) * n_elements);
    ASSERT_EQ(hipSuccess, ret);

    k_set<<<n_blocks, n_threads, 0, stream>>>(tex, n_elements);
    ret = hipStreamSynchronize(stream);
    ASSERT_EQ(hipSuccess, ret);

    const struct hipChannelFormatDesc desc = hipCreateChannelDesc<tex_t>();
    tex_src.addressMode[0] = hipAddressModeClamp;
    tex_src.filterMode = hipFilterModePoint;
    tex_src.normalized = false;

    ret = hipBindTexture(NULL, tex_src, tex, desc, sizeof(*tex) * n_elements);
    ASSERT_EQ(hipSuccess, ret);

    /* Allocate output. */
    tex_t *out;
    ret = hipMalloc((void **) &out, sizeof(*out));
    ASSERT_EQ(hipSuccess, ret);

    /* Run kernel. */
    int index = 0;
    VALGRIND_MAKE_MEM_UNDEFINED(&index, sizeof(index));
    k_load<<<1, 1, 0, stream>>>(out, index);

    ret = hipStreamSynchronize(stream);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipStreamDestroy(stream);
    ASSERT_EQ(hipSuccess, ret);

    tex_t hout;
    ret = hipMemcpy(&hout, out, sizeof(*out), hipMemcpyDeviceToHost);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipFree(out);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipUnbindTexture(tex_src);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipFree(tex);
    ASSERT_EQ(hipSuccess, ret);

    uint32_t vout;
    BOOST_STATIC_ASSERT(sizeof(vout) == sizeof(hout));
    int vret = VALGRIND_GET_VBITS(&hout, &vout, sizeof(hout));
    EXPECT_EQ(1, vret);
    EXPECT_EQ(0xFFFFFFFF, vout);
}

static __global__ void k_txq(unsigned * out) {
    unsigned tmp;
    asm("txq.width.b32 %0, [tex_src];" : "=r"(tmp));
    out[0] = tmp;

    asm("txq.height.b32 %0, [tex_src];" : "=r"(tmp));
    out[1] = tmp;

    asm("txq.depth.b32 %0, [tex_src];" : "=r"(tmp));
    out[2] = tmp;

    asm("txq.channel_data_type.b32 %0, [tex_src];" : "=r"(tmp));
    out[3] = tmp;

    asm("txq.channel_order.b32 %0, [tex_src];" : "=r"(tmp));
    out[4] = tmp;

    asm("txq.normalized_coords.b32 %0, [tex_src];" : "=r"(tmp));
    out[5] = tmp;

    asm("txq.filter_mode.b32 %0, [tex_src];" : "=r"(tmp));
    out[6] = tmp;

    asm("txq.addr_mode_0.b32 %0, [tex_src];" : "=r"(tmp));
    out[7] = tmp;

    asm("txq.addr_mode_1.b32 %0, [tex_src];" : "=r"(tmp));
    out[8] = tmp;

    asm("txq.addr_mode_2.b32 %0, [tex_src];" : "=r"(tmp));
    out[9] = tmp;
}

TEST(Textures, Query) {
    hipError_t ret;
    hipStream_t stream;
    ret = hipStreamCreate(&stream);
    ASSERT_EQ(hipSuccess, ret);

    const size_t n_elements = 1 << 16;

    tex_t *tex;
    ret = hipMalloc((void **) &tex, sizeof(*tex) * n_elements);
    ASSERT_EQ(hipSuccess, ret);

    const struct hipChannelFormatDesc desc = hipCreateChannelDesc<tex_t>();
    tex_src.addressMode[0] = hipAddressModeClamp;
    tex_src.filterMode = hipFilterModePoint;
    tex_src.normalized = false;

    ret = hipBindTexture(NULL, tex_src, tex, desc, sizeof(*tex) * n_elements);
    ASSERT_EQ(hipSuccess, ret);

    /* Allocate output. */
    unsigned hout[10];
    unsigned *out;
    ret = hipMalloc((void **) &out, sizeof(hout));
    ASSERT_EQ(hipSuccess, ret);

    ret = hipMemset(out, 0xFF, sizeof(hout));
    ASSERT_EQ(hipSuccess, ret);

    /* Run kernel. */
    k_txq<<<1, 1, 0, stream>>>(out);

    ret = hipStreamSynchronize(stream);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipStreamDestroy(stream);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipMemcpy(hout, out, sizeof(hout), hipMemcpyDeviceToHost);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipFree(out);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipUnbindTexture(tex_src);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipFree(tex);
    ASSERT_EQ(hipSuccess, ret);

    /*
     * The values returned by txq return 0.
     *
     * 0: width
     * 1: height
     * 2: depth
     * 3: channel_data_type
     * 4: channel_order
     * 5: normalized_coords
     * 6: filter_mode
     * 7: addr_mode_0
     * 8: addr_mode_1
     * 9: addr_mode_2
     */
    EXPECT_EQ(0, hout[0]);
    EXPECT_EQ(0, hout[1]);
    EXPECT_EQ(0, hout[2]);
    EXPECT_EQ(0, hout[3]);
    EXPECT_EQ(0, hout[4]);
    EXPECT_EQ(0, hout[5]);
    EXPECT_EQ(0, hout[6]);
    EXPECT_EQ(0, hout[7]);
    EXPECT_EQ(0, hout[8]);
    EXPECT_EQ(0, hout[9]);
}

int main(int argc, char **argv) {
    ::testing::InitGoogleTest(&argc, argv);
    return RUN_ALL_TESTS();
}
