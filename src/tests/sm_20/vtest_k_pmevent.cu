#include "hip/hip_runtime.h"
/**
 * Panoptes - A Binary Translation Framework for CUDA
 * (c) 2011-2012 Chris Kennelly <chris@ckennelly.com>
 *
 * This program is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.
 */

#include <hip/hip_runtime.h>
#include <gtest/gtest.h>

extern "C" __global__ void k_pmevent() {
    prof_trigger(0);
}

extern "C" __global__ void k_pmmask() {
    /* This must be an immediate */
    asm volatile("pmevent.mask 3;");
}

TEST(kPMEVENT, ExplicitStream) {
    hipError_t ret;
    hipStream_t stream;

    ret = hipStreamCreate(&stream);
    ASSERT_EQ(hipSuccess, ret);

    k_pmevent<<<256, 16, 0, stream>>>();

    ret = hipStreamSynchronize(stream);
    EXPECT_EQ(hipSuccess, ret);

    ret = hipStreamDestroy(stream);
    ASSERT_EQ(hipSuccess, ret);
}

TEST(kPMEVENT, Mask) {
    hipError_t ret;
    hipStream_t stream;

    ret = hipStreamCreate(&stream);
    ASSERT_EQ(hipSuccess, ret);

    k_pmmask<<<256, 16, 0, stream>>>();

    ret = hipStreamSynchronize(stream);
    EXPECT_EQ(hipSuccess, ret);

    ret = hipStreamDestroy(stream);
    ASSERT_EQ(hipSuccess, ret);
}

int main(int argc, char **argv) {
    ::testing::InitGoogleTest(&argc, argv);
    return RUN_ALL_TESTS();
}
