/**
 * Panoptes - A Binary Translation Framework for CUDA
 * (c) 2011-2012 Chris Kennelly <chris@ckennelly.com>
 *
 * This program is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.
 */

#include <boost/scoped_array.hpp>
#include <boost/static_assert.hpp>
#include <hip/hip_runtime.h>
#include <gtest/gtest.h>
#include <stdint.h>
#include <valgrind/memcheck.h>
#include <cstdio>

TEST(EventCreate, NullArgument) {
    hipError_t ret;

    ret = hipEventCreate(NULL);
    EXPECT_EQ(hipErrorInvalidValue, ret);
}

TEST(EventCreate, CreateDestroy) {
    hipError_t ret;
    hipEvent_t event;

    ret = hipEventCreate(&event);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipEventDestroy(event);
    EXPECT_EQ(hipSuccess, ret);
}

TEST(EventCreate, EventLeak) {
    hipError_t ret;
    hipEvent_t event;

    ret = hipEventCreate(&event);
    ASSERT_EQ(hipSuccess, ret);
}

int main(int argc, char **argv) {
    ::testing::InitGoogleTest(&argc, argv);
    return RUN_ALL_TESTS();
}
