#include "hip/hip_runtime.h"
/**
 * Panoptes - A Binary Translation Framework for CUDA
 * (c) 2011-2012 Chris Kennelly <chris@ckennelly.com>
 *
 * This program is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.
 */

#include <boost/scoped_array.hpp>
#include <boost/static_assert.hpp>
#include <hip/hip_runtime.h>
#include <gtest/gtest.h>
#include <stdint.h>
#include <valgrind/memcheck.h>
#include <cstdio>

extern "C" __global__ void k_set_const_int(int * a) {
    *a = 0;
}

extern "C" __global__ void k_set_const_int_fixed_offset(int * a) {
    a[1] = 0;
}

TEST(kSetConstInt, ExplicitStream) {
    hipError_t ret;
    hipStream_t stream;
    int * a;

    ret = hipMalloc(&a, 2u * sizeof(*a));
    ASSERT_EQ(hipSuccess, ret);

    ret = hipStreamCreate(&stream);
    ASSERT_EQ(hipSuccess, ret);

    k_set_const_int<<<1, 1, 0, stream>>>(a);
    k_set_const_int_fixed_offset<<<1, 1, 0, stream>>>(a);

    ret = hipStreamSynchronize(stream);
    EXPECT_EQ(hipSuccess, ret);

    ret = hipStreamDestroy(stream);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipFree(a);
    ASSERT_EQ(hipSuccess, ret);
}

int main(int argc, char **argv) {
    ::testing::InitGoogleTest(&argc, argv);
    return RUN_ALL_TESTS();
}
