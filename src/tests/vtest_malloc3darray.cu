/**
 * Panoptes - A Binary Translation Framework for CUDA
 * (c) 2011-2012 Chris Kennelly <chris@ckennelly.com>
 *
 * This program is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.
 */

#include <hip/hip_runtime.h>
#include <gtest/gtest.h>

TEST(Malloc3DArray, NullArguments) {
    struct hipArray * ary;
    struct hipChannelFormatDesc dsc;
    dsc.x = dsc.y = dsc.z = dsc.w = 8;
    dsc.f = hipChannelFormatKindSigned;

    // Commented out cases segfault.

    EXPECT_EQ(hipErrorInvalidValue, hipMalloc3DArray(NULL, NULL, make_hipExtent(0, 0, 0), 0));
    EXPECT_EQ(hipErrorInvalidValue, hipMalloc3DArray(NULL, NULL, make_hipExtent(0, 0, 8), 0));
    EXPECT_EQ(hipErrorInvalidValue, hipMalloc3DArray(NULL, NULL, make_hipExtent(0, 8, 0), 0));
    EXPECT_EQ(hipErrorInvalidValue, hipMalloc3DArray(NULL, NULL, make_hipExtent(0, 8, 8), 0));
    // EXPECT_EQ(hipErrorInvalidValue, hipMalloc3DArray(NULL, NULL, make_hipExtent(8, 0, 0), 0));
    EXPECT_EQ(hipErrorInvalidValue, hipMalloc3DArray(NULL, NULL, make_hipExtent(8, 0, 8), 0));
    // EXPECT_EQ(hipErrorInvalidValue, hipMalloc3DArray(NULL, NULL, make_hipExtent(8, 8, 0), 0));
    // EXPECT_EQ(hipErrorInvalidValue, hipMalloc3DArray(NULL, NULL, make_hipExtent(8, 8, 8), 0));

    EXPECT_EQ(hipSuccess,           hipMalloc3DArray(&ary, NULL, make_hipExtent(0, 0, 0), 0));
    EXPECT_EQ(hipSuccess,           hipFreeArray(ary));
    EXPECT_EQ(hipSuccess,           hipMalloc3DArray(&ary, NULL, make_hipExtent(0, 0, 8), 0));
    EXPECT_EQ(hipSuccess,           hipFreeArray(ary));

    EXPECT_EQ(hipSuccess,           hipMalloc3DArray(&ary, NULL, make_hipExtent(0, 8, 0), 0));
    EXPECT_EQ(hipSuccess,           hipFreeArray(ary));

    EXPECT_EQ(hipSuccess,           hipMalloc3DArray(&ary, NULL, make_hipExtent(0, 8, 8), 0));
    EXPECT_EQ(hipSuccess,           hipFreeArray(ary));

    // EXPECT_EQ(hipErrorInvalidValue, hipMalloc3DArray(&ary, NULL, make_hipExtent(8, 0, 0), 0));

    /**
     * There's no reason why this should pass...

    EXPECT_EQ(hipSuccess,           hipMalloc3DArray(&ary, NULL, make_hipExtent(8, 0, 8), 0));
    EXPECT_EQ(hipSuccess,           hipFreeArray(ary));
     */

    // EXPECT_EQ(hipErrorInvalidValue, hipMalloc3DArray(&ary, NULL, make_hipExtent(8, 8, 0), 0));
    // EXPECT_EQ(hipErrorInvalidValue, hipMalloc3DArray(&ary, NULL, make_hipExtent(8, 8, 8), 0));

    EXPECT_EQ(hipErrorInvalidValue, hipMalloc3DArray(NULL, &dsc, make_hipExtent(0, 0, 0), 0));
    EXPECT_EQ(hipErrorInvalidValue, hipMalloc3DArray(NULL, &dsc, make_hipExtent(0, 0, 8), 0));
    EXPECT_EQ(hipErrorInvalidValue, hipMalloc3DArray(NULL, &dsc, make_hipExtent(0, 8, 0), 0));
    EXPECT_EQ(hipErrorInvalidValue, hipMalloc3DArray(NULL, &dsc, make_hipExtent(0, 8, 8), 0));
    // EXPECT_EQ(hipErrorInvalidValue, hipMalloc3DArray(NULL, &dsc, make_hipExtent(8, 0, 0), 0));
    EXPECT_EQ(hipErrorInvalidValue, hipMalloc3DArray(NULL, &dsc, make_hipExtent(8, 0, 8), 0));
    // EXPECT_EQ(hipErrorInvalidValue, hipMalloc3DArray(NULL, &dsc, make_hipExtent(8, 8, 0), 0));
    // EXPECT_EQ(hipErrorInvalidValue, hipMalloc3DArray(NULL, &dsc, make_hipExtent(8, 8, 8), 0));
}

TEST(Malloc3DArray, Limits) {
    struct hipArray * ary;
    struct hipChannelFormatDesc dsc;
    dsc.x = dsc.y = dsc.z = dsc.w = 8;
    dsc.f = hipChannelFormatKindSigned;

    hipError_t ret;

    ret = hipMalloc3DArray(&ary, &dsc,
        make_hipExtent(0,         0,          0),        0);
    EXPECT_EQ(hipSuccess, ret);
    if (ret == hipSuccess) {
        EXPECT_EQ(hipSuccess, hipFreeArray(ary));
    }

    int device;
    ret = hipGetDevice(&device);
    ASSERT_EQ(hipSuccess, ret);

    struct hipDeviceProp_t prop;
    ret = hipGetDeviceProperties(&prop, device);
    ASSERT_EQ(hipSuccess, ret);

    /* Adapt to what's available by a safe margin */
    size_t targetable = prop.totalGlobalMem / 8;

    if ((size_t) prop.maxTexture1D < targetable) {
        ret = hipMalloc3DArray(&ary, &dsc,
            make_hipExtent(prop.maxTexture1D, 0, 0), 0);
        EXPECT_EQ(hipSuccess, ret);
        if (ret == hipSuccess) {
            EXPECT_EQ(hipSuccess, hipFreeArray(ary));
        }

        ret = hipMalloc3DArray(&ary, &dsc,
            make_hipExtent(prop.maxTexture1D + 1, 0, 0), 0);
        EXPECT_EQ(hipErrorInvalidValue, ret);
        if (ret == hipSuccess) {
            EXPECT_EQ(hipSuccess, hipFreeArray(ary));
        }
    }

    if ((size_t) prop.maxTexture2D[0] < targetable) {
        ret = hipMalloc3DArray(&ary, &dsc,
            make_hipExtent(prop.maxTexture2D[0],     1, 0), 0);
        EXPECT_EQ(hipSuccess, ret);
        if (ret == hipSuccess) {
            EXPECT_EQ(hipSuccess, hipFreeArray(ary));
        }

        ret = hipMalloc3DArray(&ary, &dsc,
            make_hipExtent(prop.maxTexture2D[0] + 1, 1, 0), 0);
        EXPECT_EQ(hipErrorInvalidValue, ret);
        if (ret == hipSuccess) {
            EXPECT_EQ(hipSuccess, hipFreeArray(ary));
        }
    }

    if ((size_t) prop.maxTexture2D[1] < targetable) {
        ret = hipMalloc3DArray(&ary, &dsc,
            make_hipExtent(1, prop.maxTexture2D[1],     0), 0);
        EXPECT_EQ(hipSuccess, ret);
        if (ret == hipSuccess) {
            EXPECT_EQ(hipSuccess, hipFreeArray(ary));
        }

        ret = hipMalloc3DArray(&ary, &dsc,
            make_hipExtent(1, prop.maxTexture2D[1] + 1, 0), 0);
        EXPECT_EQ(hipErrorInvalidValue, ret);
        if (ret == hipSuccess) {
            EXPECT_EQ(hipSuccess, hipFreeArray(ary));
        }
    }

    if ((size_t) prop.maxTexture2D[0] * prop.maxTexture2D[1] < targetable) {
        ret = hipMalloc3DArray(&ary, &dsc,
            make_hipExtent(prop.maxTexture2D[0],
                prop.maxTexture2D[1], 0), 0);
        EXPECT_EQ(hipSuccess, ret);
        if (ret == hipSuccess) {
            EXPECT_EQ(hipSuccess, hipFreeArray(ary));
        }

        ret = hipMalloc3DArray(&ary, &dsc,
            make_hipExtent(prop.maxTexture2D[0],
                prop.maxTexture2D[1] + 1, 0), 0);
        EXPECT_EQ(hipErrorInvalidValue, ret);
        if (ret == hipSuccess) {
            EXPECT_EQ(hipSuccess, hipFreeArray(ary));
        }

        ret = hipMalloc3DArray(&ary, &dsc,
            make_hipExtent(prop.maxTexture2D[0] + 1,
                prop.maxTexture2D[1], 0), 0);
        EXPECT_EQ(hipErrorInvalidValue, ret);
        if (ret == hipSuccess) {
            EXPECT_EQ(hipSuccess, hipFreeArray(ary));
        }

        ret = hipMalloc3DArray(&ary, &dsc,
            make_hipExtent(prop.maxTexture2D[0] + 1,
                prop.maxTexture2D[1] + 1, 0), 0);
        EXPECT_EQ(hipErrorInvalidValue, ret);
        if (ret == hipSuccess) {
            EXPECT_EQ(hipSuccess, hipFreeArray(ary));
        }
    } else if ((size_t) prop.maxTexture2D[0] * prop.maxTexture2D[1] >
            prop.totalGlobalMem) {
        EXPECT_EQ(hipErrorOutOfMemory,
            hipMalloc3DArray(&ary, &dsc,
            make_hipExtent(prop.maxTexture2D[0], prop.maxTexture2D[1], 0), 0));
    }

   ret = hipMalloc3DArray(&ary, &dsc,
        make_hipExtent(1,         1,         1),        0);
    EXPECT_EQ(hipSuccess, ret);
    if (ret == hipSuccess) {
        EXPECT_EQ(hipSuccess, hipFreeArray(ary));
    }

    ret = hipMalloc3DArray(&ary, &dsc,
        make_hipExtent(64,        64,        64),     0);
    EXPECT_EQ(hipSuccess, ret);
    if (ret == hipSuccess) {
        EXPECT_EQ(hipSuccess, hipFreeArray(ary));
    }

    /* TODO:  More 3D tests. */
}

TEST(Malloc3DArray, Attributes) {
    struct hipArray * ary;
    struct hipChannelFormatDesc dsc;
    dsc.x = dsc.y = dsc.z = dsc.w = 8;
    dsc.f = hipChannelFormatKindSigned;

    hipError_t ret;

    ret = hipMalloc3DArray(&ary, &dsc, make_hipExtent(1, 1, 1), 0);
    ASSERT_EQ(hipSuccess, ret);

    struct hipPointerAttribute_t attr;
    ret = hipPointerGetAttributes(&attr, ary);
    EXPECT_EQ(hipErrorInvalidValue, ret);

    EXPECT_EQ(hipSuccess, hipFreeArray(ary));
}

TEST(Malloc3DArray, NegativeChannels) {
    struct hipArray * ary;
    struct hipChannelFormatDesc dsc;
    dsc.x = dsc.y = dsc.z = 8;
    dsc.w = -8;
    dsc.f = hipChannelFormatKindSigned;

    hipError_t ret;

    ret = hipMalloc3DArray(&ary, &dsc, make_hipExtent(1, 1, 1), 0);
    EXPECT_EQ(hipErrorInvalidChannelDescriptor, ret);
}

int main(int argc, char **argv) {
    ::testing::InitGoogleTest(&argc, argv);
    return RUN_ALL_TESTS();
}
