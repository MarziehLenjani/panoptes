/**
 * Panoptes - A Binary Translation Framework for CUDA
 * (c) 2011-2012 Chris Kennelly <chris@ckennelly.com>
 *
 * This program is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.
 */

#include <boost/scoped_array.hpp>
#include <boost/static_assert.hpp>
#include <hip/hip_runtime.h>
#include <gtest/gtest.h>
#include <stdint.h>
#include <valgrind/memcheck.h>
#include <cstdio>

typedef int symbol_t;
__constant__ symbol_t const_symbol;
__device__   symbol_t device_symbol;

TEST(GetSymbolSize, Invalid) {
    const char missing[] = "this_symbol_does_not_exist";
    hipError_t ret;
    size_t size;

    ret = hipGetSymbolSize(NULL, HIP_SYMBOL(missing));
    EXPECT_EQ(hipErrorInvalidValue, ret);

    ret = hipGetSymbolSize(&size, HIP_SYMBOL(missing));
    EXPECT_EQ(hipErrorInvalidSymbol, ret);

    ret = hipGetSymbolSize(NULL, HIP_SYMBOL(NULL));
    EXPECT_EQ(hipErrorInvalidValue, ret);

    ret = hipGetSymbolSize(&size, HIP_SYMBOL(NULL));
    EXPECT_EQ(hipErrorInvalidSymbol, ret);
}

TEST(GetSymbolSize, ConstantSymbol) {
    hipError_t ret;
    size_t size;

    ret = hipGetSymbolSize(&size, HIP_SYMBOL(const_symbol));
    ASSERT_EQ(hipSuccess, ret);
    EXPECT_EQ(sizeof(symbol_t), size);
}

TEST(GetSymbolSize, DeviceSymbol) {
    hipError_t ret;
    size_t size;

    ret = hipGetSymbolSize(&size, HIP_SYMBOL(device_symbol));
    ASSERT_EQ(hipSuccess, ret);
    EXPECT_EQ(sizeof(symbol_t), size);
}

int main(int argc, char **argv) {
    ::testing::InitGoogleTest(&argc, argv);
    return RUN_ALL_TESTS();
}
