#include "hip/hip_runtime.h"
/**
 * Panoptes - A Binary Translation Framework for CUDA
 * (c) 2011-2012 Chris Kennelly <chris@ckennelly.com>
 *
 * This program is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.
 */

#include <hip/hip_runtime.h>
#include <gtest/gtest.h>
#include <stdint.h>

extern "C" __global__ void k_clz(const int32_t * in, int * out, int n) {
    for (int idx = threadIdx.x + blockDim.x * blockIdx.x;
            idx < n; idx += blockDim.x * gridDim.x) {
        out[idx] = __clz(in[idx]);
    }
}

extern "C" __global__ void k_clzll(const int64_t * in, int * out, int n) {
    for (int idx = threadIdx.x + blockDim.x * blockIdx.x;
            idx < n; idx += blockDim.x * gridDim.x) {
        out[idx] = __clzll(in[idx]);
    }
}

TEST(kCLZ, Int32) {
    hipError_t ret;
    hipStream_t stream;

    const int N = 1 << 20;
    const int n_blocks = 32;

    int32_t * in;
    int * out;

    ret = hipMalloc((void **) &in, sizeof(*in) * N);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipMalloc((void **) &out, sizeof(*out) * N);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipStreamCreate(&stream);
    ASSERT_EQ(hipSuccess, ret);

    k_clz<<<256, n_blocks, 0, stream>>>(in, out, N);

    ret = hipStreamSynchronize(stream);
    EXPECT_EQ(hipSuccess, ret);

    ret = hipStreamDestroy(stream);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipFree(in);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipFree(out);
    ASSERT_EQ(hipSuccess, ret);
}

TEST(kCLZ, Int64) {
    hipError_t ret;
    hipStream_t stream;

    const int N = 1 << 20;
    const int n_blocks = 32;

    int64_t * in;
    int * out;

    ret = hipMalloc((void **) &in, sizeof(*in) * N);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipMalloc((void **) &out, sizeof(*out) * N);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipStreamCreate(&stream);
    ASSERT_EQ(hipSuccess, ret);

    k_clzll<<<256, n_blocks, 0, stream>>>(in, out, N);

    ret = hipStreamSynchronize(stream);
    EXPECT_EQ(hipSuccess, ret);

    ret = hipStreamDestroy(stream);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipFree(in);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipFree(out);
    ASSERT_EQ(hipSuccess, ret);
}

int main(int argc, char **argv) {
    ::testing::InitGoogleTest(&argc, argv);
    return RUN_ALL_TESTS();
}
