/**
 * Panoptes - A Binary Translation Framework for CUDA
 * (c) 2011-2012 Chris Kennelly <chris@ckennelly.com>
 *
 * This program is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.
 */

#include <boost/scoped_array.hpp>
#include <boost/static_assert.hpp>
#include <hip/hip_runtime.h>
#include <gtest/gtest.h>
#include <stdint.h>
#include <valgrind/memcheck.h>
#include <cstdio>

TEST(StreamWaitEvent, InvalidFlags) {
    hipError_t ret;
    hipEvent_t event;

    ret = hipEventCreate(&event);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipStreamWaitEvent(NULL, event, 1);
    EXPECT_EQ(hipErrorInvalidValue, ret);

    ret = hipEventDestroy(event);
    ASSERT_EQ(hipSuccess, ret);
}

TEST(StreamWaitEvent, NeverRecorded) {
    hipError_t ret;
    hipEvent_t event[2];
    hipStream_t stream;

    ret = hipStreamCreate(&stream);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipEventCreate(&event[0]);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipEventCreate(&event[1]);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipStreamWaitEvent(NULL, event[0], 0);
    EXPECT_EQ(hipSuccess, ret);

    ret = hipEventRecord(event[1], stream);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipEventSynchronize(event[1]);
    EXPECT_EQ(hipSuccess, ret);

    ret = hipEventDestroy(event[0]);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipEventDestroy(event[1]);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipStreamDestroy(stream);
    ASSERT_EQ(hipSuccess, ret);
}

TEST(StreamWaitEvent, Ordering) {
    hipError_t ret;
    hipEvent_t event[2];
    hipStream_t stream[2];

    ret = hipStreamCreate(&stream[0]);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipStreamCreate(&stream[1]);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipEventCreate(&event[0]);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipEventCreate(&event[1]);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipEventRecord(event[0], stream[0]);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipStreamWaitEvent(stream[1], event[0], 0);
    EXPECT_EQ(hipSuccess, ret);

    ret = hipEventRecord(event[1], stream[1]);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipEventSynchronize(event[1]);
    ASSERT_EQ(hipSuccess, ret);

    float ms;
    ret = hipEventElapsedTime(&ms, event[0], event[1]);
    EXPECT_EQ(hipSuccess, ret);

    if (ret == hipSuccess) {
        EXPECT_LE(0.f, ms);
    }

    ret = hipEventDestroy(event[0]);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipEventDestroy(event[1]);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipStreamDestroy(stream[0]);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipStreamDestroy(stream[1]);
    ASSERT_EQ(hipSuccess, ret);
}

TEST(StreamWaitEvent, GlobalOrdering) {
    hipError_t ret;
    hipEvent_t event[2];
    hipStream_t stream[2];

    ret = hipStreamCreate(&stream[0]);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipStreamCreate(&stream[1]);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipEventCreate(&event[0]);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipEventCreate(&event[1]);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipEventRecord(event[0], stream[0]);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipStreamWaitEvent(NULL, event[0], 0);
    EXPECT_EQ(hipSuccess, ret);

    ret = hipEventRecord(event[1], stream[1]);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipEventSynchronize(event[1]);
    ASSERT_EQ(hipSuccess, ret);

    float ms;
    ret = hipEventElapsedTime(&ms, event[0], event[1]);
    EXPECT_EQ(hipSuccess, ret);

    if (ret == hipSuccess) {
        EXPECT_LE(0.f, ms);
    }

    ret = hipEventDestroy(event[0]);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipEventDestroy(event[1]);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipStreamDestroy(stream[0]);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipStreamDestroy(stream[1]);
    ASSERT_EQ(hipSuccess, ret);
}

int main(int argc, char **argv) {
    ::testing::InitGoogleTest(&argc, argv);
    return RUN_ALL_TESTS();
}
