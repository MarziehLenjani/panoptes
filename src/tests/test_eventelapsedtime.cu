/**
 * Panoptes - A Binary Translation Framework for CUDA
 * (c) 2011-2012 Chris Kennelly <chris@ckennelly.com>
 *
 * This program is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.
 */

#include <boost/scoped_array.hpp>
#include <boost/static_assert.hpp>
#include <hip/hip_runtime.h>
#include <gtest/gtest.h>
#include <stdint.h>
#include <valgrind/memcheck.h>
#include <cstdio>

TEST(EventElapsedTime, NullMS) {
    ::testing::FLAGS_gtest_death_test_style = "threadsafe";

    hipError_t ret;
    hipEvent_t event[2];
    hipStream_t stream;

    ret = hipStreamCreate(&stream);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipEventCreate(&event[0]);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipEventCreate(&event[1]);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipEventRecord(event[0], stream);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipEventRecord(event[1], stream);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipEventSynchronize(event[1]);
    ASSERT_EQ(hipSuccess, ret);

    EXPECT_EXIT(
        hipEventElapsedTime(NULL, event[0], event[1]),
        ::testing::KilledBySignal(SIGSEGV), "");

    ret = hipEventDestroy(event[0]);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipEventDestroy(event[1]);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipStreamDestroy(stream);
    ASSERT_EQ(hipSuccess, ret);
}

int main(int argc, char **argv) {
    ::testing::InitGoogleTest(&argc, argv);
    return RUN_ALL_TESTS();
}
