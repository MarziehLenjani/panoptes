/**
 * Panoptes - A Binary Translation Framework for CUDA
 * (c) 2011-2012 Chris Kennelly <chris@ckennelly.com>
 *
 * This program is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.
 */

#include <hip/hip_runtime.h>
#include <gtest/gtest.h>

TEST(MallocHost, MallocFree) {
    hipError_t ret;
    int * ptr;

    ret = hipHostMalloc((void **) &ptr, sizeof(*ptr));
    ASSERT_EQ(hipSuccess, ret);

    ASSERT_FALSE(NULL == ptr);
    *ptr = 0;

    ret = hipHostFree(ptr);
    EXPECT_EQ(hipSuccess, ret);
}

TEST(MallocHost, NullArguments) {
    hipError_t ret;

    ret = hipHostMalloc(NULL, 0);
    EXPECT_EQ(hipErrorInvalidValue, ret);

    ret = hipHostMalloc(NULL, 4);
    EXPECT_EQ(hipErrorInvalidValue, ret);

    ret = hipHostFree(NULL);
    EXPECT_EQ(hipSuccess, ret);
}

TEST(MallocHost, FlagRetrieval) {
    hipError_t ret;
    int * ptr;

    ret = hipHostMalloc((void **) &ptr, sizeof(*ptr));
    ASSERT_EQ(hipSuccess, ret);

    ASSERT_FALSE(NULL == ptr);

    unsigned int flags;
    ret = hipHostGetFlags(&flags, ptr);
    EXPECT_EQ(hipSuccess, ret);

    int device;
    ret = hipGetDevice(&device);
    ASSERT_EQ(hipSuccess, ret);

    struct hipDeviceProp_t prop;
    ret = hipGetDeviceProperties(&prop, device);
    ASSERT_EQ(hipSuccess, ret);

    if (prop.unifiedAddressing) {
        EXPECT_EQ(hipHostMallocMapped, flags);
    } else {
        EXPECT_EQ(hipHostMallocDefault, flags);
    }

    ret = hipHostFree(ptr);
    EXPECT_EQ(hipSuccess, ret);

}

/** TODO:  Mismatched */
/** TODO:  leak detection */

int main(int argc, char **argv) {
    ::testing::InitGoogleTest(&argc, argv);
    return RUN_ALL_TESTS();
}
