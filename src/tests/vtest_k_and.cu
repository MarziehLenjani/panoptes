#include "hip/hip_runtime.h"
/**
 * Panoptes - A Binary Translation Framework for CUDA
 * (c) 2011-2012 Chris Kennelly <chris@ckennelly.com>
 *
 * This program is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.
 */

#include <hip/hip_runtime.h>
#include <gtest/gtest.h>
#include <stdint.h>
#include <valgrind/memcheck.h>

static __global__ void k_and_const11(uint32_t * out) {
    uint32_t _out;
    asm("and.b32 %0, 1234567890, 987654321;\n" : "=r"(_out));
    *out = _out;
}

static __global__ void k_and_constA1(uint32_t * out, uint32_t in) {
    uint32_t _out;
    asm("and.b32 %0, %1, 1234567890;\n" : "=r"(_out) : "r"(in));
    *out = _out;
}

static __global__ void k_and_const1B(uint32_t * out, uint32_t in) {
    uint32_t _out;
    asm("and.b32 %0, 1234567890, %1;\n" : "=r"(_out) : "r"(in));
    *out = _out;
}

TEST(And, BinaryConstant) {
    hipError_t ret;
    hipStream_t stream;

    uint32_t * out;
    ret = hipMalloc((void **) &out, 5 * sizeof(*out));
    ASSERT_EQ(hipSuccess, ret);

    ret = hipStreamCreate(&stream);
    ASSERT_EQ(hipSuccess, ret);

    const uint32_t in = 987654321;
    uint32_t invalid_in = in;
    VALGRIND_MAKE_MEM_UNDEFINED(&invalid_in, sizeof(invalid_in));
    k_and_const11<<<1, 1, 0, stream>>>(out + 0);
    k_and_constA1<<<1, 1, 0, stream>>>(out + 1, in);
    k_and_constA1<<<1, 1, 0, stream>>>(out + 2, invalid_in);
    k_and_const1B<<<1, 1, 0, stream>>>(out + 3, in);
    k_and_const1B<<<1, 1, 0, stream>>>(out + 4, invalid_in);

    ret = hipStreamSynchronize(stream);
    EXPECT_EQ(hipSuccess, ret);

    ret = hipStreamDestroy(stream);
    ASSERT_EQ(hipSuccess, ret);

    uint32_t hout[5];
    ret = hipMemcpy(&hout, out, sizeof(hout), hipMemcpyDeviceToHost);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipFree(out);
    ASSERT_EQ(hipSuccess, ret);

    EXPECT_EQ(1234567890 & 987654321, hout[0]);
    EXPECT_EQ(1234567890 & 987654321, hout[1]);
    EXPECT_EQ(1234567890 & 987654321, hout[1]);

    uint32_t vout[5];
    const int vret = VALGRIND_GET_VBITS(&hout, &vout, sizeof(hout));
    if (vret == 1) {
        EXPECT_EQ(0x00000000, vout[0]);
        EXPECT_EQ(0x00000000, vout[1]);
        EXPECT_EQ(0xFFFFFFFF, vout[2]);
        EXPECT_EQ(0x00000000, vout[3]);
        EXPECT_EQ(0xFFFFFFFF, vout[4]);
    }
}

int main(int argc, char **argv) {
    ::testing::InitGoogleTest(&argc, argv);
    return RUN_ALL_TESTS();
}
