#include "hip/hip_runtime.h"
/**
 * Panoptes - A Binary Translation Framework for CUDA
 * (c) 2011-2012 Chris Kennelly <chris@ckennelly.com>
 *
 * This program is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.
 */

#include <hip/hip_runtime.h>
#include <gtest/gtest.h>

static __global__ void k_copy(int * out, int in) {
    *out = in;
}

TEST(DeviceReset, Simple) {
    hipError_t ret;
    hipStream_t stream;

    int * out;
    ret = hipMalloc((void **) &out, sizeof(*out));
    ASSERT_EQ(hipSuccess, ret);

    ret = hipStreamCreate(&stream);
    ASSERT_EQ(hipSuccess, ret);

    int in = 1;
    k_copy<<<1, 1, 0, stream>>>(out, in);

    ret = hipStreamSynchronize(stream);
    ASSERT_EQ(hipSuccess, ret);

    int hout = 0;
    ret = hipMemcpy(&hout, out, sizeof(hout), hipMemcpyDeviceToHost);
    ASSERT_EQ(hipSuccess, ret);
    ASSERT_EQ(in, hout);

    /* Pervious value of out is now invalidated. */
    ret = hipDeviceReset();
    EXPECT_EQ(hipSuccess, ret);

    ret = hipMalloc((void **) &out, sizeof(*out));
    ASSERT_EQ(hipSuccess, ret);

    ret = hipStreamCreate(&stream);
    ASSERT_EQ(hipSuccess, ret);

    in = 2;
    k_copy<<<1, 1, 0, stream>>>(out, in);

    ret = hipStreamSynchronize(stream);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipMemcpy(&hout, out, sizeof(hout), hipMemcpyDeviceToHost);
    ASSERT_EQ(hipSuccess, ret);
    ASSERT_EQ(in, hout);

    ret = hipFree(out);
    ASSERT_EQ(hipSuccess, ret);
}

TEST(DeviceReset, Alias) {
    /*
     * hipDeviceReset aliases hipDeviceReset.  Calling it should not be
     * harmful.
     */
    (void) hipGetLastError();

    const hipError_t ret = hipDeviceReset();
    EXPECT_EQ(hipSuccess, ret);
}

int main(int argc, char **argv) {
    ::testing::InitGoogleTest(&argc, argv);
    return RUN_ALL_TESTS();
}
