#include "hip/hip_runtime.h"
/**
 * Panoptes - A Binary Translation Framework for CUDA
 * (c) 2011-2012 Chris Kennelly <chris@ckennelly.com>
 *
 * This program is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.
 */

#include <boost/static_assert.hpp>
#include <hip/hip_runtime.h>
#include <gtest/gtest.h>
#include <limits>
#include <stdint.h>
#include <valgrind/memcheck.h>

/**
 * Loads a value from a hard-coded, constant address.
 */
static __global__ void k_ld_const(int2 * out, int in) {
    int2 _out;
    asm volatile(
        "{ .local .u32 l[1];\n"
        "st.local.u32 [0], %2;\n"
        "ld.local.u32 %0, [0];\n"
        "ld.local.u32 %1, [4];\n}" : "=r"(_out.x), "=r"(_out.y) : "r"(in));
    *out = _out;
}

TEST(Load, Constant) {
    hipError_t ret;
    hipStream_t stream;

    int2 * out;
    ret = hipMalloc((void **) &out, sizeof(*out));
    ASSERT_EQ(hipSuccess, ret);

    ret = hipStreamCreate(&stream);
    ASSERT_EQ(hipSuccess, ret);

    const int expected = 5;
    k_ld_const<<<1, 1, 0, stream>>>(out, expected);

    ret = hipStreamSynchronize(stream);
    EXPECT_EQ(hipSuccess, ret);

    ret = hipStreamDestroy(stream);
    ASSERT_EQ(hipSuccess, ret);

    int2 hout;
    ret = hipMemcpy(&hout, out, sizeof(hout), hipMemcpyDeviceToHost);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipFree(out);
    ASSERT_EQ(hipSuccess, ret);

    EXPECT_EQ(expected, hout.x);

    int2 vout;
    const int vret = VALGRIND_GET_VBITS(&hout, &vout, sizeof(hout));
    if (vret == 1) {
        EXPECT_EQ(0x00000000, vout.x);
        EXPECT_EQ(0xFFFFFFFF, vout.y);
    }
}

int main(int argc, char **argv) {
    ::testing::InitGoogleTest(&argc, argv);
    return RUN_ALL_TESTS();
}
