#include "hip/hip_runtime.h"
/**
 * Panoptes - A Binary Translation Framework for CUDA
 * (c) 2011-2012 Chris Kennelly <chris@ckennelly.com>
 *
 * This program is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.
 */

#include <boost/static_assert.hpp>
#include <hip/hip_runtime.h>
#include <gtest/gtest.h>
#include <stdint.h>
#include <valgrind/memcheck.h>
#include <cstdio>

template<typename S, typename T>
static __device__ __inline__ S zip(const T a, const T b) {
    BOOST_STATIC_ASSERT(sizeof(T) == 0);
}

template<>
__device__ __inline__ uint32_t zip(const uint16_t a,
        const uint16_t b) {
    uint32_t ret;
    asm volatile("mov.b32 %0, {%1, %2};\n" : "=r"(ret) : "h"(a), "h"(b));
    return ret;
}

template<>
__device__ __inline__ uint64_t zip(const uint32_t a,
        const uint32_t b) {
    uint64_t ret;
    asm volatile("mov.b64 %0, {%1, %2};\n" : "=l"(ret) : "r"(a), "r"(b));
    return ret;
}

template<typename S, typename T>
__global__ void k_zip(S * d, const T a, const T b) {
    BOOST_STATIC_ASSERT(sizeof(T) == 0);
}

template<>
__global__ void k_zip(uint32_t * d, const uint16_t a, const uint16_t b) {
    *d = zip<uint32_t, uint16_t>(a, b);
}

template<>
__global__ void k_zip(uint64_t * d, const uint32_t a, const uint32_t b) {
    *d = zip<uint64_t, uint32_t>(a, b);
}

TEST(MovTest, Zip32) {
    typedef uint16_t src_t;
    typedef uint32_t dst_t;

    hipError_t ret;

    dst_t * d;
    ret = hipMalloc((void **) &d, sizeof(*d));
    ASSERT_EQ(hipSuccess, ret);

    hipStream_t stream;
    ret = hipStreamCreate(&stream);
    ASSERT_EQ(hipSuccess, ret);

    const src_t a   = 5;
    const src_t b   = 3;
    const dst_t exp = (((dst_t) b) << (sizeof(src_t) * CHAR_BIT)) | ((dst_t) a);

    k_zip<<<1, 1, 0, stream>>>(d, a, b);
    ret = hipStreamSynchronize(stream);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipStreamDestroy(stream);
    ASSERT_EQ(hipSuccess, ret);

    dst_t hd;
    ret = hipMemcpy(&hd, d, sizeof(hd), hipMemcpyDeviceToHost);
    ASSERT_EQ(hipSuccess, ret);

    EXPECT_EQ(exp, hd);

    ret = hipFree(d);
    ASSERT_EQ(hipSuccess, ret);
}

TEST(MovTest, Zip64) {
    typedef uint32_t src_t;
    typedef uint64_t dst_t;

    hipError_t ret;

    dst_t * d;
    ret = hipMalloc((void **) &d, sizeof(*d));
    ASSERT_EQ(hipSuccess, ret);

    hipStream_t stream;
    ret = hipStreamCreate(&stream);
    ASSERT_EQ(hipSuccess, ret);

    const src_t a   = 5;
    const src_t b   = 3;
    const dst_t exp = (((dst_t) b) << (sizeof(src_t) * CHAR_BIT)) | ((dst_t) a);

    k_zip<<<1, 1, 0, stream>>>(d, a, b);
    ret = hipStreamSynchronize(stream);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipStreamDestroy(stream);
    ASSERT_EQ(hipSuccess, ret);

    dst_t hd;
    ret = hipMemcpy(&hd, d, sizeof(hd), hipMemcpyDeviceToHost);
    ASSERT_EQ(hipSuccess, ret);

    EXPECT_EQ(exp, hd);

    ret = hipFree(d);
    ASSERT_EQ(hipSuccess, ret);
}

/**
 * TODO: Test unzip.
 */

int main(int argc, char **argv) {
    ::testing::InitGoogleTest(&argc, argv);
    return RUN_ALL_TESTS();
}
