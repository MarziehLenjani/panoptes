#include "hip/hip_runtime.h"
/**
 * Panoptes - A Binary Translation Framework for CUDA
 * (c) 2011-2012 Chris Kennelly <chris@ckennelly.com>
 *
 * This program is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.
 */

#include <hip/hip_runtime.h>
#include <gtest/gtest.h>
#include <stdint.h>

__global__ void k_sad(const int * x, const int * y, const unsigned int * z,
        unsigned int * out, int32_t n) {
    for (int32_t i = threadIdx.x + blockIdx.x * blockDim.x;
            i < n; i += blockDim.x * gridDim.x) {
        out[i] = __sad(x[i], y[i], z[i]);
    }
}

__global__ void k_sad_allconst(unsigned int * out) {
    unsigned int _out;
    asm("sad.s32 %0, 1, 2, 3;\n" : "=r"(_out));
    *out = _out;
}

TEST(kSAD, SADConstant) {
    hipError_t ret;
    hipStream_t stream;

    unsigned int * out;
    ret = hipMalloc((void **) &out, sizeof(*out));
    ASSERT_EQ(hipSuccess, ret);

    ret = hipStreamCreate(&stream);
    ASSERT_EQ(hipSuccess, ret);

    k_sad_allconst<<<1, 1, 0, stream>>>(out);

    ret = hipStreamSynchronize(stream);
    EXPECT_EQ(hipSuccess, ret);

    ret = hipStreamDestroy(stream);
    ASSERT_EQ(hipSuccess, ret);

    unsigned int hout;
    ret = hipMemcpy(&hout, out, sizeof(hout), hipMemcpyDeviceToHost);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipFree(out);
    ASSERT_EQ(hipSuccess, ret);

    EXPECT_EQ(4u, hout);
}

TEST(kSAD, SAD) {
    hipError_t ret;
    hipStream_t stream;

    const int32_t n = 1 << 24;
    int * x;
    int * y;
    unsigned int * z;
    unsigned int * out;

    ret = hipMalloc((void **) &x, sizeof(*x) * n);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipMalloc((void **) &y, sizeof(*y) * n);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipMalloc((void **) &z, sizeof(*z) * n);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipMalloc((void **) &out, sizeof(*out) * n);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipStreamCreate(&stream);
    ASSERT_EQ(hipSuccess, ret);

    k_sad<<<256, 16, 0, stream>>>(x, y, z, out, n);

    ret = hipStreamSynchronize(stream);
    EXPECT_EQ(hipSuccess, ret);

    ret = hipStreamDestroy(stream);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipFree(x);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipFree(y);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipFree(z);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipFree(out);
    ASSERT_EQ(hipSuccess, ret);
}

__global__ void k_usad(const unsigned int * x, const unsigned int * y,
        const unsigned int * z, unsigned int * out, int32_t n) {
    for (int32_t i = threadIdx.x + blockIdx.x * blockDim.x;
            i < n; i += blockDim.x * gridDim.x) {
        out[i] = __sad(x[i], y[i], z[i]);
    }
}

TEST(kSAD, USAD) {
    hipError_t ret;
    hipStream_t stream;

    const int32_t n = 1 << 24;
    unsigned int * x;
    unsigned int * y;
    unsigned int * z;
    unsigned int * out;

    ret = hipMalloc((void **) &x, sizeof(*x) * n);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipMalloc((void **) &y, sizeof(*y) * n);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipMalloc((void **) &z, sizeof(*z) * n);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipMalloc((void **) &out, sizeof(*out) * n);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipStreamCreate(&stream);
    ASSERT_EQ(hipSuccess, ret);

    k_usad<<<256, 16, 0, stream>>>(x, y, z, out, n);

    ret = hipStreamSynchronize(stream);
    EXPECT_EQ(hipSuccess, ret);

    ret = hipStreamDestroy(stream);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipFree(x);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipFree(y);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipFree(z);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipFree(out);
    ASSERT_EQ(hipSuccess, ret);
}

int main(int argc, char **argv) {
    ::testing::InitGoogleTest(&argc, argv);
    return RUN_ALL_TESTS();
}
