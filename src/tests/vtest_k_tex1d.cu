#include "hip/hip_runtime.h"
/**
 * Panoptes - A Binary Translation Framework for CUDA
 * (c) 2011-2013 Chris Kennelly <chris@ckennelly.com>
 *
 * This program is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.
 */

#include <hip/hip_runtime.h>
#include <gtest/gtest.h>
#include <stdint.h>

texture<int32_t, 1, hipReadModeElementType> tex_src;

extern "C" __global__ void k_readtex1d(void * dst, int32_t bytes) {
    bytes = (bytes + 3) / 4;

    for (int32_t idx = threadIdx.x + blockIdx.x * blockDim.x;
            idx < bytes; idx += blockDim.x * gridDim.x) {
        static_cast<int32_t *>(dst)[idx] = tex1Dfetch(tex_src, idx);
    }
}

TEST(kReadTex1D, ExplicitStream) {
    hipError_t ret;
    hipStream_t stream;

    const uint32_t bytes = 1u << 20;
    int32_t *src;
    int32_t *dst;

    ret = hipMalloc(&src, bytes);
    ASSERT_EQ(hipSuccess, ret);
    ret = hipMalloc(&dst, bytes);
    ASSERT_EQ(hipSuccess, ret);

    const textureReference* texref;
    #if CUDA_VERSION < 5000
    ret = hipGetTextureReference(&texref, HIP_SYMBOL("tex_src"));
    #else
    ret = hipGetTextureReference(&texref, HIP_SYMBOL(&tex_src));
    #endif
    ASSERT_EQ(hipSuccess, ret); 

    struct hipChannelFormatDesc desc;
    desc.f = hipChannelFormatKindSigned;
    desc.x = CHAR_BIT * sizeof(*src);
    desc.y = desc.z = desc.w = 0;
    ret = hipBindTexture(NULL, texref, src, &desc, bytes);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipStreamCreate(&stream);
    ASSERT_EQ(hipSuccess, ret);

    k_readtex1d<<<1, 1, 0, stream>>>(dst, bytes);

    ret = hipStreamSynchronize(stream);
    EXPECT_EQ(hipSuccess, ret);

    ret = hipStreamDestroy(stream);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipUnbindTexture(tex_src);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipFree(src);
    ASSERT_EQ(hipSuccess, ret);
    ret = hipFree(dst);
    ASSERT_EQ(hipSuccess, ret);
}

int main(int argc, char **argv) {
    ::testing::InitGoogleTest(&argc, argv);
    return RUN_ALL_TESTS();
}
