#include "hip/hip_runtime.h"
/**
 * Panoptes - A Binary Translation Framework for CUDA
 * (c) 2011-2012 Chris Kennelly <chris@ckennelly.com>
 *
 * This program is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.
 */

#include <boost/static_assert.hpp>
#include <gtest/gtest.h>
#include <stdint.h>
#include <valgrind/memcheck.h>

typedef int32_t tex_t;
texture<tex_t, 1, hipReadModeElementType> tex_src;

class TextureValues : public ::testing::TestWithParam<int> {
    // Empty Fixture
};

static __global__ void k_set(tex_t * out, int n) {
    for (int i = threadIdx.x + blockIdx.x * blockDim.x; i < n;
            i += blockDim.x * gridDim.x) {
        out[i] = static_cast<tex_t>(i);
    }
}

static __global__ void k_copy(tex_t * out, int n) {
    for (int i = threadIdx.x + blockIdx.x * blockDim.x; i < n;
            i += blockDim.x * gridDim.x) {
        out[i] = tex1Dfetch(tex_src, i);
    }
}

TEST_P(TextureValues, DataCopy) {
    /**
     * Verify we can read the values from a texture.
     */
    const int param = GetParam();
    const int alloc = 1 << param;

    const int n_threads = 256;
    const int n_blocks  = (alloc + n_threads - 1) / n_threads;

    hipError_t ret;
    int device;
    ret = hipGetDevice(&device);
    ASSERT_EQ(hipSuccess, ret);

    struct hipDeviceProp_t prop;
    ret = hipGetDeviceProperties(&prop, device);
    ASSERT_EQ(hipSuccess, ret);

    if (alloc > prop.maxTexture1DLinear) {
        return;
    }

    hipStream_t stream;
    ret = hipStreamCreate(&stream);
    ASSERT_EQ(hipSuccess, ret);

    tex_t *tex;
    ret = hipMalloc((void **) &tex, 2 * sizeof(*tex) * alloc);
    ASSERT_EQ(hipSuccess, ret);

    /* Only half of the values are initialized. */
    k_set<<<n_blocks, n_threads, 0, stream>>>(tex, alloc);
    ret = hipStreamSynchronize(stream);
    ASSERT_EQ(hipSuccess, ret);

    const struct hipChannelFormatDesc desc = hipCreateChannelDesc<tex_t>();
    tex_src.addressMode[0] = hipAddressModeClamp;
    tex_src.filterMode = hipFilterModePoint;
    tex_src.normalized = false;

    ret = hipBindTexture(NULL, tex_src, tex, desc, 2 * sizeof(*tex) * alloc);
    ASSERT_EQ(hipSuccess, ret);

    /* Allocate output. */
    tex_t *out;
    ret = hipMalloc((void **) &out, 2 * sizeof(*out) * alloc);
    ASSERT_EQ(hipSuccess, ret);

    /* Run kernel. */
    k_copy<<<n_blocks, n_threads, 0, stream>>>(out, 2 * alloc);

    ret = hipStreamSynchronize(stream);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipStreamDestroy(stream);
    ASSERT_EQ(hipSuccess, ret);

    std::vector<tex_t> hout(2 * alloc);
    ret = hipMemcpy(&hout[0], out, 2 * sizeof(*out) * alloc,
        hipMemcpyDeviceToHost);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipFree(out);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipUnbindTexture(tex_src);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipFree(tex);
    ASSERT_EQ(hipSuccess, ret);

    if (RUNNING_ON_VALGRIND) {
        /* Check validity bits. */
        std::vector<uint32_t> vout(2 * alloc);
        BOOST_STATIC_ASSERT(sizeof(vout[0]) == sizeof(hout[0]));
        int vret = VALGRIND_GET_VBITS(&hout[0], &vout[0],
            2 * sizeof(hout[0]) * alloc);
        ASSERT_EQ(1, vret);

        bool error = false;
        int i;
        /* First half is fully initialized. */
        for (i = 0; i < alloc; i++) {
            error |= vout[i] != 0;
        }
        EXPECT_FALSE(error);

        /* Second half is uninitialized. */
        for (; i < 2 * alloc; i++) {
            error |= vout[i] != 0xFFFFFFFF;
        }
        EXPECT_FALSE(error);
    }

    bool error = false;
    for (int i = 0; i < alloc; i++) {
        const tex_t expected = static_cast<tex_t>(i);
        error |= expected != hout[i];
    }
    EXPECT_FALSE(error);
}

INSTANTIATE_TEST_CASE_P(TextureInst, TextureValues, ::testing::Range(1, 22));

static __global__ void k_load(tex_t * out, int index) {
    *out = tex1Dfetch(tex_src, index);
}

TEST(Textures, WildLoad) {
    /**
     * Allocate a buffer and initialize it.  Then perform a read from the
     * texture at an uninitialized index.
     */
    if (!(RUNNING_ON_VALGRIND)) {
        return;
    }

    hipError_t ret;
    hipStream_t stream;
    ret = hipStreamCreate(&stream);
    ASSERT_EQ(hipSuccess, ret);

    const size_t n_elements = 1u << 16;
    const size_t n_threads  = 256;
    const size_t n_blocks   = (n_elements + n_threads - 1) / n_threads;

    tex_t *tex;
    ret = hipMalloc((void **) &tex, sizeof(*tex) * n_elements);
    ASSERT_EQ(hipSuccess, ret);

    k_set<<<n_blocks, n_threads, 0, stream>>>(tex, n_elements);
    ret = hipStreamSynchronize(stream);
    ASSERT_EQ(hipSuccess, ret);

    const struct hipChannelFormatDesc desc = hipCreateChannelDesc<tex_t>();
    tex_src.addressMode[0] = hipAddressModeClamp;
    tex_src.filterMode = hipFilterModePoint;
    tex_src.normalized = false;

    ret = hipBindTexture(NULL, tex_src, tex, desc, sizeof(*tex) * n_elements);
    ASSERT_EQ(hipSuccess, ret);

    /* Allocate output. */
    tex_t *out;
    ret = hipMalloc((void **) &out, sizeof(*out));
    ASSERT_EQ(hipSuccess, ret);

    /* Run kernel. */
    int index = 0;
    VALGRIND_MAKE_MEM_UNDEFINED(&index, sizeof(index));
    k_load<<<1, 1, 0, stream>>>(out, index);

    ret = hipStreamSynchronize(stream);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipStreamDestroy(stream);
    ASSERT_EQ(hipSuccess, ret);

    tex_t hout;
    ret = hipMemcpy(&hout, out, sizeof(*out), hipMemcpyDeviceToHost);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipFree(out);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipUnbindTexture(tex_src);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipFree(tex);
    ASSERT_EQ(hipSuccess, ret);

    uint32_t vout;
    BOOST_STATIC_ASSERT(sizeof(vout) == sizeof(hout));
    int vret = VALGRIND_GET_VBITS(&hout, &vout, sizeof(hout));
    EXPECT_EQ(1, vret);
    EXPECT_EQ(0xFFFFFFFF, vout);
}

int main(int argc, char **argv) {
    ::testing::InitGoogleTest(&argc, argv);
    return RUN_ALL_TESTS();
}
