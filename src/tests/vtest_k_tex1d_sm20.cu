#include "hip/hip_runtime.h"
/**
 * Panoptes - A Binary Translation Framework for CUDA
 * (c) 2011-2012 Chris Kennelly <chris@ckennelly.com>
 *
 * This program is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.
 */

#include <gtest/gtest.h>
#include <stdint.h>

typedef int32_t tex_t;
texture<tex_t, 1, hipReadModeElementType> tex_src;

class TextureValues : public ::testing::TestWithParam<int> {
    // Empty Fixture
};

static __global__ void k_set(tex_t * out, int n) {
    for (int i = threadIdx.x + blockIdx.x * blockDim.x; i < n;
            i += blockDim.x * gridDim.x) {
        out[i] = static_cast<tex_t>(i);
    }
}

static __global__ void k_copy(tex_t * out, int n) {
    for (int i = threadIdx.x + blockIdx.x * blockDim.x; i < n;
            i += blockDim.x * gridDim.x) {
        out[i] = tex1Dfetch(tex_src, i);
    }
}

TEST_P(TextureValues, DataCopy) {
    /**
     * Verify we can read the values from a texture.
     */
    const int param = GetParam();
    const int alloc = 1 << param;

    const int n_threads = 256;
    const int n_blocks  = (alloc + n_threads - 1) / n_threads;

    hipError_t ret;
    int device;
    ret = hipGetDevice(&device);
    ASSERT_EQ(hipSuccess, ret);

    struct hipDeviceProp_t prop;
    ret = hipGetDeviceProperties(&prop, device);
    ASSERT_EQ(hipSuccess, ret);

    if (alloc > prop.maxTexture1DLinear) {
        return;
    }

    hipStream_t stream;
    ret = hipStreamCreate(&stream);
    ASSERT_EQ(hipSuccess, ret);

    tex_t *tex;
    ret = hipMalloc((void **) &tex, sizeof(*tex) * alloc);
    ASSERT_EQ(hipSuccess, ret);

    k_set<<<n_blocks, n_threads, 0, stream>>>(tex, alloc);
    ret = hipStreamSynchronize(stream);
    ASSERT_EQ(hipSuccess, ret);

    const struct hipChannelFormatDesc desc = hipCreateChannelDesc<tex_t>();
    tex_src.addressMode[0] = hipAddressModeClamp;
    tex_src.filterMode = hipFilterModePoint;
    tex_src.normalized = false;

    ret = hipBindTexture(NULL, tex_src, tex, desc, sizeof(*tex) * alloc);
    ASSERT_EQ(hipSuccess, ret);

    /* Allocate output. */
    tex_t *out;
    ret = hipMalloc((void **) &out, sizeof(*out) * alloc);
    ASSERT_EQ(hipSuccess, ret);

    /* Run kernel. */
    k_copy<<<n_blocks, n_threads, 0, stream>>>(out, alloc);

    ret = hipStreamSynchronize(stream);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipStreamDestroy(stream);
    ASSERT_EQ(hipSuccess, ret);

    std::vector<tex_t> hout(alloc);
    ret = hipMemcpy(&hout[0], out, sizeof(*out) * alloc,
        hipMemcpyDeviceToHost);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipFree(out);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipUnbindTexture(tex_src);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipFree(tex);
    ASSERT_EQ(hipSuccess, ret);

    bool error = false;
    for (int i = 0; i < alloc; i++) {
        const tex_t expected = static_cast<tex_t>(i);
        error |= expected != hout[i];
    }
    EXPECT_FALSE(error);
}

INSTANTIATE_TEST_CASE_P(TextureInst, TextureValues, ::testing::Range(1, 22));

int main(int argc, char **argv) {
    ::testing::InitGoogleTest(&argc, argv);
    return RUN_ALL_TESTS();
}
