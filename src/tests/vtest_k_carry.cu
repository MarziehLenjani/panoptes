#include "hip/hip_runtime.h"
/**
 * Panoptes - A Binary Translation Framework for CUDA
 * (c) 2011-2012 Chris Kennelly <chris@ckennelly.com>
 *
 * This program is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.
 */

#include <boost/static_assert.hpp>
#include <hip/hip_runtime.h>
#include <gtest/gtest.h>
#include <valgrind/memcheck.h>

template<typename T>
static __device__ __inline__ T wide_add(const T & a, const T & b) {
    BOOST_STATIC_ASSERT(sizeof(T) == 0);
}

template<>
__device__ __inline__ uint2 wide_add(const uint2 & a, const uint2 & b) {
    uint2 ret;
    asm volatile(
        "add.cc.u32 %0, %2, %4;\n"
        "addc.u32 %1, %3, %5;\n" : "=r"(ret.x), "=r"(ret.y) :
        "r"(a.x), "r"(a.y), "r"(b.x), "r"(b.y));
    return ret;
}

template<typename T>
__global__ void k_wide_add(T * d, const T a, const T b) {
    *d = wide_add(a, b);
}

TEST(CarryTest, AddSingle) {
    hipError_t ret;
    hipStream_t stream;

    uint2 * d;
    ret = hipMalloc((void **) &d, sizeof(*d));
    ASSERT_EQ(hipSuccess, ret);

    ret = hipStreamCreate(&stream);
    ASSERT_EQ(hipSuccess, ret);

    const uint2 a   = make_uint2(0xFFFFFFFF, 0x0);
    const uint2 b   = make_uint2(0x00000002, 0x0);
    const uint2 exp = make_uint2(0x00000001, 0x1);

    k_wide_add<<<1, 1, 0, stream>>>(d, a, b);

    ret = hipStreamSynchronize(stream);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipStreamDestroy(stream);
    ASSERT_EQ(hipSuccess, ret);

    uint2 hd;
    BOOST_STATIC_ASSERT(sizeof(hd) == sizeof(*d));

    ret = hipMemcpy(&hd, d, sizeof(*d), hipMemcpyDeviceToHost);
    ASSERT_EQ(hipSuccess, ret);

    /**
     * TODO:  Do not suppress validity bits.
     */
    (void) VALGRIND_MAKE_MEM_DEFINED_IF_ADDRESSABLE(&hd, sizeof(hd));
    EXPECT_EQ(exp.x, hd.x);
    EXPECT_EQ(exp.y, hd.y);

    ret = hipFree(d);
    ASSERT_EQ(hipSuccess, ret);
}

/**
 * TODO:  Add a validity check to see that we propagate invalid bits
 * during a carry operation.
 */

template<typename T>
static __device__ __inline__ T wide_sub(const T & a, const T & b) {
    BOOST_STATIC_ASSERT(sizeof(T) == 0);
}

template<>
__device__ __inline__ uint2 wide_sub(const uint2 & a, const uint2 & b) {
    uint2 ret;
    asm volatile(
        "sub.cc.u32 %0, %2, %4;\n"
        "subc.u32 %1, %3, %5;\n" : "=r"(ret.x), "=r"(ret.y) :
        "r"(a.x), "r"(a.y), "r"(b.x), "r"(b.y));
    return ret;
}

template<typename T>
__global__ void k_wide_sub(T * d, const T a, const T b) {
    *d = wide_sub(a, b);
}

TEST(CarryTest, SubSingle) {
    hipError_t ret;
    hipStream_t stream;

    uint2 * d;
    ret = hipMalloc((void **) &d, sizeof(*d));
    ASSERT_EQ(hipSuccess, ret);

    ret = hipStreamCreate(&stream);
    ASSERT_EQ(hipSuccess, ret);

    const uint2 a   = make_uint2(0x0000000F, 0x1);
    const uint2 b   = make_uint2(0x00000010, 0x0);
    const uint2 exp = make_uint2(0xFFFFFFFF, 0x0);

    k_wide_sub<<<1, 1, 0, stream>>>(d, a, b);

    ret = hipStreamSynchronize(stream);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipStreamDestroy(stream);
    ASSERT_EQ(hipSuccess, ret);

    uint2 hd;
    BOOST_STATIC_ASSERT(sizeof(hd) == sizeof(*d));

    ret = hipMemcpy(&hd, d, sizeof(*d), hipMemcpyDeviceToHost);
    ASSERT_EQ(hipSuccess, ret);

    /**
     * TODO:  Do not suppress validity bits.
     */
    (void) VALGRIND_MAKE_MEM_DEFINED_IF_ADDRESSABLE(&hd, sizeof(hd));
    EXPECT_EQ(exp.x, hd.x);
    EXPECT_EQ(exp.y, hd.y);

    ret = hipFree(d);
    ASSERT_EQ(hipSuccess, ret);
}

template<typename S, typename T>
static __device__ __inline__ S wide_mul(const T & a, const T & b) {
    BOOST_STATIC_ASSERT(sizeof(T) == 0);
}

template<>
__device__ __forceinline__ uint4 wide_mul(const uint2 & a, const uint2 & b) {
    /**
     * This is based on the extended precision multiplication PTX given
     * in the PTX version 3.0 ISA documentation for madc.
     *
     * [r3, r2, r1, r0] = [r5, r4] * [r7, r6]
     */
    uint4 ret;
    asm volatile(
        "mul.lo.u32 %0, %4, %6;\n"
        "mul.hi.u32 %1, %4, %6;\n"
        "mad.lo.cc.u32 %1, %5, %6, %1;\n"
        "madc.hi.u32 %2, %5, %6, 0;\n"
        "mad.lo.cc.u32 %1, %4, %7, %1;\n"
        "madc.hi.cc.u32 %2, %4, %7, %2;\n"
        "addc.u32 %3, 0, 0;\n"
        "mad.lo.cc.u32 %2, %5, %7, %2;\n"
        "madc.hi.u32 %3, %5, %7, %3;\n" :
        "=r"(ret.x), "=r"(ret.y), "=r"(ret.z), "=r"(ret.w) :
        "r"(a.x), "r"(a.y), "r"(b.x), "r"(b.y));
    return ret;
}

template<typename S, typename T>
__global__ void k_wide_mul(S * d, const T a, const T b) {
    *d = wide_mul<S, T>(a, b);
}

TEST(CarryTest, MulSingle) {
    hipError_t ret;
    hipStream_t stream;

    uint4 * d;
    ret = hipMalloc((void **) &d, sizeof(*d));
    ASSERT_EQ(hipSuccess, ret);

    ret = hipStreamCreate(&stream);
    ASSERT_EQ(hipSuccess, ret);

    const uint2 a   = make_uint2(0x00000002, 0x00000001);
    const uint2 b   = make_uint2(0xFFFFFFFF, 0x00000000);
    const uint4 exp = make_uint4(0xFFFFFFFE, 0x00000000, 0x1, 0x0);

    k_wide_mul<<<1, 1, 0, stream>>>(d, a, b);

    ret = hipStreamSynchronize(stream);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipStreamDestroy(stream);
    ASSERT_EQ(hipSuccess, ret);

    uint4 hd;
    BOOST_STATIC_ASSERT(sizeof(hd) == sizeof(*d));

    ret = hipMemcpy(&hd, d, sizeof(*d), hipMemcpyDeviceToHost);
    ASSERT_EQ(hipSuccess, ret);

    /**
     * TODO:  Do not suppress validity bits.
     */
    (void) VALGRIND_MAKE_MEM_DEFINED_IF_ADDRESSABLE(&hd, sizeof(hd));
    EXPECT_EQ(exp.x, hd.x);
    EXPECT_EQ(exp.y, hd.y);
    EXPECT_EQ(exp.z, hd.z);
    EXPECT_EQ(exp.w, hd.w);

    ret = hipFree(d);
    ASSERT_EQ(hipSuccess, ret);
}



int main(int argc, char **argv) {
    ::testing::InitGoogleTest(&argc, argv);
    return RUN_ALL_TESTS();
}
