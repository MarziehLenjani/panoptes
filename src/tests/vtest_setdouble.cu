/**
 * Panoptes - A Binary Translation Framework for CUDA
 * (c) 2011-2012 Chris Kennelly <chris@ckennelly.com>
 *
 * This program is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.
 */

#include <boost/scoped_array.hpp>
#include <boost/static_assert.hpp>
#include <hip/hip_runtime.h>
#include <gtest/gtest.h>
#include <stdint.h>
#include <valgrind/memcheck.h>
#include <cstdio>

TEST(SetDouble, FixedPoint) {
    hipError_t ret;
    const double start = 0.5;

    double device = start;
    ret = cudaSetDoubleForDevice(&device);
    ASSERT_EQ(hipSuccess, ret);

    double host = device;
    ret = cudaSetDoubleForHost(&host);
    ASSERT_EQ(hipSuccess, ret);

    const int cmp = memcmp(&host, &start, sizeof(start));
    ASSERT_EQ(0, cmp);
}

int main(int argc, char **argv) {
    ::testing::InitGoogleTest(&argc, argv);
    return RUN_ALL_TESTS();
}
