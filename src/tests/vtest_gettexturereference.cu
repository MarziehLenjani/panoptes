/**
 * Panoptes - A Binary Translation Framework for CUDA
 * (c) 2011-2012 Chris Kennelly <chris@ckennelly.com>
 *
 * This program is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.
 */

#include <boost/scoped_array.hpp>
#include <boost/static_assert.hpp>
#include <hip/hip_runtime.h>
#include <gtest/gtest.h>
#include <stdint.h>
#include <valgrind/memcheck.h>
#include <cstdio>

texture<int32_t, 1, hipReadModeElementType> tex_src;

TEST(GetTextureReference, Simple) {
    hipError_t ret;
    const struct textureReference * texref;

    ret = hipGetTextureReference(&texref, HIP_SYMBOL("tex_src"));
    ASSERT_EQ(hipSuccess, ret);
}

TEST(GetTextureReference, NonTexture) {
    hipError_t ret;
    const struct textureReference * texref;

    ret = hipGetTextureReference(&texref, HIP_SYMBOL((const char *)) &tex_src);
    ASSERT_EQ(hipErrorInvalidTexture, ret);
}

TEST(GetTextureReference, NullArguments) {
    hipError_t ret;
    const struct textureReference * texref;

    ret = hipGetTextureReference(&texref, HIP_SYMBOL(NULL));
    ASSERT_EQ(hipErrorUnknown, ret);

    ret = hipGetTextureReference(NULL, HIP_SYMBOL(NULL));
    ASSERT_EQ(hipErrorUnknown, ret);
}

int main(int argc, char **argv) {
    ::testing::InitGoogleTest(&argc, argv);
    return RUN_ALL_TESTS();
}
