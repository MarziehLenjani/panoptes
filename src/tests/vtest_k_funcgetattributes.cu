#include "hip/hip_runtime.h"
/**
 * Panoptes - A Binary Translation Framework for CUDA
 * (c) 2011-2012 Chris Kennelly <chris@ckennelly.com>
 *
 * This program is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.
 */

#include <boost/scoped_array.hpp>
#include <boost/static_assert.hpp>
#include <hip/hip_runtime.h>
#include <gtest/gtest.h>
#include <stdint.h>
#include <valgrind/memcheck.h>
#include <cstdio>

extern "C" __global__ void k_noop() {

}

TEST(kNOOP, FuncGetAttributes) {
    struct hipFuncAttributes attr;
    hipError_t ret;

    ret = hipFuncGetAttributes(&attr, reinterpret_cast<const void*>(k_noop));
    ASSERT_EQ(hipSuccess, ret);
}

static void not_a_device_function() {

}

TEST(FuncGetAttributes, HostFunction) {
    struct hipFuncAttributes attr;
    hipError_t ret;

    ret = hipFuncGetAttributes(&attr, reinterpret_cast<const void*>(not_a_device_function));
    ASSERT_EQ(hipErrorInvalidDeviceFunction, ret);
}

TEST(FuncGetAttributes, OtherPointer) {
    struct hipFuncAttributes attr;
    hipError_t ret;

    ret = hipFuncGetAttributes(&attr, reinterpret_cast<const void*>(&attr));
    ASSERT_EQ(hipErrorInvalidDeviceFunction, ret);
}

TEST(FuncGetAttributes, NullArguments) {
    struct hipFuncAttributes attr;
    hipError_t ret;

    ret = hipFuncGetAttributes(NULL, reinterpret_cast<const void*>(k_noop));
    ASSERT_EQ(hipErrorInvalidValue, ret);

    ret = hipFuncGetAttributes(NULL, reinterpret_cast<const void*>(NULL));
    ASSERT_EQ(hipErrorInvalidValue, ret);

    ret = hipFuncGetAttributes(&attr, reinterpret_cast<const void*>(NULL));
    ASSERT_EQ(hipErrorUnknown, ret);
}

int main(int argc, char **argv) {
    ::testing::InitGoogleTest(&argc, argv);
    return RUN_ALL_TESTS();
}
