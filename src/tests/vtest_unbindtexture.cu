/**
 * Panoptes - A Binary Translation Framework for CUDA
 * (c) 2011-2012 Chris Kennelly <chris@ckennelly.com>
 *
 * This program is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.
 */

#include <hip/hip_runtime.h>
#include <gtest/gtest.h>
#include <stdint.h>

texture<int32_t, 1, hipReadModeElementType> tex_src;

TEST(UnbindTexture, DoubleUnbind) {
    hipError_t ret;
    const struct textureReference * texref;

    const uint32_t bytes = 1u << 20;
    int32_t * data;
    ret = hipMalloc((void **) &data, sizeof(*data) * bytes);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipGetTextureReference(&texref, HIP_SYMBOL("tex_src"));
    ASSERT_EQ(hipSuccess, ret);

    struct hipChannelFormatDesc desc;
    desc.f = hipChannelFormatKindSigned;
    desc.x = CHAR_BIT * sizeof(*data);
    desc.y = desc.z = desc.w = 0;
    ret = hipBindTexture(NULL, texref, data, &desc, bytes);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipUnbindTexture(tex_src);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipUnbindTexture(tex_src);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipFree(data);
    ASSERT_EQ(hipSuccess, ret);
}

int main(int argc, char **argv) {
    ::testing::InitGoogleTest(&argc, argv);
    return RUN_ALL_TESTS();
}
