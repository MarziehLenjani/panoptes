#include "hip/hip_runtime.h"
/**
 * Panoptes - A Binary Translation Framework for CUDA
 * (c) 2011-2012 Chris Kennelly <chris@ckennelly.com>
 *
 * This program is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.
 */

#include <boost/static_assert.hpp>
#include <hip/hip_runtime.h>
#include <gtest/gtest.h>
#include <stdint.h>
#include <valgrind/memcheck.h>

extern "C" __global__ void k_copy_scalar_float(float * a, float b) {
    *a = b;
}

extern "C" __global__ void k_copy_scalar_double(double * a, double b) {
    *a = b;
}

struct simple_t {
    double a;
    float  b;
};

TEST(kCopyScalarFloat, ExplicitStream) {
    hipError_t ret;
    hipStream_t stream;
    simple_t * d;

    ret = hipMalloc(&d, sizeof(*d));
    ASSERT_EQ(hipSuccess, ret);

    ret = hipStreamCreate(&stream);
    ASSERT_EQ(hipSuccess, ret);

    k_copy_scalar_double<<<1, 1, 0, stream>>>(&d->a, 0.1);
    k_copy_scalar_float <<<1, 1, 0, stream>>>(&d->b, 0.1f);

    ret = hipStreamSynchronize(stream);
    EXPECT_EQ(hipSuccess, ret);

    ret = hipStreamDestroy(stream);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipFree(d);
    ASSERT_EQ(hipSuccess, ret);
}

int main(int argc, char **argv) {
    ::testing::InitGoogleTest(&argc, argv);
    return RUN_ALL_TESTS();
}
