#include "hip/hip_runtime.h"
/**
 * Panoptes - A Binary Translation Framework for CUDA
 * (c) 2011-2012 Chris Kennelly <chris@ckennelly.com>
 *
 * This program is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.
 */

#include <hip/hip_runtime.h>
#include <gtest/gtest.h>

extern "C" __global__ void k_all_evens(const int * in, bool * out,
        const int N) {
    bool local = true;

    for (int idx = threadIdx.x + blockDim.x * blockIdx.x; idx < N;
            idx += blockDim.x * gridDim.x) {
        local = local & (in[idx] % 2 == 0);
    }

    out[blockIdx.x] = __syncthreads_and(local);
}

extern "C" __global__ void k_any_evens(const int * in, bool * out,
        const int N) {
    bool local = true;

    for (int idx = threadIdx.x + blockDim.x * blockIdx.x; idx < N;
            idx += blockDim.x * gridDim.x) {
        local = local & (in[idx] % 2 == 0);
    }

    out[blockIdx.x] = __syncthreads_or(local);
}

extern "C" __global__ void k_count_evens(const int * in, int * out,
        const int N) {
    const int idx = threadIdx.x + blockDim.x * blockIdx.x;
    bool val = false;
    if (idx < N) {
        val = (in[idx] % 2 == 0);
    }

    out[blockIdx.x] = __syncthreads_count(val);
}

TEST(kSyncThreads, AllEvens) {
    hipError_t ret;
    hipStream_t stream;

    const int N = 1 << 20;
    const int n_blocks = 32;
    const int block_size = 256;

    int * in;
    bool * out;

    ret = hipMalloc((void **) &in, sizeof(*in) * N);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipMalloc((void **) &out, sizeof(*out) * n_blocks);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipStreamCreate(&stream);
    ASSERT_EQ(hipSuccess, ret);

    k_all_evens<<<n_blocks, block_size, 0, stream>>>(in, out, N);

    ret = hipStreamSynchronize(stream);
    EXPECT_EQ(hipSuccess, ret);

    ret = hipStreamDestroy(stream);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipFree(in);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipFree(out);
    ASSERT_EQ(hipSuccess, ret);
}

TEST(kSyncThreads, AnyEvens) {
    hipError_t ret;
    hipStream_t stream;

    const int N = 1 << 20;
    const int n_blocks = 32;
    const int block_size = 256;

    int * in;
    bool * out;

    ret = hipMalloc((void **) &in, sizeof(*in) * N);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipMalloc((void **) &out, sizeof(*out) * n_blocks);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipStreamCreate(&stream);
    ASSERT_EQ(hipSuccess, ret);

    k_any_evens<<<n_blocks, block_size, 0, stream>>>(in, out, N);

    ret = hipStreamSynchronize(stream);
    EXPECT_EQ(hipSuccess, ret);

    ret = hipStreamDestroy(stream);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipFree(in);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipFree(out);
    ASSERT_EQ(hipSuccess, ret);
}

TEST(kSyncThreads, CountEvens) {
    hipError_t ret;
    hipStream_t stream;

    const int N = 1 << 20;
    const int block_size = 256;
    const int n_blocks = 32;

    int * in;
    int * out;

    ret = hipMalloc((void **) &in, sizeof(*in) * N);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipMalloc((void **) &out, sizeof(*out) * N /
        (sizeof(*out) * CHAR_BIT));
    ASSERT_EQ(hipSuccess, ret);

    ret = hipStreamCreate(&stream);
    ASSERT_EQ(hipSuccess, ret);

    k_count_evens<<<block_size, n_blocks, 0, stream>>>(in, out, N);

    ret = hipStreamSynchronize(stream);
    EXPECT_EQ(hipSuccess, ret);

    ret = hipStreamDestroy(stream);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipFree(in);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipFree(out);
    ASSERT_EQ(hipSuccess, ret);
}

int main(int argc, char **argv) {
    ::testing::InitGoogleTest(&argc, argv);
    return RUN_ALL_TESTS();
}
