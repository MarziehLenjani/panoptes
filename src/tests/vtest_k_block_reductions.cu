#include "hip/hip_runtime.h"
/**
 * Panoptes - A Binary Translation Framework for CUDA
 * (c) 2011-2012 Chris Kennelly <chris@ckennelly.com>
 *
 * This program is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.
 */

#include <boost/static_assert.hpp>
#include <gtest/gtest.h>
#include <valgrind/memcheck.h>

extern "C" __global__ void k_all_evens(const int * in, bool * out,
        const int N) {
    bool local = true;

    for (int idx = threadIdx.x + blockDim.x * blockIdx.x; idx < N;
            idx += blockDim.x * gridDim.x) {
        local = local & (in[idx] % 2 == 0);
    }

    out[blockIdx.x] = __syncthreads_and(local);
}

extern "C" __global__ void k_const_all(bool * out) {
    int tmp;
    asm("{ .reg .pred %tmp;\n"
        "bar.red.and.pred %tmp, 0, 1;\n"
        "selp.s32 %0, 1, 0, %tmp;\n}" : "=r"(tmp));
    out[blockIdx.x] = tmp;
}

extern "C" __global__ void k_any_evens(const int * in, bool * out,
        const int N) {
    bool local = true;

    for (int idx = threadIdx.x + blockDim.x * blockIdx.x; idx < N;
            idx += blockDim.x * gridDim.x) {
        local = local & (in[idx] % 2 == 0);
    }

    out[blockIdx.x] = __syncthreads_or(local);
}

extern "C" __global__ void k_const_any(bool * out) {
    int tmp;
    asm("{ .reg .pred %tmp;\n"
        "bar.red.or.pred %tmp, 0, 1;\n"
        "selp.s32 %0, 1, 0, %tmp;\n}" : "=r"(tmp));
    out[blockIdx.x] = tmp;
}

extern "C" __global__ void k_count_evens(const int * in, int * out,
        const int N) {
    const int idx = threadIdx.x + blockDim.x * blockIdx.x;
    bool val = false;
    if (idx < N) {
        val = (in[idx] % 2 == 0);
    }

    out[blockIdx.x] = __syncthreads_count(val);
}

extern "C" __global__ void k_const_count(int * out) {
    int tmp;
    asm("bar.red.popc.u32 %0, 0, 1;" : "=r"(tmp));
    out[blockIdx.x] = tmp;
}

TEST(kSyncThreads, AllEvens) {
    hipError_t ret;
    hipStream_t stream;

    const int N = 1 << 20;
    const int n_blocks = 32;
    const int block_size = 256;

    int * in;
    bool * out;

    ret = hipMalloc((void **) &in, sizeof(*in) * N);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipMalloc((void **) &out, 2 * sizeof(*out) * n_blocks);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipStreamCreate(&stream);
    ASSERT_EQ(hipSuccess, ret);

    k_all_evens<<<n_blocks, block_size, 0, stream>>>(in, out, N);
    k_const_all<<<n_blocks, block_size, 0, stream>>>(out + n_blocks);

    ret = hipStreamSynchronize(stream);
    EXPECT_EQ(hipSuccess, ret);

    ret = hipStreamDestroy(stream);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipFree(in);
    ASSERT_EQ(hipSuccess, ret);

    /* Avoid the std::vector<bool> specialization. */
    std::vector<char> hout(2 * n_blocks);
    BOOST_STATIC_ASSERT(sizeof(hout[0]) == sizeof(*out));
    ret = hipMemcpy(&hout[0], out, 2 * sizeof(*out) * n_blocks,
        hipMemcpyDeviceToHost);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipFree(out);
    ASSERT_EQ(hipSuccess, ret);

    if (RUNNING_ON_VALGRIND) {
        std::vector<unsigned char> vout(2 * n_blocks);
        BOOST_STATIC_ASSERT(sizeof(hout[0]) == sizeof(vout[0]));
        int v = VALGRIND_GET_VBITS(&hout[0], &vout[0],
            2 * sizeof(hout[0]) * n_blocks);
        ASSERT_EQ(1, v);

        bool error = false;
        for (int i = 0; i < n_blocks; i++) {
            /**
             * setp is imprecise and marks the validity bits to be entirely
             * invalid, even when the propagation of uninitialized state may
             * cause a single bitflip.
             */
            error |= vout[i] != 0xFF;
        }
        EXPECT_FALSE(error);

        for (int i = n_blocks; i < 2 * n_blocks; i++) {
            error |= vout[i] != 0x0;
        }
        EXPECT_FALSE(error);
    }

    bool error = false;
    for (int i = n_blocks; i < 2 * n_blocks; i++) {
        error |= hout[i] != 1;
    }
    EXPECT_FALSE(error);
}

TEST(kSyncThreads, AnyEvens) {
    hipError_t ret;
    hipStream_t stream;

    const int N = 1 << 20;
    const int n_blocks = 32;
    const int block_size = 256;

    int * in;
    bool * out;

    ret = hipMalloc((void **) &in, sizeof(*in) * N);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipMalloc((void **) &out, 2 * sizeof(*out) * n_blocks);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipStreamCreate(&stream);
    ASSERT_EQ(hipSuccess, ret);

    k_any_evens<<<n_blocks, block_size, 0, stream>>>(in, out, N);
    k_const_any<<<n_blocks, block_size, 0, stream>>>(out + n_blocks);

    ret = hipStreamSynchronize(stream);
    EXPECT_EQ(hipSuccess, ret);

    ret = hipStreamDestroy(stream);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipFree(in);
    ASSERT_EQ(hipSuccess, ret);

    /* Avoid the std::vector<bool> specialization. */
    std::vector<char> hout(2 * n_blocks);
    BOOST_STATIC_ASSERT(sizeof(hout[0]) == sizeof(*out));
    ret = hipMemcpy(&hout[0], out, 2 * sizeof(*out) * n_blocks,
        hipMemcpyDeviceToHost);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipFree(out);
    ASSERT_EQ(hipSuccess, ret);

    if (RUNNING_ON_VALGRIND) {
        std::vector<unsigned char> vout(2 * n_blocks);
        BOOST_STATIC_ASSERT(sizeof(hout[0]) == sizeof(vout[0]));
        int v = VALGRIND_GET_VBITS(&hout[0], &vout[0],
            2 * sizeof(hout[0]) * n_blocks);
        ASSERT_EQ(1, v);

        bool error = false;
        for (int i = 0; i < n_blocks; i++) {
            /**
             * setp is imprecise and marks the validity bits to be entirely
             * invalid, even when the propagation of uninitialized state may
             * cause a single bitflip.
             */
            error |= vout[i] != 0xFF;
        }
        EXPECT_FALSE(error);

        for (int i = n_blocks; i < 2 * n_blocks; i++) {
            error |= vout[i] != 0x0;
        }
        EXPECT_FALSE(error);
    }

    bool error = false;
    for (int i = n_blocks; i < 2 * n_blocks; i++) {
        error |= hout[i] != 1;
    }
    EXPECT_FALSE(error);
}

TEST(kSyncThreads, CountEvens) {
    hipError_t ret;
    hipStream_t stream;

    const int N = 1 << 20;
    const int block_size = 256;
    const int n_blocks = (N + block_size - 1) / block_size;

    int * in;
    int * out;

    ret = hipMalloc((void **) &in, sizeof(*in) * N);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipMalloc((void **) &out, 2 * sizeof(*out) * n_blocks);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipStreamCreate(&stream);
    ASSERT_EQ(hipSuccess, ret);

    k_count_evens<<<n_blocks, block_size, 0, stream>>>(in, out, N);
    k_const_count<<<n_blocks, block_size, 0, stream>>>(out + n_blocks);

    ret = hipStreamSynchronize(stream);
    EXPECT_EQ(hipSuccess, ret);

    ret = hipStreamDestroy(stream);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipFree(in);
    ASSERT_EQ(hipSuccess, ret);

    std::vector<int> hout(2 * n_blocks);
    ret = hipMemcpy(&hout[0], out, 2 * sizeof(*out) * n_blocks,
        hipMemcpyDeviceToHost);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipFree(out);
    ASSERT_EQ(hipSuccess, ret);

    if (RUNNING_ON_VALGRIND) {
        std::vector<unsigned> vout(2 * n_blocks);
        BOOST_STATIC_ASSERT(sizeof(hout[0]) == sizeof(vout[0]));
        int v = VALGRIND_GET_VBITS(&hout[0], &vout[0],
            2 * sizeof(hout[0]) * n_blocks);
        ASSERT_EQ(1, v);

        /* Round block_size to next power of two. */
        unsigned rbs = block_size;
        rbs--;
        rbs |= rbs >> 1;
        rbs |= rbs >> 2;
        rbs |= rbs >> 4;
        rbs |= rbs >> 8;
        rbs |= rbs >> 16;
        const unsigned mask = (rbs + 1) | rbs;

        bool error = false;
        for (int i = 0; i < n_blocks; i++) {
            EXPECT_EQ(mask, vout[i]);
            error |= vout[i] != mask;
        }
        EXPECT_FALSE(error);

        for (int i = n_blocks; i < 2 * n_blocks; i++) {
            error |= vout[i] != 0;
        }
        EXPECT_FALSE(error);
    }

    bool error = false;
    for (int i = n_blocks; i < 2 * n_blocks; i++) {
        error |= hout[i] != block_size;
    }
    EXPECT_FALSE(error);
}

int main(int argc, char **argv) {
    ::testing::InitGoogleTest(&argc, argv);
    return RUN_ALL_TESTS();
}
