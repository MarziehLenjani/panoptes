#include "hip/hip_runtime.h"
/**
 * Panoptes - A Binary Translation Framework for CUDA
 * (c) 2011-2012 Chris Kennelly <chris@ckennelly.com>
 *
 * This program is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.
 */

#include <boost/scoped_array.hpp>
#include <boost/static_assert.hpp>
#include <hip/hip_runtime.h>
#include <gtest/gtest.h>
#include <stdint.h>
#include <valgrind/memcheck.h>
#include <cstdio>

extern "C" __global__ void k_noop() {

}

TEST(kNOOP, ZeroThreads) {
    hipError_t ret;
    hipStream_t stream;

    ret = hipStreamCreate(&stream);
    ASSERT_EQ(hipSuccess, ret);

    k_noop<<<1, 0, 0, stream>>>();

    ret = hipStreamSynchronize(stream);
    EXPECT_EQ(hipSuccess, ret);

    ret = hipStreamDestroy(stream);
    ASSERT_EQ(hipSuccess, ret);
}

TEST(kNOOP, GlobalZeroThreads) {
    hipError_t ret;

    k_noop<<<1, 0, 0>>>();

    ret = hipDeviceSynchronize();
    EXPECT_EQ(hipSuccess, ret);
}

TEST(kNOOP, ZeroBlocks) {
    hipError_t ret;
    hipStream_t stream;

    ret = hipStreamCreate(&stream);
    ASSERT_EQ(hipSuccess, ret);

    k_noop<<<0, 1, 0, stream>>>();

    ret = hipPeekAtLastError();
    EXPECT_EQ(hipErrorInvalidConfiguration, ret);

    ret = hipStreamSynchronize(stream);
    EXPECT_EQ(hipSuccess, ret);

    ret = hipPeekAtLastError();
    EXPECT_EQ(hipErrorInvalidConfiguration, ret);

    ret = hipStreamDestroy(stream);
    ASSERT_EQ(hipSuccess, ret);

    ret = hipGetLastError();
    EXPECT_EQ(hipErrorInvalidConfiguration, ret);
}

TEST(kNOOP, GlobalZeroBlocks) {
    hipError_t ret;

    k_noop<<<0, 1, 0>>>();

    ret = hipPeekAtLastError();
    EXPECT_EQ(hipErrorInvalidConfiguration, ret);

    /**
     * Why the error doesn't show up at this call isn't clear.
     */
    ret = hipDeviceSynchronize();
    EXPECT_EQ(hipSuccess, ret);

    ret = hipGetLastError();
    EXPECT_EQ(hipErrorInvalidConfiguration, ret);
}



TEST(kNOOP, ExplicitStream) {
    hipError_t ret;
    hipStream_t stream;

    ret = hipStreamCreate(&stream);
    ASSERT_EQ(hipSuccess, ret);

    k_noop<<<1, 1, 0, stream>>>();

    ret = hipStreamSynchronize(stream);
    EXPECT_EQ(hipSuccess, ret);

    ret = hipStreamDestroy(stream);
    ASSERT_EQ(hipSuccess, ret);
}

TEST(kNOOP, GlobalStream) {
    hipError_t ret;

    k_noop<<<1, 1, 0>>>();

    ret = hipDeviceSynchronize();
    EXPECT_EQ(hipSuccess, ret);
}

TEST(kNOOP, ExplicitGlobalStream) {
    hipError_t ret;

    k_noop<<<1, 1, 0, NULL>>>();

    ret = hipDeviceSynchronize();
    EXPECT_EQ(hipSuccess, ret);
}

int main(int argc, char **argv) {
    ::testing::InitGoogleTest(&argc, argv);
    return RUN_ALL_TESTS();
}
